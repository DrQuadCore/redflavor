#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

__global__ void add(int *a, int *b, int *c, int tmp) {
	*c = *a + *b + tmp;
  printf("add\n");
  printf("%d %d\n", *a, tmp);
}

int main() {
	int a, b, c;
	int *d_a, *d_b, *d_c;
	int size = sizeof(int);

	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_b, size);
	hipMalloc((void**)&d_c, size);
	
	a = 0;
	b = 1;
  c = 2;
	printf("[before]%d %d %d\n", a, b, c);
	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);
	
	add<<< 1, 1 >>>(d_a, d_b, d_c, 4);
		
	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);
	printf("[after]%d %d %d\n", a, b, c);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	return 0;
}
