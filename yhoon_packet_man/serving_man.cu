#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <memory.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <stdarg.h>
#include <string.h>
#include <sys/types.h>
#include <sys/socket.h>
#include <sys/time.h>
#include <sys/stat.h>
#include <sys/mman.h>
#include <fcntl.h>
#include <errno.h>
#include <netdb.h>
#include <malloc.h>
#include <getopt.h>
#include <arpa/inet.h>
#include <sys/ioctl.h>
#include <time.h>
#include <asm/types.h>

#include <linux/if_ether.h>
#include <linux/ip.h>
#include <linux/udp.h>
#include "icmp.cu.h"
#include "arp.h" 

#include <linux/netdevice.h>   /* struct device, and other headers */
#include <linux/tcp.h>         /* struct tcphdr */
#include <linux/in6.h>
#define ETH_ALEN  6 // YHOON
#define ARP_PAD_LEN 18 // YHOON
#define IP_HEADER_LEN 20

#ifndef TRUE
#define TRUE (1)
#endif

#ifndef FALSE
#define FALSE (0)
#endif

#ifndef ERROR
#define ERROR (-1)
#endif


#define HTONS(n) (((((unsigned short)(n) & 0xFF)) << 8) | (((unsigned short)(n) & 0xFF00) >> 8))
#define NTOHS(n) (((((unsigned short)(n) & 0xFF)) << 8) | (((unsigned short)(n) & 0xFF00) >> 8))

#define HTONL(n) (((((unsigned long)(n) & 0xFF)) << 24) | \
	((((unsigned long)(n) & 0xFF00)) << 8) | \
	((((unsigned long)(n) & 0xFF0000)) >> 8) | \
		  ((((unsigned long)(n) & 0xFF000000)) >> 24))

#define NTOHL(n) (((((unsigned long)(n) & 0xFF)) << 24) | \
	((((unsigned long)(n) & 0xFF00)) << 8) | \
	((((unsigned long)(n) & 0xFF0000)) >> 8) | \
		  ((((unsigned long)(n) & 0xFF000000)) >> 24))


#define cudaCheckErrors(msg) do { hipError_t __err = hipGetLastError(); if (__err != hipSuccess) { \
			fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
					msg, hipGetErrorString(__err), \
					__FILE__, __LINE__); \
				fprintf(stderr, "*** FAILED - ABORTING\n"); \
				exit(1); \
		} \
	} while (0)

#include <asm/types.h>
using namespace std;

#include "packet_man.h"
#include "mydrv/mydrv.h"
#include "common.hpp"


#define OUT cout


enum mycopy_msg_level {
    MYCOPY_MSG_DEBUG = 1,
    MYCOPY_MSG_INFO,
    MYCOPY_MSG_WARN,
    MYCOPY_MSG_ERROR
};

#define IXGBE_TDT(_i)   (0x06018 + ((_i) * 0x40))

//#define __USE_PKT_MONITOR__
#define NUM_PACKETS 500
#define THREADS_PER_TB 512 

#define RX_TB 0
#define PP_TB 1
#define TX_TB 2

struct my {
  int fd;
};

/*----------------------------------------------------------------------------*/
enum arp_hrd_format
{
	arp_hrd_ethernet = 1
};
/*----------------------------------------------------------------------------*/
enum arp_opcode
{
	arp_op_request = 1, 
	arp_op_reply = 2, 
};

// YHOON
struct arphdr
{
  uint16_t ar_hrd;      /* hardware address format */
  uint16_t ar_pro;      /* protocol address format */
  uint8_t ar_hln;       /* hardware address length */
  uint8_t ar_pln;       /* protocol address length */
  uint16_t ar_op;       /* arp opcode */

  uint8_t ar_sha[ETH_ALEN]; /* sender hardware address */
  uint32_t ar_sip;      /* sender ip address */
  uint8_t ar_tha[ETH_ALEN]; /* targe hardware address */
  uint32_t ar_tip;      /* target ip address */

  uint8_t pad[ARP_PAD_LEN];
} __attribute__ ((packed));

static int my_msg_level = MYCOPY_MSG_ERROR;
static int my_enable_logging = 1;

static void my_msg(enum mycopy_msg_level lvl, const char* fmt, ...)
{
    if (-1 == my_enable_logging) {
        const char *env = getenv("MYCOPY_ENABLE_LOGGING");
        if (env)
            my_enable_logging = 1;
        else
            my_enable_logging = 0;

        env = getenv("MYCOPY_LOG_LEVEL");
        if (env)
            my_msg_level = atoi(env);
    }
    if (my_enable_logging) {
        if (lvl >= my_msg_level) {
            va_list ap;
            va_start(ap, fmt);
            vfprintf(stderr, fmt, ap);
        }
    }
}

#define my_dbg(FMT, ARGS...)  my_msg(MYCOPY_MSG_DEBUG, "DBG:  " FMT, ## ARGS)
#define my_dbgc(C, FMT, ARGS...)  do { static int my_dbg_cnt=(C); if (my_dbg_cnt) { my_dbg(FMT, ## ARGS); --my_dbg_cnt; }} while (0)
#define my_info(FMT, ARGS...) my_msg(MYCOPY_MSG_INFO,  "INFO: " FMT, ## ARGS)
#define my_warn(FMT, ARGS...) my_msg(MYCOPY_MSG_WARN,  "WARN: " FMT, ## ARGS)
#define my_err(FMT, ARGS...)  my_msg(MYCOPY_MSG_ERROR, "ERR:  " FMT, ## ARGS)

#define DBGMSG 1

__device__ uint32_t d_curr_of_processing_queue = 0;
__device__ uint32_t d_processing_queue_size = 8 * 512;

void h_DumpARPPacket(struct arphdr *arph)
{
	uint8_t *t;

	printf("ARP header: \n");
	printf("Hardware type: %d (len: %d), "
			"protocol type: %d (len: %d), opcode: %d\n", 
			//ntohs(arph->ar_hrd), arph->ar_hln, 
			NTOHS(arph->ar_hrd), arph->ar_hln, 
			//ntohs(arph->ar_pro), arph->ar_pln, ntohs(arph->ar_op));
			NTOHS(arph->ar_pro), arph->ar_pln, NTOHS(arph->ar_op));
	t = (uint8_t *)&arph->ar_sip;
	printf("Sender IP: %u.%u.%u.%u, "
			"haddr: %02X:%02X:%02X:%02X:%02X:%02X\n", 
			t[0], t[1], t[2], t[3], 
			arph->ar_sha[0], arph->ar_sha[1], arph->ar_sha[2], 
			arph->ar_sha[3], arph->ar_sha[4], arph->ar_sha[5]);
	t = (uint8_t *)&arph->ar_tip;
	printf("Target IP: %u.%u.%u.%u, "
			"haddr: %02X:%02X:%02X:%02X:%02X:%02X\n", 
			t[0], t[1], t[2], t[3], 
			arph->ar_tha[0], arph->ar_tha[1], arph->ar_tha[2], 
			arph->ar_tha[3], arph->ar_tha[4], arph->ar_tha[5]);
}

__device__ void DumpARPPacket(struct arphdr *arph)
//void DumpARPPacket(struct arphdr *arph)
{
	uint8_t *t;

	printf("ARP header: \n");
	printf("Hardware type: %d (len: %d), "
			"protocol type: %d (len: %d), opcode: %d\n", 
			//ntohs(arph->ar_hrd), arph->ar_hln, 
			NTOHS(arph->ar_hrd), arph->ar_hln, 
			//ntohs(arph->ar_pro), arph->ar_pln, ntohs(arph->ar_op));
			NTOHS(arph->ar_pro), arph->ar_pln, NTOHS(arph->ar_op));
	t = (uint8_t *)&arph->ar_sip;
	printf("Sender IP: %u.%u.%u.%u, "
			"haddr: %02X:%02X:%02X:%02X:%02X:%02X\n", 
			t[0], t[1], t[2], t[3], 
			arph->ar_sha[0], arph->ar_sha[1], arph->ar_sha[2], 
			arph->ar_sha[3], arph->ar_sha[4], arph->ar_sha[5]);
	t = (uint8_t *)&arph->ar_tip;
	printf("Target IP: %u.%u.%u.%u, "
			"haddr: %02X:%02X:%02X:%02X:%02X:%02X\n", 
			t[0], t[1], t[2], t[3], 
			arph->ar_tha[0], arph->ar_tha[1], arph->ar_tha[2], 
			arph->ar_tha[3], arph->ar_tha[4], arph->ar_tha[5]);
}


__device__ uint8_t * EthernetOutput(uint8_t *buf, uint16_t h_proto, unsigned char* src_haddr, unsigned char* dst_haddr, uint16_t iplen)
{
	struct ethhdr *ethh;
	int i;

	ethh = (struct ethhdr *)buf;

#if 0
	printf("dst_hwaddr: %02X:%02X:%02X:%02X:%02X:%02X\n",
				dst_haddr[0], dst_haddr[1], 
				dst_haddr[2], dst_haddr[3], 
				dst_haddr[4], dst_haddr[5]);
	printf("src_hwaddr: %02X:%02X:%02X:%02X:%02X:%02X\n",
				src_haddr[0], src_haddr[1], 
				src_haddr[2], src_haddr[3], 
				src_haddr[4], src_haddr[5]);
#endif

	for (i = 0; i < ETH_ALEN; i++) {
		ethh->h_source[i] = src_haddr[i];
		ethh->h_dest[i] = dst_haddr[i];
	}
	ethh->h_proto = HTONS(h_proto);

	return (uint8_t *)(ethh + 1);
}

__device__ void DumpPacket(uint8_t *buf, int len)
//void DumpPacket(uint8_t *buf, int len)
{
  printf("<<<DumpPacket>>>----------------------------------------------------------------\n");
	struct ethhdr *ethh;
	struct iphdr *iph;
	struct udphdr *udph;
	//struct tcphdr *tcph;
	uint8_t *t;

	ethh = (struct ethhdr *)buf;
	//if (ntohs(ethh->h_proto) != ETH_P_IP) {
	if (NTOHS(ethh->h_proto) != ETH_P_IP) {
		printf("%02X:%02X:%02X:%02X:%02X:%02X -> %02X:%02X:%02X:%02X:%02X:%02X ",
				ethh->h_source[0],
				ethh->h_source[1],
				ethh->h_source[2],
				ethh->h_source[3],
				ethh->h_source[4],
				ethh->h_source[5],
				ethh->h_dest[0],
				ethh->h_dest[1],
				ethh->h_dest[2],
				ethh->h_dest[3],
				ethh->h_dest[4],
				ethh->h_dest[5]);

		//printf("protocol %04hx  \n", ntohs(ethh->h_proto));
		printf("protocol %04hx  \n", NTOHS(ethh->h_proto));

    //if(ntohs(ethh->h_proto) == ETH_P_ARP)
    if(NTOHS(ethh->h_proto) == ETH_P_ARP)
      DumpARPPacket((struct arphdr *) (ethh + 1));
		goto done;
	}

	iph = (struct iphdr *)(ethh + 1);
	udph = (struct udphdr *)((uint32_t *)iph + iph->ihl);
	//tcph = (struct tcphdr *)((uint32_t *)iph + iph->ihl);

	t = (uint8_t *)&iph->saddr;
	printf("%u.%u.%u.%u", t[0], t[1], t[2], t[3]);
	if (iph->protocol == IPPROTO_TCP || iph->protocol == IPPROTO_UDP)
		//printf("(%d)", ntohs(udph->source));
		printf("(%d)", NTOHS(udph->source));

	printf(" -> ");

	t = (uint8_t *)&iph->daddr;
	printf("%u.%u.%u.%u", t[0], t[1], t[2], t[3]);
	if (iph->protocol == IPPROTO_TCP || iph->protocol == IPPROTO_UDP)
		//printf("(%d)", ntohs(udph->dest));
		printf("(%d)", NTOHS(udph->dest));

	//printf(" IP_ID=%d", ntohs(iph->id));
	printf(" IP_ID=%d", NTOHS(iph->id));
	printf(" TTL=%d ", iph->ttl);

	switch (iph->protocol) {
	case IPPROTO_TCP:
		printf("TCP ");
		break;
	case IPPROTO_UDP:
		printf("UDP ");
		break;
	default:
		printf("protocol %d ", iph->protocol);
		goto done;
	}
done:
	printf("len=%d\n", len);
  printf("<<<DumpPacket>>>-----------------------------------END--------------------------\n");

}


__device__ static int ARPOutput(uint8_t * d_tx_pkt_buffer, int opcode, uint32_t src_ip, uint32_t dst_ip, unsigned char *dst_haddr)
{
	if (!dst_haddr)
		return -1;

  //printf("\n\n\n[%s][%d] Enters\n", __FUNCTION__, __LINE__);
  // ckjung: 00:1b:21:bc:11:52
  uint8_t src_haddr[ETH_ALEN] = {0x00, 0x1b, 0x21, 0xbc, 0x11, 0x52};
	struct arphdr *arph = 
    (struct arphdr *)(uintptr_t)EthernetOutput(d_tx_pkt_buffer, ETH_P_ARP, src_haddr, dst_haddr, sizeof(struct arphdr));

	if (!arph) {
    printf("ERROR\n");
		return -1;
	}
	/* Fill arp header */
	arph->ar_hrd = HTONS(arp_hrd_ethernet);
	arph->ar_pro = HTONS(ETH_P_IP);
	arph->ar_hln = ETH_ALEN;
	arph->ar_pln = 4;
	arph->ar_op = HTONS(opcode);

	/* Fill arp body */
#if 0 // HONESTCHOI : TODO
	arph->ar_sip = CONFIG.eths[nif].ip_addr;
#endif 
	arph->ar_sip = src_ip;
	arph->ar_tip = dst_ip;

#if 0 // HONESTCHOI : TODO
	memcpy(arph->ar_sha, CONFIG.eths[nif].haddr, arph->ar_hln);
	if (target_haddr) {
		memcpy(arph->ar_tha, target_haddr, arph->ar_hln);
	} else {
		memcpy(arph->ar_tha, dst_haddr, arph->ar_hln);
	}
#endif
  for(int i=0; i<arph->ar_hln; i++) {
    arph->ar_sha[i] = src_haddr[i];
    arph->ar_tha[i] = dst_haddr[i];
  }
	//memcpy(arph->ar_sha, src_haddr, arph->ar_hln);
  //memcpy(arph->ar_tha, dst_haddr, arph->ar_hln);
	memset(arph->pad, 0, ARP_PAD_LEN);

#if 0
	DumpARPPacket(arph);
#endif

	return 0;
}

__device__ uint32_t offset_for_rx = 512 * 0x1000;
__device__ static volatile uint8_t *tx_tail_for_queue_zero;

__device__ static int ProcessARPRequest(struct arphdr *arph, uint8_t* d_tx_pkt_buffer)
{
  //printf("[%s][%d] Enters", __FUNCTION__, __LINE__);
	ARPOutput(d_tx_pkt_buffer, arp_op_reply, arph->ar_tip, arph->ar_sip, arph->ar_sha);
  return 0;
}

// TODO
__device__ static int ProcessARPReply(struct arphdr *arph)
{
  DumpARPPacket(arph);
	//unsigned char *temp;
	return 0;
}

__device__ int ProcessARPPacket(unsigned char* d_tx_pkt_buffer, unsigned char *pkt_data, int len)
{
	struct arphdr *arph = (struct arphdr *)(pkt_data + sizeof(struct ethhdr));

  switch (NTOHS(arph->ar_op)) {
    case arp_op_request:
      //printf("[%s][%d] arp_op_request\n", __FUNCTION__, __LINE__);
      ProcessARPRequest(arph, d_tx_pkt_buffer);
      break;

    case arp_op_reply:
      //printf("[%s][%d] arp_op_reply\n", __FUNCTION__, __LINE__);
      // TODO
      ProcessARPReply(arph);
      break;

    default:
      printf("[%s][%d] ERROR. KNOWN OP CODE (%d)\n", __FUNCTION__, __LINE__, NTOHS(arph->ar_op));
      DumpPacket(pkt_data, 1500);
      break;
  }

  return 1;
}
uint8_t * h_EthernetOutput(uint8_t *buf, uint16_t h_proto, unsigned char* src_haddr, unsigned char* dst_haddr, uint16_t iplen)
{
	struct ethhdr *ethh;
	int i;

	ethh = (struct ethhdr *)buf;

#if 1
	printf("dst_hwaddr: %02X:%02X:%02X:%02X:%02X:%02X\n",
				dst_haddr[0], dst_haddr[1], 
				dst_haddr[2], dst_haddr[3], 
				dst_haddr[4], dst_haddr[5]);
	printf("src_hwaddr: %02X:%02X:%02X:%02X:%02X:%02X\n",
				src_haddr[0], src_haddr[1], 
				src_haddr[2], src_haddr[3], 
				src_haddr[4], src_haddr[5]);
#endif

	for (i = 0; i < ETH_ALEN; i++) {
		ethh->h_source[i] = src_haddr[i];
		ethh->h_dest[i] = dst_haddr[i];
	}
	ethh->h_proto = HTONS(h_proto);

	return (uint8_t *)(ethh + 1);
}

int my_pin_buffer(my_t g, unsigned long addr, size_t size, uint64_t p2p_token, uint32_t va_space, my_mh_t *handle)
{
    int ret = 0;
    int retcode;

    struct MYDRV_IOC_PIN_BUFFER_PARAMS params;
    params.addr = addr;
    params.size = size;
    params.p2p_token = p2p_token;
    params.va_space = va_space;
    params.handle = 0;
    //params.buf_name = bname;

    retcode = ioctl(g->fd, MYDRV_IOC_PIN_BUFFER, &params);
    if (0 != retcode) {
        ret = errno;
        my_err("ioctl error (errno=%d)\n", ret);
    }
    *handle = params.handle;

    return ret;
}

my_t my_open()
{
    my_t m = NULL;
    //const char *myinode = "/dev/mydrv";
    const char *myinode = "/dev/ixgbe";

    m = (my_t) calloc(1, sizeof(*m));
    if (!m) {
        //my_err("error while allocating memory\n");
        return NULL;
    }

    int fd = open(myinode, O_RDWR);
    if (-1 == fd ) {
        int ret = errno;
        //my_err("error opening driver (errno=%d/%s)\n", ret, strerror(ret));
        free(m);
        return NULL;
    }

    m->fd = fd;

    return m;
}
#if 0
static inline uint32_t myrand(uint64_t *seed) 
{
	*seed = *seed * 1103515245 + 12345;
	return (uint32_t)(*seed >> 32);
}
#endif

int curr_num;
__device__ void d_check_data(int size, volatile int* d_pkt_buffer, volatile int* flag)
{
  printf("[%s][%d] \n", __FUNCTION__ , __LINE__);
  volatile unsigned char * d_mem = (volatile unsigned char*)d_pkt_buffer;
  d_mem += offset_for_rx;

  int pkt_cnt = 0;
  //const int DUMP_SIZE = 30;
  //__threadfence_system();
  //printf("[%s][%d] before waiting %d\n", __FUNCTION__ , __LINE__, *flag);
  //WAIT_ON_MEM_NE(d_mem[2093068], 0);
  //WAIT_ON_MEM(*flag, 1);
  //printf("[%s][%d] after waiting %d\n", __FUNCTION__ , __LINE__, *flag);

#if 1
  while(pkt_cnt < 30000) {
    //printf("[%s][%d] BEFORE WAIT_ON_MEM %d\n", __FUNCTION__ , __LINE__, threadIdx.x);
    int head_index = 12 + 0x1000*threadIdx.x; // + 0x1000*(THREADS_PER_TB * pkt_cnt);
    //WAIT_ON_MEM_NE(d_mem[head_index], 0);
    pkt_cnt++;
    if(*((uint16_t*)&d_mem[head_index]) == 0x0608 || *((uint16_t*)&d_mem[head_index]) == 0x0806 ) {
      printf("[%s][%d] Ethernet type 0x%02x%02x on %dth memory. (threadIdx.x:%d %dpkts)\n", __FUNCTION__ , __LINE__, d_mem[head_index], d_mem[head_index+1], head_index, threadIdx.x, pkt_cnt);
      *((uint16_t*)&d_mem[head_index]) = 0;     
    }
  }
#endif

}

#if 0
__device__ void wait_for_something(volatile int * something_finished)
{
  BEGIN_SINGLE_THREAD_PART {
    while(!*something_finished) {
    }
  }
}
#endif

__device__ unsigned long tail_val;
__device__ volatile int server_done;
#define NUM_TB 1 
#define NUM_THREADS 512 

__global__ void clean_buffer(unsigned char* buffer, unsigned char* buffer2, int size, char* bm_worked_thread, int* tb_status_table) 
{
  for(int i=0; i<size; i++) {
    buffer[i] = 0;
  }
  for(int i=0; i<4*size; i++) {
    buffer2[i] = 0;
  }
  for(int i=0; i<NUM_THREADS; i++) {
    bm_worked_thread[i] = 0;
    tb_status_table[i] = 0;
  }

}

//#define NUM_TURN 100
__device__ int NUM_TURN;
#define COMPILER_BARRIER() asm volatile("" ::: "memory")
#define cpu_to_le32(x) ((__le32)(__swab32)(x))

union ixgbe_adv_tx_desc {
	struct {
		__le64 buffer_addr; /* Address of descriptor's data buf */
		__le32 cmd_type_len;
		__le32 olinfo_status;
	} read;
	struct {
		__le64 rsvd; /* Reserved */
		__le32 nxtseq_seed;
		__le32 status;
	} wb;
};

#define __force
typedef unsigned int u32;

__device__ static inline __sum16 csum_fold(unsigned int csum)
{
	u32 sum = (__force u32)csum;;

	sum += (sum << 16);
	csum = (sum < csum);
	sum >>= 16;
	sum += csum;

	return (__force __sum16)~sum;
}

__device__ static inline __sum16 ip_fast_csum(const void *iph, unsigned int ihl)
{
	const unsigned int *word = (const unsigned int*) iph;
	const unsigned int *stop = word + ihl;
	unsigned int csum = 0;
	int carry;

  uint32_t tmp = 0;
  memcpy(&tmp, (uint16_t*)&word[0], 4);
  csum = tmp;
  memcpy(&tmp, (uint16_t*)&word[1], 4);
	csum += tmp;
	carry = (csum < tmp);
	csum += carry;

  memcpy(&tmp, (uint16_t*)&word[2], 4);
	csum += tmp;
	carry = (csum < tmp);
	csum += carry;

  memcpy(&tmp, (uint16_t*)&word[3], 4);
	csum += tmp;
	carry = (csum < tmp);
	csum += carry;

	word += 4;
	do {
    memcpy(&tmp, (uint16_t*)word, 4);
		csum += tmp;
		carry = (csum < tmp);
		csum += carry;
		word++;
	} while (word != stop);

	return csum_fold(csum);
}


#define IP_NEXT_PTR(iph) ((uint8_t *)iph + (iph->ihl << 2))


__device__ void 
DumpICMPPacket(const char* type, struct icmphdr *icmph, uint32_t saddr, uint32_t daddr)
{
  uint8_t* _saddr = (uint8_t*) &saddr;
  uint8_t* _daddr = (uint8_t*) &daddr;

	printf("ICMP header: \n");
  printf("Type: %d, "
      "Code: %d, ID: %d, Sequence: %d\n", 
      icmph->icmp_type, icmph->icmp_code,
      NTOHS(ICMP_ECHO_GET_ID(icmph)), NTOHS(ICMP_ECHO_GET_SEQ(icmph)));

  printf("Sender IP: %u.%u.%u.%u\n",
      *_saddr++, *_saddr++, *_saddr++, *_saddr);
  printf("Target IP: %u.%u.%u.%u\n",
      *_daddr++, *_daddr++, *_daddr++, *_daddr);

  printf("%s--------------------------------------------\n", type);
  for(int i=0; i<64; i+=2) {
    printf("%x ", *(((uint8_t*)icmph) + i));
    printf("%x ", *(((uint8_t*)icmph) + i+1));
    if(i%20==0)
      printf("\n");
  }
  printf("\n--------------------------------------------\n");
}


__device__ void 
DumpICMPPacket(struct icmphdr *icmph, uint32_t saddr, uint32_t daddr)
{
  uint8_t* _saddr = (uint8_t*) &saddr;
  uint8_t* _daddr = (uint8_t*) &daddr;

	printf("ICMP header: \n");
  printf("Type: %d, "
      "Code: %d, ID: %d, Sequence: %d\n", 
      icmph->icmp_type, icmph->icmp_code,
      NTOHS(ICMP_ECHO_GET_ID(icmph)), NTOHS(ICMP_ECHO_GET_SEQ(icmph)));

  printf("Sender IP: %u.%u.%u.%u\n",
      *_saddr++, *_saddr++, *_saddr++, *_saddr);
  printf("Target IP: %u.%u.%u.%u\n",
      *_daddr++, *_daddr++, *_daddr++, *_daddr);

  printf("--------------------------------------------\n");
  for(int i=0; i<100; i+=2) {
    printf("%x ", *(((uint8_t*)icmph) + i));
    printf("%x ", *(((uint8_t*)icmph) + i+1));
    if(i%20==0)
      printf("\n");
  }
  printf("\n--------------------------------------------\n");
}

#if 0
__device__ void 
DumpICMPPacket(struct icmphdr *icmph, uint8_t* saddr, uint8_t* daddr)
{
	printf("ICMP header: \n");
  printf("Type: %d, "
      "Code: %d, ID: %d, Sequence: %d\n", 
      icmph->icmp_type, icmph->icmp_code,
      NTOHS(ICMP_ECHO_GET_ID(icmph)), NTOHS(ICMP_ECHO_GET_SEQ(icmph)));

  printf("Sender IP: %u.%u.%u.%u\n",
      *saddr++, *saddr++, *saddr++, *saddr);
  printf("Target IP: %u.%u.%u.%u\n",
      *daddr++, *daddr++, *daddr++, *daddr);
}
#endif

__device__ uint8_t *
IPOutputStandalone(unsigned char* d_tx_pkt_buffer, uint8_t protocol, 
		uint16_t ip_id, uint32_t saddr, uint32_t daddr, uint16_t payloadlen)
{
	struct iphdr *iph;
	int nif;
	unsigned char * haddr;
	int rc = -1;

// TODO: when daddr is not known yet.
// This should be done with handling arp reply
#if 0
	nif = GetOutputInterface(daddr);
	if (nif < 0)
		return NULL;

	haddr = GetDestinationHWaddr(daddr);
	if (!haddr) {
#if 0
		uint8_t *da = (uint8_t *)&daddr;
		TRACE_INFO("[WARNING] The destination IP %u.%u.%u.%u "
				"is not in ARP table!\n",
				da[0], da[1], da[2], da[3]);
#endif
		RequestARP(mtcp, daddr, nif, mtcp->cur_ts);
		return NULL;
	}
#endif

  //TODO for now, statically sets mac addrs
  uint8_t src_haddr[ETH_ALEN] = {0x00, 0x1b, 0x21, 0xbc, 0x11, 0x52};
  // hw addr for yoon
  uint8_t dst_haddr[ETH_ALEN] = {0x3c, 0xa3, 0x15, 0x04, 0x88, 0xd3};
	iph = (struct iphdr *)EthernetOutput(d_tx_pkt_buffer, ETH_P_IP, src_haddr, dst_haddr, payloadlen + IP_HEADER_LEN);
	if (!iph) {
		return NULL;
	}

	iph->ihl = IP_HEADER_LEN >> 2;
	iph->version = 4;
	iph->tos = 0;
	iph->tot_len = HTONS(IP_HEADER_LEN + payloadlen);

  // to avoid memory misalignment
  // maybe, __be16 is not recognized as 2bytes by cuda
  // maybe, we need to redevine ip header structure using __align__
	*(uint16_t*)(&(iph->id)) = HTONS(ip_id);

  // XXX ??? why undefined?
#define IP_DF   0x4000
	*(uint16_t*)(&(iph->frag_off)) = (uint16_t)HTONS(IP_DF);	// no fragmentation
	*(uint8_t*)&iph->ttl = 64;
	*(uint8_t*)&iph->protocol = protocol;
	//iph->saddr = saddr;
	memcpy((uint16_t*)&iph->saddr,&saddr,4);
	//*(uint32_t*)&iph->saddr = saddr;
	//*(uint32_t*)&iph->daddr = daddr;
	memcpy((uint16_t*)&iph->daddr,&daddr,4);

  __sum16 tmp = ip_fast_csum(iph, iph->ihl);
	//memcpy((uint16_t*)&iph->check, &tmp, 2);
  //iph->check = 0;
	iph->check = ip_fast_csum(iph, iph->ihl);
	return (uint8_t *)(iph + 1);
}

__device__ static uint16_t
ICMPChecksum(uint16_t *icmph, int len)
{
	uint16_t ret = 0;
	uint32_t sum = 0;
	uint16_t odd_byte;
	
	while (len > 1) {
		sum += *icmph++;
		len -= 2;
	}
	
	if (len == 1) {
		*(uint8_t*)(&odd_byte) = * (uint8_t*)icmph;
		sum += odd_byte;
	}
	
	sum =  (sum >> 16) + (sum & 0xffff);
	sum += (sum >> 16);
	ret =  ~sum;
	return ret; 
}

__device__ static int
ICMPOutput(unsigned char* d_tx_pkt_buffer, uint32_t saddr, uint32_t daddr,
	   uint8_t icmp_type, uint8_t icmp_code, uint16_t icmp_id, uint16_t icmp_seq,
	   uint8_t *icmpd, uint16_t len)
{
	struct icmphdr *icmph;

	icmph = (struct icmphdr *)IPOutputStandalone(
      d_tx_pkt_buffer, IPPROTO_ICMP, 0, saddr, daddr, sizeof(struct icmphdr) + len);
	if (!icmph)
		return -1;
	/* Fill in the icmp header */
	icmph->icmp_type = icmp_type;
	icmph->icmp_code = icmp_code;
	icmph->icmp_checksum = 0;
	ICMP_ECHO_SET_ID(icmph, HTONS(icmp_id));
	ICMP_ECHO_SET_SEQ(icmph, HTONS(icmp_seq));
	
	/* Fill in the icmp data */
	if (len > 0)
		memcpy((void *)(icmph + 1), icmpd, len);
	
#if 1
	//DumpICMPPacket("ICMPChecksum", icmph, saddr, daddr);
	/* Calculate ICMP Checksum with header and data */
	icmph->icmp_checksum = 
		ICMPChecksum((uint16_t *)icmph, sizeof(struct icmphdr) + len);
#endif
	
	//DumpICMPPacket("TX", icmph, saddr, daddr);

#if 0
	if (ICMPChecksum((uint16_t *) icmph, 64) ) {
    printf("ICMPChecksum returns ERROR\n");
  }
#endif
	return 0;
}

__device__ static int 
ProcessICMPECHORequest(unsigned char* d_tx_pkt_buffer, struct iphdr *iph, int len)
{
	int ret = 0;
	struct icmphdr *icmph = (struct icmphdr *) IP_NEXT_PTR(iph);
	
	/* Check correctness of ICMP checksum and send ICMP echo reply */
  // TODO
#if 1 
	if (ICMPChecksum((uint16_t *) icmph, len - (iph->ihl << 2)) ) {
		ret = ERROR;
  }
	else 
#endif
  {
    // RESOLVING MISALINGED ERROR
    uint16_t* _saddr = (uint16_t*)&(iph->saddr);
    uint16_t* _daddr = (uint16_t*)&(iph->daddr);
    uint32_t saddr = 0;
    uint32_t daddr = 0;
    memcpy(&saddr, _saddr, 4);
    memcpy(&daddr, _daddr, 4);

    ICMPOutput(d_tx_pkt_buffer, daddr, saddr, ICMP_ECHOREPLY, 0, 
        NTOHS(ICMP_ECHO_GET_ID(icmph)), NTOHS(ICMP_ECHO_GET_SEQ(icmph)), 
        (uint8_t *) (icmph + 1),
        (uint16_t) (len - (iph->ihl << 2) - sizeof(struct icmphdr)) );
  }

  return ret;
}

__device__ int ProcessICMPPacket(unsigned char* d_tx_pkt_buffer, struct iphdr *iph, int len)
{
  //uint8_t* _saddr = (uint8_t*) &(iph->saddr);
  //uint8_t* _daddr = (uint8_t*) &(iph->daddr);
	struct icmphdr *icmph = (struct icmphdr *) IP_NEXT_PTR(iph);
	int i;
  // TODO : should we do the following?
#if 0
	int to_me = -1;
	
	/* process the icmp messages destined to me */
	for (i = 0; i < CONFIG.eths_num; i++) {
		if (iph->daddr == CONFIG.eths[i].ip_addr) {
			to_me = TRUE;
		}
	}
	
	if (!to_me)
		return TRUE;
#endif
	
  // need to re-align for cuda
#if 0
  uint16_t* _saddr = (uint16_t*)&(iph->saddr);
  uint16_t* _daddr = (uint16_t*)&(iph->daddr);
  uint32_t saddr = 0;
  uint32_t daddr = 0;
  memcpy(&saddr, _saddr, 4);
  memcpy(&daddr, _daddr, 4);
#endif

  switch (icmph->icmp_type) {
        case ICMP_ECHO:
          //printf("[%s][%d] [INFO] ICMP_ECHO received\n", __FUNCTION__, __LINE__);
          //DumpICMPPacket("RX", icmph, saddr, daddr);
          ProcessICMPECHORequest(d_tx_pkt_buffer, iph, len);
          break;
		
        case ICMP_DEST_UNREACH:
          printf("[INFO] ICMP Destination Unreachable message received\n");
          break;
		
        case ICMP_TIME_EXCEEDED:
          printf("[INFO] ICMP Time Exceeded message received\n");
          break;

        default:
          printf("[INFO] Unsupported ICMP message type %x received\n", icmph->icmp_type);
          break;
  }
  return TRUE;
}


__device__ inline int ProcessIPv4Packet(unsigned char* d_tx_pkt_buffer, unsigned char *pkt_data, int len)
{
  //printf("[%s][%d]\n",__FUNCTION__, __LINE__);
	/* check and process IPv4 packets */
	struct iphdr* iph = (struct iphdr *)(pkt_data + sizeof(struct ethhdr));
	int ip_len = NTOHS(iph->tot_len);

	/* drop the packet shorter than ip header */
	if (ip_len < sizeof(struct iphdr))
    // TODO: define ERROR and FALSE
		//return ERROR;
    return -1;

  // TODO: should handle checksum and promiscuous mode
#if 0
#ifndef DISABLE_HWCSUM
	if (mtcp->iom->dev_ioctl != NULL)
		rc = mtcp->iom->dev_ioctl(mtcp->ctx, ifidx, PKT_RX_IP_CSUM, iph);
	if (rc == -1 && ip_fast_csum(iph, iph->ihl))
		return ERROR;
#else
	UNUSED(rc);
	if (ip_fast_csum(iph, iph->ihl))
		return ERROR;
#endif

#if !PROMISCUOUS_MODE
	/* if not promiscuous mode, drop if the destination is not myself */
	if (iph->daddr != CONFIG.eths[ifidx].ip_addr)
		//DumpIPPacketToFile(stderr, iph, ip_len);
		return TRUE;
#endif
#endif 

	// see if the version is correct
	if (iph->version != 0x4 ) {
    // TODO: define ERROR and FALSE
		//return FALSE;
		return -1;
	}
	
  switch (iph->protocol) {
#if 0
		case IPPROTO_TCP:
			return ProcessTCPPacket(mtcp, cur_ts, ifidx, iph, ip_len);
#endif
		case IPPROTO_ICMP:
			return ProcessICMPPacket(d_tx_pkt_buffer, iph, ip_len);
		default:
			/* currently drop other protocols */
      // TODO: define FALSE
			return -1;
      //return FALSE
	}
  //return FALSE
}

__global__ void packet_processor(unsigned char* d_pkt_processing_queue, unsigned char* d_tx_pkt_buffer, int * tb_status_table, volatile int* num_turns, volatile uint8_t* io_addr)
{
  if(blockIdx.x == 0) {
    BEGIN_SINGLE_THREAD_PART {
    // can be placed somewhere else.
      tx_tail_for_queue_zero = io_addr + IXGBE_TDT(0);
      printf("[%s][%d] in pp. (%d, %d)\n", __FUNCTION__, __LINE__, readNoCache(&tb_status_table[PP_TB]), *num_turns );
    } END_SINGLE_THREAD_PART;
      
    while(*num_turns < NUM_TURN) {
      while(!readNoCache(&tb_status_table[PP_TB])) {
      } 
      // currently d_curr_of_processing_queue is fixed to zero.
      unsigned char* rx_packet = &d_pkt_processing_queue[d_curr_of_processing_queue * 512 * 0x1000 + 0x1000*threadIdx.x];
      unsigned char* tx_packet = &d_tx_pkt_buffer[0x1000*threadIdx.x];
      //__threadfence_system();
      if(*(uint16_t*)(rx_packet+12) != 0) {
        struct ethhdr *ethh = (struct ethhdr *)rx_packet;
        u_short ip_proto = NTOHS(ethh->h_proto);
        if (ip_proto == ETH_P_ARP) {
          ProcessARPPacket(tx_packet, rx_packet, 60);
          //printf("[%s][%d] %d thread setting tx packet for ARP\n", __FUNCTION__, __LINE__, threadIdx.x);
        } else if(ip_proto == ETH_P_IP) {
          // TODO: passing len from below
          ProcessIPv4Packet(tx_packet, rx_packet, 1500);
          //printf("[%s][%d] %d thread setting tx packet for IP\n", __FUNCTION__, __LINE__, threadIdx.x);
        } else {
          printf("[%s][%d] %d thread unknown protocol\n", __FUNCTION__, __LINE__, threadIdx.x);
        }
        //DumpPacket((uint8_t*)tx_packet, 60);
        *(uint16_t*)(rx_packet+12) = 0;
      } else {
        //printf("[%s][%d] %d thread not set\n", __FUNCTION__, __LINE__, threadIdx.x);
      }
      //__threadfence();

      tb_status_table[TX_TB] = 1; // set for tx 
      tb_status_table[PP_TB] = 0; // set for pp
    }

    BEGIN_SINGLE_THREAD_PART {
    // can be placed somewhere else.
      printf("[%s][%d] out pp.\n", __FUNCTION__, __LINE__);
    } END_SINGLE_THREAD_PART;
  }
}


__device__ int tx_curr = 0;

#if 0
__global__ void tx_handler(volatile unsigned char* d_pkt_buffer, int * tb_status_table, volatile uint8_t* io_addr, volatile union ixgbe_adv_tx_desc* tx_desc, volatile int* num_turns)
{
  BEGIN_SINGLE_THREAD_PART {
    while(*num_turns < NUM_TURN) {
      while(!readNoCache(&tb_status_table[TX_TB])) { } 
      //printf("TX [%2d,%2d]\n", blockIdx.x, threadIdx.x);
      tb_status_table[TX_TB]=0;

      //volatile unsigned char* tx_packet = &d_pkt_buffer[0x1000*threadIdx.x];
      //DumpPacket((uint8_t*)tx_packet, 60);
#if 1

      COMPILER_BARRIER();
      int tmp = tx_curr;
      //printf("+(%d) ",tmp);
      for(int i=0; i<512; i++) {
        int curr_index = (tmp+ i)%512;
        printf("(%d,%d) ",i, curr_index);

        if(*(uint16_t*)(d_pkt_buffer+ 0x1000*curr_index) != 0) {
          tx_curr = curr_index;
          printf("-(%d) ",tx_curr);
          //printf("%dth memory, tx handler finds a packet to send.\n", curr_index);
          //DumpPacket((uint8_t*)(d_pkt_buffer+0x1000*curr_index), 60);

          // TODO: currently, back to back. batching need to be implemented
          //printf("TX: Try to send packets using %dth tx_desc.\n",curr_index);
          //printf("tx_tail_for_queue_zero:%p\n", tx_tail_for_queue_zero);
          volatile union ixgbe_adv_tx_desc *desc = tx_desc + curr_index;
          //desc->read.olinfo_status = 0xf0002;
          COMPILER_BARRIER();
          struct ethhdr *ethh = (struct ethhdr *) (d_pkt_buffer + 0x1000*curr_index);
          u_short ip_proto = NTOHS(ethh->h_proto);
          if (ip_proto == ETH_P_ARP) {
            desc->read.cmd_type_len |= 60;
            // TODO
            desc->read.olinfo_status = 0xf0000;
          } else if(ip_proto == ETH_P_IP) {
            desc->read.cmd_type_len |= 98;
            // TODO
            // temporal value for ping msgs
            desc->read.olinfo_status = 0x188000;
            //printf("%p %p\n", &(desc->read.cmd_type_len), &(desc->read.olinfo_status));
          } else {
            desc->read.cmd_type_len |= 60;
            desc->read.olinfo_status = 0xf0000;
          }
          tx_tail_for_queue_zero = io_addr + IXGBE_TDT(0);
          *(uint16_t*)(d_pkt_buffer+ 0x1000*(curr_index-1)) = 0;
          *(volatile unsigned long*) tx_tail_for_queue_zero = (unsigned long)(curr_index + 1);
          //uint32_t curr_tx_index_q_zero = *(volatile unsigned int *)tx_tail_for_queue_zero;
          //printf("curr_tx_index_q_zero: %u\n", curr_tx_index_q_zero);

          // TODO cleaning sent data
          //*(uint16_t*)(d_pkt_buffer+ 0x1000*curr_index) = 0;
          //break;
        }
        //printf("\n");
      }
      COMPILER_BARRIER();
#endif
      //__threadfence_system();
    }
    printf("[%s][%d] out tx.\n", __FUNCTION__, __LINE__);
  } END_SINGLE_THREAD_PART;
}

#else

__global__ void tx_handler(volatile unsigned char* d_pkt_buffer, int * tb_status_table, volatile uint8_t* io_addr, volatile union ixgbe_adv_tx_desc* tx_desc, volatile int* num_turns)
{
  while(*num_turns < NUM_TURN) {
    BEGIN_SINGLE_THREAD_PART {
      while(!readNoCache(&tb_status_table[TX_TB])) { } 
      tb_status_table[TX_TB]=0;
    } END_SINGLE_THREAD_PART;
    __syncthreads();
#if 1
    int my_index = threadIdx.x;
    if(*(uint16_t*)(d_pkt_buffer+ 0x1000*my_index) != 0) {
      volatile union ixgbe_adv_tx_desc *desc = tx_desc + my_index;
      struct ethhdr *ethh = (struct ethhdr *) (d_pkt_buffer + 0x1000*my_index);
      u_short ip_proto = NTOHS(ethh->h_proto);
      if (ip_proto == ETH_P_ARP) {
        desc->read.cmd_type_len |= 60;
        // TODO
        desc->read.olinfo_status = 0xf0000;
      } else if(ip_proto == ETH_P_IP) {
        desc->read.cmd_type_len |= 98;
        // TODO
        // temporal value for ping msgs
        desc->read.olinfo_status = 0x188000;
        //printf("%p %p\n", &(desc->read.cmd_type_len), &(desc->read.olinfo_status));
      } else {
        desc->read.cmd_type_len |= 60;
        desc->read.olinfo_status = 0xf0000;
      }
      tx_tail_for_queue_zero = io_addr + IXGBE_TDT(0);
      // TODO: following code is wrong
      *(uint16_t*)(d_pkt_buffer+ 0x1000*(my_index-1)) = 0;
      *(volatile unsigned long*) tx_tail_for_queue_zero = (unsigned long)(my_index + 1);
    }
#endif
  }
  //printf("[%s][%d] out tx.\n", __FUNCTION__, __LINE__);
}
#endif

__global__ void rx_handler(volatile unsigned char* d_pkt_buffer, int * tb_status_table, char *bm_worked_thread, volatile int* num_turns, int fd, unsigned char* d_pkt_processing_queue) // bm: bitmap
{
  *num_turns = 0;
  //volatile unsigned char * d_mem = (volatile unsigned char*)d_pkt_buffer;
  tb_status_table[RX_TB] = 0;
  tb_status_table[PP_TB] = 0;
  tb_status_table[TX_TB] = 0;
  volatile unsigned char* rx_buf = d_pkt_buffer + offset_for_rx;

#if 0
  if(blockIdx.x == 0) {
#if 0
    BEGIN_SINGLE_THREAD_PART {
      printf("Entering rx_handler. (Block ID:%d)\n", blockIdx.x);
      int mem_index = 0; // why 12??
      while(*num_turns < NUM_TURN) { 
        if(readNoCache(((uint16_t*)&rx_buf[mem_index])) != 0 ) {
          //for(int i=0; i<120; i=i+2) {
          //  printf("%d:0x%02x%02x\n", i, rx_buf[mem_index+i],rx_buf[mem_index+i+1]);
          //}
          //printf("1[%2d,%2d] %d, %d\n", blockIdx.x, threadIdx.x, readNoCache((uint16_t*)&rx_buf[mem_index]), mem_index);
          //printf("Setting tb_status_table[1] = 1 %d(%d)\n", (uint16_t)rx_buf[mem_index], mem_index);
          tb_status_table[RX_TB] = 1;
        }
        mem_index += 0x1000;
        //__threadfence_system();

        if(mem_index >= offset_for_rx) {
          mem_index -= offset_for_rx;
        }
      }
    } END_SINGLE_THREAD_PART;
#else
    //printf("Entering rx_handler. (Block ID:%d)\n", blockIdx.x);
    int mem_index = 0x1000*threadIdx.x;
    while(*num_turns < NUM_TURN) { 
      if(readNoCache(((uint16_t*)&rx_buf[mem_index])) != 0 ) {
        //for(int i=0; i<120; i=i+2) {
        //  printf("%d:0x%02x%02x\n", i, rx_buf[mem_index+i],rx_buf[mem_index+i+1]);
        //}
        //printf("1[%2d,%2d] %d, %d\n", blockIdx.x, threadIdx.x, readNoCache((uint16_t*)&rx_buf[mem_index]), mem_index);
        //printf("Setting tb_status_table[1] = 1 %d(%d)\n", (uint16_t)rx_buf[mem_index], threadIdx.x);
        tb_status_table[RX_TB] = 1;
      }
    }
#endif
  } else {
    while(*num_turns < NUM_TURN) {
      while(!readNoCache(&tb_status_table[RX_TB])) { } 
      //__threadfence_system();
      int mem_index = 0x1000 * threadIdx.x;
      if(readNoCache((uint16_t*)&rx_buf[mem_index]) != 0) {
        //printf("RX [%2d,%2d] %d from %d\n", blockIdx.x, threadIdx.x, tb_status_table[blockIdx.x], *num_turns);
        //DumpPacket((uint8_t*)&rx_buf[mem_index], 60);
        memcpy(d_pkt_processing_queue + mem_index,(const void*)(rx_buf + mem_index), 0x1000);
        for(int i=mem_index; i<mem_index+0x1000; i++)
          rx_buf[i] = 0;
        bm_worked_thread[threadIdx.x] = 1;
      }
#if 1
      BEGIN_SINGLE_THREAD_PART {
        //printf("[%2d,%2d] %d from %d\n", blockIdx.x, threadIdx.x, tb_status_table[blockIdx.x], *num_turns);
        //__threadfence_system();
        tb_status_table[RX_TB] = 0; // set for rx
        tb_status_table[PP_TB] = 1; // set for pp
        (*num_turns)++;
        int num_worked_threads = 0;
        for(int i=0; i<NUM_THREADS; i++) {
          if(bm_worked_thread[i]) {
            num_worked_threads++;
            bm_worked_thread[i] = 0;
          }
        }
        //printf("RX [%2d,%2d] %d from %d, num_worked_threads:%d\n", blockIdx.x, threadIdx.x, tb_status_table[RX_TB], *num_turns, num_worked_threads);
      } END_SINGLE_THREAD_PART;
#endif
    }
  }
#else
  
  while(*num_turns < NUM_TURN) {
    //while(!readNoCache(&tb_status_table[RX_TB])) { } 
    //__threadfence_system();
    int mem_index = 0x1000 * threadIdx.x;
    if(readNoCache((uint16_t*)&rx_buf[mem_index]) != 0) {
      //DumpPacket((uint8_t*)&rx_buf[mem_index], 60);
      memcpy(d_pkt_processing_queue + mem_index, (const void*)(rx_buf + mem_index), 0x800);
#if 0
      clock_t start = clock();
      clock_t now;
      for (;;) {
        now = clock();
        clock_t cycles = now > start ? now - start : now + (0xffffffff - start);
        if (cycles >= 10000) {
          break;
        }
      }
#endif
      //printf("RX [%2d,%2d] status:%d num_turn:%d(%d) in buf:%x\n", blockIdx.x, threadIdx.x, tb_status_table[0], *num_turns, NUM_TURN, rx_buf[mem_index]);
      //for(int i=mem_index; i<mem_index+0x1000; i++) {
       // rx_buf[i] = 0;
      //}
      rx_buf[mem_index] = 0;
      rx_buf[mem_index+1] = 0;
      //bm_worked_thread[threadIdx.x] = 1;
      (*num_turns)++;
      tb_status_table[PP_TB] = 1; // set for pp
    }
    //__threadfence_system();
#if 0
    BEGIN_SINGLE_THREAD_PART {
      //printf("[%2d,%2d] %d from %d\n", blockIdx.x, threadIdx.x, tb_status_table[blockIdx.x], *num_turns);
      //__threadfence_system();
      //tb_status_table[RX_TB] = 0; // set for rx
      //tb_status_table[PP_TB] = 1; // set for pp
      int num_worked_threads = 0;
     // for(int i=0; i<NUM_THREADS; i++) {
     //   if(bm_worked_thread[i]) {
     //     num_worked_threads++;
     //     bm_worked_thread[i] = 0;
     //   }
     // }
      //printf("RX [%2d,%2d] %d from %d, num_worked_threads:%d\n", blockIdx.x, threadIdx.x, tb_status_table[RX_TB], *num_turns, num_worked_threads);
    } END_SINGLE_THREAD_PART;
#endif
  }

#endif
}




__global__ void doorbell_test(void * io_addr, void * desc, uint32_t curr, int* d_mem, int size)
{
  printf("[%s][%d]\n", __FUNCTION__, __LINE__);
  if (blockIdx.x == 0) {
    BEGIN_SINGLE_THREAD_PART {
      printf("[%s][%d] in doorbell_test First Block.\n", __FUNCTION__, __LINE__);
      if(desc == 0) {
        printf("[%s][%d]desc==NULL.\n", __FUNCTION__, __LINE__);
        return;
      }
      // ARP call
      uint8_t* pktBuf;
      pktBuf = (uint8_t *)malloc(60);

      // For now, static ip address 
      uint8_t src_tmp[] = {0x0a, 0x00, 0x00, 0x02};
      uint8_t dst_tmp[] = {0x0a, 0x00, 0x00, 0x01};
      uint32_t src_ip;                             
      memcpy(&src_ip, src_tmp, 4);                 
      uint32_t dst_ip;                             
      memcpy(&dst_ip, dst_tmp, 4);  
      unsigned char dst_haddr[ETH_ALEN];
      memset(dst_haddr, 0xFF, ETH_ALEN);

      for(int i=0; i< size/sizeof(int); i++) {
        d_mem[i] = 0;
      }

      for(int i=0; i< 512; i++) {
        memcpy(d_mem+4096*i/4, pktBuf, 60);
      }

      unsigned char *db[12];
      for(int i=0; i<12; i++)
        db[i] = ((unsigned char *)io_addr) + IXGBE_TDT(i);

      COMPILER_BARRIER();
      volatile union ixgbe_adv_tx_desc* tx_desc;
      tx_desc = (union ixgbe_adv_tx_desc*) desc;

      int num_packets = NUM_PACKETS;
      int index = curr;
      for(int i=0; i<curr; i++) {
        tx_desc++;
      }
      for(int i=0; i<num_packets; i++) {
        tx_desc->read.cmd_type_len |= 60;
        tx_desc->read.olinfo_status = 0xf0000;
        tx_desc++;
        index++;
        if(index == 512)
          tx_desc = (union ixgbe_adv_tx_desc*) desc;
      }
      tail_val = (unsigned long)((curr + num_packets) % 512);
      *(volatile unsigned long*)db[0] = tail_val;

      COMPILER_BARRIER();

      //__threadfence_system();
    } END_SINGLE_THREAD_PART;
    
  } else {
#ifndef __USE_PKT_MONITOR__
    // Second Block
    //printf("[%s][%d] in doorbell_test Second Block.\n", __FUNCTION__, __LINE__);
    //d_check_data(size, d_mem, flag);
#endif
  }
}

// YHOON~ for test
int tx_rx_ring_setup()
{
  const char *myinode = "/dev/ixgbe";
  int fd = open(myinode, O_RDWR);
  ioctl(fd, 1);
  return fd;
}

void yhoon_initializer(int fd, void *ixgbe_bar0_host_addr, ixgbe_adv_tx_desc* desc_addr, void **io_addr, void **tx_desc)
{
  const size_t IXGBE_BAR0_SIZE = 4096*8; // A rough calculation

  ixgbe_bar0_host_addr = mmap(0, IXGBE_BAR0_SIZE*5 , PROT_READ | PROT_WRITE, MAP_SHARED, fd, 0);
  ASSERTRT(hipHostRegister(ixgbe_bar0_host_addr, IXGBE_BAR0_SIZE, hipHostRegisterIoMemory));
  ASSERTRT(hipHostGetDevicePointer((void**)io_addr, (void*)ixgbe_bar0_host_addr, 0));
  printf("io_addr: %p\n", *io_addr);

  desc_addr = (ixgbe_adv_tx_desc*)((unsigned char*)ixgbe_bar0_host_addr + IXGBE_BAR0_SIZE);
  ASSERTRT(hipHostRegister(desc_addr, IXGBE_BAR0_SIZE * 4, hipHostRegisterIoMemory));
  //ASSERTRT(hipMalloc(&tx_desc, IXGBE_BAR0_SIZE * 4));
  ASSERTRT(hipHostGetDevicePointer((void**)tx_desc, (void*)desc_addr, 0));
  printf("tx_desc: %p\n", *tx_desc);

}

void yhoon_finalizer(void* ixgbe_bar0_host_addr, ixgbe_adv_tx_desc* desc_addr)
{
  const size_t IXGBE_BAR0_SIZE = 4096*8; // A rough calculation
  hipHostUnregister(desc_addr);
  hipHostUnregister(ixgbe_bar0_host_addr);
  munmap(ixgbe_bar0_host_addr, IXGBE_BAR0_SIZE*5);
}


void yhoon_xmit_arp(int *d_mem, int size, int fd)
{
  printf("[%s][%d]START---yhoon_xmit_arp------------HOST.\n", __FUNCTION__, __LINE__);
  printf("[%s][%d] sizeof tx_desc:%lu \n", __FUNCTION__, __LINE__, sizeof(union ixgbe_adv_tx_desc));
  // sample code for copying address to drivers
  //uint64_t ptr = 1234;
  //ioctl(fd, 0, &ptr);

  //printf("cpu_to_le32 test:%x\n", htonl(60));

  void* dBAR;
  const size_t IXGBE_BAR0_SIZE = 4096*8; // A rough calculation
  void* ixgbe_bar0_host_addr = mmap(0, IXGBE_BAR0_SIZE*5 , PROT_READ | PROT_WRITE, MAP_SHARED, fd, 0);
  ASSERTRT(hipHostRegister(ixgbe_bar0_host_addr, IXGBE_BAR0_SIZE, hipHostRegisterIoMemory));
  ASSERTRT(hipHostGetDevicePointer((void**)&dBAR, (void*)ixgbe_bar0_host_addr, 0));

  unsigned char *db[12];
  for(int i=0; i<1; i++) {
    db[i] = ((unsigned char *)ixgbe_bar0_host_addr) + IXGBE_TDT(i);
#if 0
    printf("db[%d]: %d\n", i, *(volatile unsigned int *)db[i]);
#endif
  }

  ixgbe_adv_tx_desc* desc = (ixgbe_adv_tx_desc*)((unsigned char*)ixgbe_bar0_host_addr + IXGBE_BAR0_SIZE);
  ASSERTRT(hipHostRegister(desc, IXGBE_BAR0_SIZE * 4, hipHostRegisterIoMemory));
  //hipHostRegister(desc, sizeof(ixgbe_adv_tx_desc), hipHostRegisterIoMemory);
  void* tx_desc;
  ASSERTRT(hipMalloc(&tx_desc, IXGBE_BAR0_SIZE * 4));
  if(hipSuccess != hipHostGetDevicePointer((void**)&tx_desc, (void*)desc, 0)) {
    cudaCheckErrors("hipHostGetDevicePointer fails");
  }

	hipStream_t cuda_stream2;
  ASSERT_CUDA(hipStreamCreate(&cuda_stream2));

  uint32_t curr_tx_index_q_zero = *(volatile unsigned int *)db[0];
  printf("curr_tx_index_q_zero: %u\n", curr_tx_index_q_zero);
  doorbell_test<<< 1, 1, 0, cuda_stream2 >>>(dBAR, tx_desc, curr_tx_index_q_zero, d_mem, size);

  //if(hipSuccess != hipDeviceSynchronize())
	//  cudaCheckErrors("doorbell_sync_error!");

  curr_num = curr_tx_index_q_zero;
  hipHostUnregister(desc);
  hipHostUnregister(ixgbe_bar0_host_addr);
  munmap(ixgbe_bar0_host_addr, IXGBE_BAR0_SIZE*5);

  // ~YHOON
  printf("[%s][%d]END-----yhoon_xmit_arp------------HOST.\n", __FUNCTION__, __LINE__);
}

void check_data(int size, unsigned char* h_mem, int* d_A)
{
  printf("[%s][%d] BEGINS--------------------------------------------------------------------------------\n", __FUNCTION__, __LINE__);
  hipMemcpy(h_mem, d_A, size, hipMemcpyDeviceToHost);
#if 0
  for(int outer = 0; outer < size-4095; outer+=4096) {
	if(iter[outer+4095] != 0) {
      int inner = outer;
      while(iter[inner] != 0)
	    printf("%c", iter[inner++]);
	}
  }
#else
  int dirty_cnt = 0;
  uint32_t offset_for_rx = 512 * 4096;
  //const int DUMP_SIZE = 30;
  for(int i = offset_for_rx; i < size; i++) {
    if(*((uint16_t*)&h_mem[i]) == 0x0608 || *((uint16_t*)&h_mem[i]) == 0x0008 ) {//(h_mem[i] == 0x08 && h_mem[i+1] == 0x00) ) {
      i = i;
      //printf("\n%s][%d] PACKET BEGINS-------------------------------------\n", __FUNCTION__, __LINE__);
      printf("[%s][%d] We found ethernet type 0x%02x%02x on %dth memory.\n", __FUNCTION__ , __LINE__, h_mem[i], h_mem[i+1],i-offset_for_rx);
      printf("[%s][%d] We found ethernet type 0x%04x on %dth memory.\n", __FUNCTION__ , __LINE__, *((uint16_t*)&h_mem[i]), i-offset_for_rx);
    } else if (h_mem[i] == 0xAB || h_mem[i] == 0xBA) {
      for(int packet_iter = i; packet_iter < (i+20); packet_iter++) {
        if((packet_iter-i) % 4 == 0)
          printf("\n");
        printf("%3d:0x%02x\t", packet_iter-i, h_mem[packet_iter]);
      }
	}
  }
  for(int i = 0; i < size; i++) 
    if(h_mem[i] != 0) 
      dirty_cnt++;
#endif
  printf("[%s][%d] dirty_cnt:[%d]\n", __FUNCTION__ , __LINE__, dirty_cnt);
  printf("[%s][%d] ENDS\n\n\n\n", __FUNCTION__, __LINE__);
}

__global__ void init_data(int num_turn)
{
  NUM_TURN = num_turn;  
}

int main(int argc, char *argv[])
{
  int num_turn = 10;
#if 1
  if(argc != 1) {
    num_turn = atoi(argv[1]);
  }
#endif

  printf("[%s][%d] main\n", __FUNCTION__, __LINE__);
  int dev_id = 0;
  size_t _pkt_buffer_size = 2*512*4096; // 4MB, for rx,tx ring

  // CKJUNG, meaning of this?
  size_t pkt_buffer_size = (_pkt_buffer_size + GPU_PAGE_SIZE - 1) & GPU_PAGE_MASK;
  printf("[%s][%d]____CKJUNG__pkt_buffer_size: %lu\n", __FUNCTION__, __LINE__, pkt_buffer_size);

  int n_devices = 0;

  ASSERTRT(hipGetDeviceCount(&n_devices));

  hipDeviceProp_t prop;
  for (int n=0; n<n_devices; ++n) {
    hipGetDeviceProperties(&prop,n);
    OUT << "GPU id:" << n << " name:" << prop.name 
      << " PCI domain: " << prop.pciDomainID 
      << " bus: " << prop.pciBusID 
      << " device: " << prop.pciDeviceID << endl;
  }
  OUT << "selecting device " << dev_id << endl;
  OUT << "_pkt_buffer_size: " << _pkt_buffer_size << "  pkt_buffer_size: " << pkt_buffer_size << endl;

  ASSERTRT(hipSetDevice(dev_id));
  ASSERTRT(hipSetDeviceFlags(hipDeviceMapHost));

  unsigned char* d_pkt_buffer;
  unsigned char* d_pkt_processing_queue;
  ASSERTRT(hipMalloc((void**)&d_pkt_buffer, pkt_buffer_size));
  ASSERTRT(hipMalloc((void**)&d_pkt_processing_queue, 4*pkt_buffer_size));
  ASSERTRT(hipMemset(d_pkt_buffer, 0, pkt_buffer_size));
  ASSERTRT(hipMemset(d_pkt_processing_queue, 0, 4*pkt_buffer_size));
 
  unsigned int flag = 1;
  ASSERTDRV(hipPointerSetAttribute(&flag, HIP_POINTER_ATTRIBUTE_SYNC_MEMOPS, (hipDeviceptr_t) d_pkt_buffer));

  my_t g = my_open();

  ASSERT_NEQ(g, (void*)0);

  my_mh_t mh;
  if (my_pin_buffer(g, (hipDeviceptr_t)d_pkt_buffer, pkt_buffer_size, 0, 0, &mh)  != 0)
    OUT << "NOT_EQ" << endl;

#if 0
  int *h_tmp = (int*) malloc(sizeof(int) * pkt_buffer_size);
  for(int i=0; i<pkt_buffer_size; i++)
    h_tmp[i] = 0;

  hipMemcpy((void *)d_pkt_buffer, (const void*)h_tmp, pkt_buffer_size * sizeof(int), hipMemcpyHostToDevice);

  for(int i=512*4096; i<512*4096*2; i++) {
    if(h_tmp[i] != 0) {
      printf("I%d %c\n", i, (uint16_t)h_tmp[i]);
    }
  }
  hipMemcpy((void*)h_tmp, (const void *)d_pkt_buffer, pkt_buffer_size * sizeof(int), hipMemcpyDeviceToHost);

  for(int i=512*4096; i<512*4096*2; i++) {
    if(h_tmp[i] != 0) {
      printf("O%d %c\n", i, (uint16_t)h_tmp[i]);
    }
  }
#endif 

  void *ixgbe_bar0_host_addr = 0, *io_addr=0, *tx_desc=0;
  ixgbe_adv_tx_desc* desc_addr=0;
  int fd = tx_rx_ring_setup();
  yhoon_initializer(fd, ixgbe_bar0_host_addr, desc_addr, &io_addr, &tx_desc);
  init_data<<< 1,1>>> (num_turn);

	hipStream_t cuda_stream1;
  //ASSERT_CUDA(hipStreamCreate(&cuda_stream1));
  ASSERT_CUDA(hipStreamCreateWithFlags(&cuda_stream1,hipStreamNonBlocking));
	hipStream_t cuda_stream2;
  //ASSERT_CUDA(hipStreamCreate(&cuda_stream2));
  ASSERT_CUDA(hipStreamCreateWithFlags(&cuda_stream2,hipStreamNonBlocking));
	hipStream_t cuda_stream3;
  //ASSERT_CUDA(hipStreamCreate(&cuda_stream3));
  ASSERT_CUDA(hipStreamCreateWithFlags(&cuda_stream3,hipStreamNonBlocking));

	int *dev_tb_status_table, *num_turns;
  char *bm_worked_thread;
	ASSERT_CUDA(hipMalloc(&dev_tb_status_table, NUM_THREADS * sizeof(*dev_tb_status_table)));
	ASSERT_CUDA(hipMemset(dev_tb_status_table, 0, NUM_THREADS* sizeof(*dev_tb_status_table)));
  ASSERTRT(hipMalloc((void**)&num_turns, sizeof(int)));
  ASSERTRT(hipMalloc((void**)&bm_worked_thread, NUM_THREADS * sizeof(char)));

  clean_buffer<<< 1, 1 >>> (d_pkt_buffer, d_pkt_processing_queue, pkt_buffer_size, bm_worked_thread, dev_tb_status_table);

  if(hipSuccess != hipDeviceSynchronize())
	  cudaCheckErrors("hipDeviceSynchronize Error"); 

#if 1
  printf("PP\n");
  packet_processor<<< NUM_TB, NUM_THREADS, 0, cuda_stream2 >>> (d_pkt_processing_queue, d_pkt_buffer, dev_tb_status_table, num_turns, (volatile uint8_t *)io_addr);
  //usleep(1*1000*1000);
  printf("RX\n");
  rx_handler<<< NUM_TB, NUM_THREADS, 0, cuda_stream1 >>> (d_pkt_buffer, dev_tb_status_table, bm_worked_thread, num_turns, fd, d_pkt_processing_queue);
  //usleep(1*1000*1000);
  printf("TX\n");
  tx_handler<<< NUM_TB, NUM_THREADS, 0, cuda_stream3 >>> (d_pkt_buffer, dev_tb_status_table, (volatile uint8_t*)io_addr, (volatile union ixgbe_adv_tx_desc*) tx_desc, num_turns);
#endif

  // call ixgbe_xmit_yhoon in ixgbe_main.c
  //yhoon_xmit_arp(d_pkt_buffer, pkt_buffer_size, fd);

#if 0
  int one = 1;
  usleep(1*1000*1000);
  hipMemcpyToSymbol(HIP_SYMBOL(server_done), &one, sizeof(int));

  unsigned char* h_mem = (unsigned char*)malloc(pkt_buffer_size*sizeof(unsigned char)); 
  hipMemcpy(d_pkt_buffer, h_mem, pkt_buffer_size, hipMemcpyHostToDevice);
  int count = 0;
  while(count < 1000) {
    check_data(pkt_buffer_size, h_mem, d_pkt_buffer);
    usleep(1*1000*1000);
    count++;
  }
#endif

 
  if(hipSuccess != hipDeviceSynchronize())
	  cudaCheckErrors("hipDeviceSynchronize Error"); 
  hipStreamDestroy(cuda_stream1);
  hipStreamDestroy(cuda_stream2);
  hipStreamDestroy(cuda_stream3);
  yhoon_finalizer(ixgbe_bar0_host_addr, desc_addr);

  ASSERT_CUDA(hipFree(dev_tb_status_table));
  ASSERT_CUDA(hipFree(d_pkt_buffer));
  return 0;
}

#undef IP_NEXT_PTR
