#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <memory.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <stdarg.h>
#include <string.h>
#include <sys/types.h>
#include <sys/socket.h>
#include <sys/time.h>
#include <sys/stat.h>
#include <sys/mman.h>
#include <fcntl.h>
#include <errno.h>
#include <netdb.h>
#include <malloc.h>
#include <getopt.h>
#include <arpa/inet.h>
#include <sys/ioctl.h>
#include <time.h>
#include <asm/types.h>

#include <linux/if_ether.h>
#include <linux/ip.h>
#include <linux/udp.h>
#include "arp.h" 

#include <linux/netdevice.h>   /* struct device, and other headers */
#include <linux/tcp.h>         /* struct tcphdr */
#include <linux/in6.h>
#define ETH_ALEN  6 // YHOON
#define ARP_PAD_LEN 18 // YHOON

#define HTONS(n) (((((unsigned short)(n) & 0xFF)) << 8) | (((unsigned short)(n) & 0xFF00) >> 8))
#define NTOHS(n) (((((unsigned short)(n) & 0xFF)) << 8) | (((unsigned short)(n) & 0xFF00) >> 8))

#define HTONL(n) (((((unsigned long)(n) & 0xFF)) << 24) | \
	((((unsigned long)(n) & 0xFF00)) << 8) | \
	((((unsigned long)(n) & 0xFF0000)) >> 8) | \
		  ((((unsigned long)(n) & 0xFF000000)) >> 24))

#define NTOHL(n) (((((unsigned long)(n) & 0xFF)) << 24) | \
	((((unsigned long)(n) & 0xFF00)) << 8) | \
	((((unsigned long)(n) & 0xFF0000)) >> 8) | \
		  ((((unsigned long)(n) & 0xFF000000)) >> 24))


#define cudaCheckErrors(msg) do { hipError_t __err = hipGetLastError(); if (__err != hipSuccess) { \
			fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
					msg, hipGetErrorString(__err), \
					__FILE__, __LINE__); \
				fprintf(stderr, "*** FAILED - ABORTING\n"); \
				exit(1); \
		} \
	} while (0)

#include <asm/types.h>
using namespace std;

#include "packet_man.h"
#include "mydrv/mydrv.h"
#include "common.hpp"


#define OUT cout


#define HTONS(n) (((((unsigned short)(n) & 0xFF)) << 8) | (((unsigned short)(n) & 0xFF00) >> 8))
#define NTOHS(n) (((((unsigned short)(n) & 0xFF)) << 8) | (((unsigned short)(n) & 0xFF00) >> 8))
enum mycopy_msg_level {
    MYCOPY_MSG_DEBUG = 1,
    MYCOPY_MSG_INFO,
    MYCOPY_MSG_WARN,
    MYCOPY_MSG_ERROR
};

#define IXGBE_TDT(_i)   (0x06018 + ((_i) * 0x40))

#define NUM_PACKETS 50
#define THREADS_PER_TB 512 

struct my {
  int fd;
};

/*----------------------------------------------------------------------------*/
enum arp_hrd_format
{
	arp_hrd_ethernet = 1
};
/*----------------------------------------------------------------------------*/
enum arp_opcode
{
	arp_op_request = 1, 
	arp_op_reply = 2, 
};

// YHOON
struct arphdr
{
  uint16_t ar_hrd;      /* hardware address format */
  uint16_t ar_pro;      /* protocol address format */
  uint8_t ar_hln;       /* hardware address length */
  uint8_t ar_pln;       /* protocol address length */
  uint16_t ar_op;       /* arp opcode */

  uint8_t ar_sha[ETH_ALEN]; /* sender hardware address */
  uint32_t ar_sip;      /* sender ip address */
  uint8_t ar_tha[ETH_ALEN]; /* targe hardware address */
  uint32_t ar_tip;      /* target ip address */

  uint8_t pad[ARP_PAD_LEN];
} __attribute__ ((packed));

static int my_msg_level = MYCOPY_MSG_ERROR;
static int my_enable_logging = 1;

static void my_msg(enum mycopy_msg_level lvl, const char* fmt, ...)
{
    if (-1 == my_enable_logging) {
        const char *env = getenv("MYCOPY_ENABLE_LOGGING");
        if (env)
            my_enable_logging = 1;
        else
            my_enable_logging = 0;

        env = getenv("MYCOPY_LOG_LEVEL");
        if (env)
            my_msg_level = atoi(env);
    }
    if (my_enable_logging) {
        if (lvl >= my_msg_level) {
            va_list ap;
            va_start(ap, fmt);
            vfprintf(stderr, fmt, ap);
        }
    }
}

#define my_dbg(FMT, ARGS...)  my_msg(MYCOPY_MSG_DEBUG, "DBG:  " FMT, ## ARGS)
#define my_dbgc(C, FMT, ARGS...)  do { static int my_dbg_cnt=(C); if (my_dbg_cnt) { my_dbg(FMT, ## ARGS); --my_dbg_cnt; }} while (0)
#define my_info(FMT, ARGS...) my_msg(MYCOPY_MSG_INFO,  "INFO: " FMT, ## ARGS)
#define my_warn(FMT, ARGS...) my_msg(MYCOPY_MSG_WARN,  "WARN: " FMT, ## ARGS)
#define my_err(FMT, ARGS...)  my_msg(MYCOPY_MSG_ERROR, "ERR:  " FMT, ## ARGS)

#define DBGMSG 1

void h_DumpARPPacket(struct arphdr *arph)
{
	uint8_t *t;

	printf("ARP header: \n");
	printf("Hardware type: %d (len: %d), "
			"protocol type: %d (len: %d), opcode: %d\n", 
			//ntohs(arph->ar_hrd), arph->ar_hln, 
			NTOHS(arph->ar_hrd), arph->ar_hln, 
			//ntohs(arph->ar_pro), arph->ar_pln, ntohs(arph->ar_op));
			NTOHS(arph->ar_pro), arph->ar_pln, NTOHS(arph->ar_op));
	t = (uint8_t *)&arph->ar_sip;
	printf("Sender IP: %u.%u.%u.%u, "
			"haddr: %02X:%02X:%02X:%02X:%02X:%02X\n", 
			t[0], t[1], t[2], t[3], 
			arph->ar_sha[0], arph->ar_sha[1], arph->ar_sha[2], 
			arph->ar_sha[3], arph->ar_sha[4], arph->ar_sha[5]);
	t = (uint8_t *)&arph->ar_tip;
	printf("Target IP: %u.%u.%u.%u, "
			"haddr: %02X:%02X:%02X:%02X:%02X:%02X\n", 
			t[0], t[1], t[2], t[3], 
			arph->ar_tha[0], arph->ar_tha[1], arph->ar_tha[2], 
			arph->ar_tha[3], arph->ar_tha[4], arph->ar_tha[5]);
}






__device__ void DumpARPPacket(struct arphdr *arph)
{
	uint8_t *t;

	printf("ARP header: \n");
	printf("Hardware type: %d (len: %d), "
			"protocol type: %d (len: %d), opcode: %d\n", 
			//ntohs(arph->ar_hrd), arph->ar_hln, 
			NTOHS(arph->ar_hrd), arph->ar_hln, 
			//ntohs(arph->ar_pro), arph->ar_pln, ntohs(arph->ar_op));
			NTOHS(arph->ar_pro), arph->ar_pln, NTOHS(arph->ar_op));
	t = (uint8_t *)&arph->ar_sip;
	printf("Sender IP: %u.%u.%u.%u, "
			"haddr: %02X:%02X:%02X:%02X:%02X:%02X\n", 
			t[0], t[1], t[2], t[3], 
			arph->ar_sha[0], arph->ar_sha[1], arph->ar_sha[2], 
			arph->ar_sha[3], arph->ar_sha[4], arph->ar_sha[5]);
	t = (uint8_t *)&arph->ar_tip;
	printf("Target IP: %u.%u.%u.%u, "
			"haddr: %02X:%02X:%02X:%02X:%02X:%02X\n", 
			t[0], t[1], t[2], t[3], 
			arph->ar_tha[0], arph->ar_tha[1], arph->ar_tha[2], 
			arph->ar_tha[3], arph->ar_tha[4], arph->ar_tha[5]);
}

void h_DumpPacket(uint8_t *buf, int len)
{
  printf("<<<h_DumpPacket>>>\n");
	struct ethhdr *ethh;
	struct iphdr *iph;
	struct udphdr *udph;
	//struct tcphdr *tcph;
	uint8_t *t;

	ethh = (struct ethhdr *)buf;
	//if (ntohs(ethh->h_proto) != ETH_P_IP) {
	if (NTOHS(ethh->h_proto) != ETH_P_IP) {
		printf("%02X:%02X:%02X:%02X:%02X:%02X -> %02X:%02X:%02X:%02X:%02X:%02X ",
				ethh->h_source[0],
				ethh->h_source[1],
				ethh->h_source[2],
				ethh->h_source[3],
				ethh->h_source[4],
				ethh->h_source[5],
				ethh->h_dest[0],
				ethh->h_dest[1],
				ethh->h_dest[2],
				ethh->h_dest[3],
				ethh->h_dest[4],
				ethh->h_dest[5]);

		//printf("protocol %04hx  \n", ntohs(ethh->h_proto));
		printf("protocol %04hx  \n", NTOHS(ethh->h_proto));

    //if(ntohs(ethh->h_proto) == ETH_P_ARP)
    if(NTOHS(ethh->h_proto) == ETH_P_ARP)
      h_DumpARPPacket((struct arphdr *) (ethh + 1));
		goto done;
	}

	iph = (struct iphdr *)(ethh + 1);
	udph = (struct udphdr *)((uint32_t *)iph + iph->ihl);
	//tcph = (struct tcphdr *)((uint32_t *)iph + iph->ihl);

	t = (uint8_t *)&iph->saddr;
	printf("%u.%u.%u.%u", t[0], t[1], t[2], t[3]);
	if (iph->protocol == IPPROTO_TCP || iph->protocol == IPPROTO_UDP)
		//printf("(%d)", ntohs(udph->source));
		printf("(%d)", NTOHS(udph->source));

	printf(" -> ");

	t = (uint8_t *)&iph->daddr;
	printf("%u.%u.%u.%u", t[0], t[1], t[2], t[3]);
	if (iph->protocol == IPPROTO_TCP || iph->protocol == IPPROTO_UDP)
		//printf("(%d)", ntohs(udph->dest));
		printf("(%d)", NTOHS(udph->dest));

	//printf(" IP_ID=%d", ntohs(iph->id));
	printf(" IP_ID=%d", NTOHS(iph->id));
	printf(" TTL=%d ", iph->ttl);

	switch (iph->protocol) {
	case IPPROTO_TCP:
		printf("TCP ");
		break;
	case IPPROTO_UDP:
		printf("UDP ");
		break;
	default:
		printf("protocol %d ", iph->protocol);
		goto done;
	}
done:
	printf("len=%d\n", len);
}





__device__ void DumpPacket(uint8_t *buf, int len)
//void DumpPacket(uint8_t *buf, int len)
{
  printf("<<<DumpPacket>>>----------------------------------------------------------------\n");
	struct ethhdr *ethh;
	struct iphdr *iph;
	struct udphdr *udph;
	//struct tcphdr *tcph;
	uint8_t *t;

	ethh = (struct ethhdr *)buf;
	//if (ntohs(ethh->h_proto) != ETH_P_IP) {
	if (NTOHS(ethh->h_proto) != ETH_P_IP) {
		printf("%02X:%02X:%02X:%02X:%02X:%02X -> %02X:%02X:%02X:%02X:%02X:%02X ",
				ethh->h_source[0],
				ethh->h_source[1],
				ethh->h_source[2],
				ethh->h_source[3],
				ethh->h_source[4],
				ethh->h_source[5],
				ethh->h_dest[0],
				ethh->h_dest[1],
				ethh->h_dest[2],
				ethh->h_dest[3],
				ethh->h_dest[4],
				ethh->h_dest[5]);

		//printf("protocol %04hx  \n", ntohs(ethh->h_proto));
		printf("protocol %04hx  \n", NTOHS(ethh->h_proto));

    //if(ntohs(ethh->h_proto) == ETH_P_ARP)
    if(NTOHS(ethh->h_proto) == ETH_P_ARP)
      DumpARPPacket((struct arphdr *) (ethh + 1));
		goto done;
	}

	iph = (struct iphdr *)(ethh + 1);
	udph = (struct udphdr *)((uint32_t *)iph + iph->ihl);
	//tcph = (struct tcphdr *)((uint32_t *)iph + iph->ihl);

	t = (uint8_t *)&iph->saddr;
	printf("%u.%u.%u.%u", t[0], t[1], t[2], t[3]);
	if (iph->protocol == IPPROTO_TCP || iph->protocol == IPPROTO_UDP)
		//printf("(%d)", ntohs(udph->source));
		printf("(%d)", NTOHS(udph->source));

	printf(" -> ");

	t = (uint8_t *)&iph->daddr;
	printf("%u.%u.%u.%u", t[0], t[1], t[2], t[3]);
	if (iph->protocol == IPPROTO_TCP || iph->protocol == IPPROTO_UDP)
		//printf("(%d)", ntohs(udph->dest));
		printf("(%d)", NTOHS(udph->dest));

	//printf(" IP_ID=%d", ntohs(iph->id));
	printf(" IP_ID=%d", NTOHS(iph->id));
	printf(" TTL=%d ", iph->ttl);

	switch (iph->protocol) {
	case IPPROTO_TCP:
		printf("TCP ");
		break;
	case IPPROTO_UDP:
		printf("UDP ");
		break;
	default:
		printf("protocol %d ", iph->protocol);
		goto done;
	}
done:
	printf("len=%d\n", len);
  printf("<<<DumpPacket>>>-----------------------------------END--------------------------\n");

}


uint8_t * h_EthernetOutput(uint8_t *buf, uint16_t h_proto, unsigned char* src_haddr, unsigned char* dst_haddr, uint16_t iplen)
{
	struct ethhdr *ethh;
	int i;

	ethh = (struct ethhdr *)buf;

#if 1
	printf("dst_hwaddr: %02X:%02X:%02X:%02X:%02X:%02X\n",
				dst_haddr[0], dst_haddr[1], 
				dst_haddr[2], dst_haddr[3], 
				dst_haddr[4], dst_haddr[5]);
	printf("src_hwaddr: %02X:%02X:%02X:%02X:%02X:%02X\n",
				src_haddr[0], src_haddr[1], 
				src_haddr[2], src_haddr[3], 
				src_haddr[4], src_haddr[5]);
#endif

	for (i = 0; i < ETH_ALEN; i++) {
		ethh->h_source[i] = src_haddr[i];
		ethh->h_dest[i] = dst_haddr[i];
	}
	ethh->h_proto = HTONS(h_proto);

	return (uint8_t *)(ethh + 1);
}


__device__ uint8_t * EthernetOutput(uint8_t *buf, uint16_t h_proto, unsigned char* src_haddr, unsigned char* dst_haddr, uint16_t iplen)
//uint8_t * EthernetOutput(uint8_t *buf, uint16_t h_proto, unsigned char* src_haddr, unsigned char* dst_haddr, uint16_t iplen)
{
	struct ethhdr *ethh;
	int i;

	ethh = (struct ethhdr *)buf;

#if 1
	printf("dst_hwaddr: %02X:%02X:%02X:%02X:%02X:%02X\n",
				dst_haddr[0], dst_haddr[1], 
				dst_haddr[2], dst_haddr[3], 
				dst_haddr[4], dst_haddr[5]);
	printf("src_hwaddr: %02X:%02X:%02X:%02X:%02X:%02X\n",
				src_haddr[0], src_haddr[1], 
				src_haddr[2], src_haddr[3], 
				src_haddr[4], src_haddr[5]);
#endif

	for (i = 0; i < ETH_ALEN; i++) {
		ethh->h_source[i] = src_haddr[i];
		ethh->h_dest[i] = dst_haddr[i];
	}
	ethh->h_proto = HTONS(h_proto);

	return (uint8_t *)(ethh + 1);
}

static int h_ARPOutput(uint8_t * buf, int opcode, uint32_t src_ip, uint32_t dst_ip, unsigned char *dst_haddr)
{
	if (!dst_haddr)
		return -1;
  printf("ARPOUTPUT\n");
	/* Allocate a buffer */

#if 1
  uint8_t src_haddr[ETH_ALEN];
  // ckjung: 00:1b:21:bc:11:52
  src_haddr[0] = 0x00;
  src_haddr[1] = 0x1b;
  src_haddr[2] = 0x21;
  src_haddr[3] = 0xbc;
  src_haddr[4] = 0x11;
  src_haddr[5] = 0x52;

	struct arphdr *arph = (struct arphdr *)(uintptr_t)h_EthernetOutput(
    buf, ETH_P_ARP, src_haddr, dst_haddr, sizeof(struct arphdr));

	if (!arph) {
    printf("ERROR\n");
		return -1;
	}
#else 
	struct arphdr *arph = NULL;
#endif
	/* Fill arp header */
	arph->ar_hrd = HTONS(arp_hrd_ethernet);
	arph->ar_pro = HTONS(ETH_P_IP);
	//arph->ar_pro = htons(0x0800);
	arph->ar_hln = ETH_ALEN;
	arph->ar_pln = 4;
	arph->ar_op = HTONS(opcode);

	/* Fill arp body */
#if 0 // HONESTCHOI : TODO
	arph->ar_sip = CONFIG.eths[nif].ip_addr;
#endif 
	arph->ar_sip = src_ip;
	arph->ar_tip = dst_ip;

#if 0 // HONESTCHOI : TODO
	memcpy(arph->ar_sha, CONFIG.eths[nif].haddr, arph->ar_hln);
	if (target_haddr) {
		memcpy(arph->ar_tha, target_haddr, arph->ar_hln);
	} else {
		memcpy(arph->ar_tha, dst_haddr, arph->ar_hln);
	}
#endif
	memcpy(arph->ar_sha, src_haddr, arph->ar_hln);
  memcpy(arph->ar_tha, dst_haddr, arph->ar_hln);
	memset(arph->pad, 0, ARP_PAD_LEN);

#if DBGMSG
//	DumpARPPacket(arph);
#endif

	return 0;
}

__device__ static int ARPOutput(uint8_t * buf, int opcode, uint32_t src_ip, uint32_t dst_ip, unsigned char *dst_haddr)
//static int ARPOutput(uint8_t * buf, int opcode, uint32_t src_ip, uint32_t dst_ip, unsigned char *dst_haddr)
{
	if (!dst_haddr)
		return -1;
  printf("ARPOUTPUT\n");
	/* Allocate a buffer */

#if 1
  uint8_t src_haddr[ETH_ALEN];
  // ckjung: 00:1b:21:bc:11:52
  src_haddr[0] = 0x00;
  src_haddr[1] = 0x1b;
  src_haddr[2] = 0x21;
  src_haddr[3] = 0xbc;
  src_haddr[4] = 0x11;
  src_haddr[5] = 0x52;

	struct arphdr *arph = (struct arphdr *)(uintptr_t)EthernetOutput(
    buf, ETH_P_ARP, src_haddr, dst_haddr, sizeof(struct arphdr));

	if (!arph) {
    printf("ERROR\n");
		return -1;
	}
#else 
	struct arphdr *arph = NULL;
#endif
	/* Fill arp header */
	arph->ar_hrd = HTONS(arp_hrd_ethernet);
	arph->ar_pro = HTONS(ETH_P_IP);
	//arph->ar_pro = htons(0x0800);
	arph->ar_hln = ETH_ALEN;
	arph->ar_pln = 4;
	arph->ar_op = HTONS(opcode);

	/* Fill arp body */
#if 0 // HONESTCHOI : TODO
	arph->ar_sip = CONFIG.eths[nif].ip_addr;
#endif 
	arph->ar_sip = src_ip;
	arph->ar_tip = dst_ip;

#if 0 // HONESTCHOI : TODO
	memcpy(arph->ar_sha, CONFIG.eths[nif].haddr, arph->ar_hln);
	if (target_haddr) {
		memcpy(arph->ar_tha, target_haddr, arph->ar_hln);
	} else {
		memcpy(arph->ar_tha, dst_haddr, arph->ar_hln);
	}
#endif
	memcpy(arph->ar_sha, src_haddr, arph->ar_hln);
  memcpy(arph->ar_tha, dst_haddr, arph->ar_hln);
	memset(arph->pad, 0, ARP_PAD_LEN);

#if DBGMSG
	DumpARPPacket(arph);
#endif

	return 0;
}
int my_pin_buffer(my_t g, unsigned long addr, size_t size, uint64_t p2p_token, uint32_t va_space, my_mh_t *handle)
{
    int ret = 0;
    int retcode;

    struct MYDRV_IOC_PIN_BUFFER_PARAMS params;
    params.addr = addr;
    params.size = size;
    params.p2p_token = p2p_token;
    params.va_space = va_space;
    params.handle = 0;
    //params.buf_name = bname;

    retcode = ioctl(g->fd, MYDRV_IOC_PIN_BUFFER, &params);
    if (0 != retcode) {
        ret = errno;
        my_err("ioctl error (errno=%d)\n", ret);
    }
    *handle = params.handle;

    return ret;
}

my_t my_open()
{
    my_t m = NULL;
    //const char *myinode = "/dev/mydrv";
    const char *myinode = "/dev/ixgbe";

    m = (my_t) calloc(1, sizeof(*m));
    if (!m) {
        //my_err("error while allocating memory\n");
        return NULL;
    }

    int fd = open(myinode, O_RDWR);
    if (-1 == fd ) {
        int ret = errno;
        //my_err("error opening driver (errno=%d/%s)\n", ret, strerror(ret));
        free(m);
        return NULL;
    }

    m->fd = fd;

    return m;
}
#if 0
static inline uint32_t myrand(uint64_t *seed) 
{
	*seed = *seed * 1103515245 + 12345;
	return (uint32_t)(*seed >> 32);
}
#endif


// CKJUNG ~

__global__ void make_pkt(int* g_mem, int size)
{
	printf("[%s][%d]----------------------------------------START---make_pkt------------DEVICE.\n", __FUNCTION__, __LINE__);

	// ARP call
	uint8_t* pktBuf;
	pktBuf = (uint8_t *)malloc(60);
	
	// For now, static ip address 
	uint8_t src_tmp[] = {0x0a, 0x00, 0x00, 0x02};
	uint8_t dst_tmp[] = {0x0a, 0x00, 0x00, 0x01};
	uint32_t src_ip;                             
	memcpy(&src_ip, src_tmp, 4);                 
	uint32_t dst_ip;                             
	memcpy(&dst_ip, dst_tmp, 4);  
	unsigned char dst_haddr[ETH_ALEN];
	memset(dst_haddr, 0xFF, ETH_ALEN);

// Just for check the value of variables
#if 0	
	uint8_t *t,*s;
	t = (uint8_t *)&src_ip;	
	s = (uint8_t *)&dst_haddr;	
	printf("[%s][%d]____CKJUNG___src_ip: %u.%u.%u.%u.\n", __FUNCTION__, __LINE__, t[0],t[1],t[2],t[3]);
	printf("[%s][%d]____CKJUNG___dst_haddr: %u.%u.%u.%u.\n", __FUNCTION__, __LINE__, s[0],s[1],s[2],s[3]);
#endif
	ARPOutput(pktBuf, arp_op_request, src_ip, dst_ip, dst_haddr);
	
	DumpPacket(pktBuf, 60);
	
#if 1
	for(int i=0; i< size/sizeof(int); i++) {
		g_mem[i] = 0;
	}

	printf("[%s][%d]____CKJUNG__ HERE\n", __FUNCTION__, __LINE__);

	for(int i=0; i< 512; i++) { // making 100 arpreq packets
		for(int j=0; j < 15; j++) { 
			//g_mem[4096*i+j] = pktBuf[j];
			memcpy(g_mem+4096*i+j, pktBuf + 4*j, 4);
		}
	}
	// For check
#if 0
	int count = 0;
	for(int j = 0; j < 4096*512; j++) {
		if(*((uint16_t*)&g_mem[j]) == 0x0608 || *((uint16_t*)&g_mem[j]) == 0x0806 ){
			count++;
			printf("[%s][%d] %d We found ethernet type 0x%02x%02x on %dth memory.\n", 
					__FUNCTION__ , __LINE__, count,g_mem[j], g_mem[j+1], j);
		}
	}
#endif
#else
#endif
	printf("[%s][%d]------------------------------------------END---make_pkt------------DEVICE.\n", __FUNCTION__, __LINE__);

}

// ~CKJUNG


int curr_num;
__device__ void d_check_data(int size, int* d_pkt_buffer, volatile int* flag)
{
  uint32_t offset_for_rx = 512 * 4096;
  volatile unsigned char * d_mem = (unsigned char*)d_pkt_buffer;
  d_mem += offset_for_rx;

  int pkt_cnt = 0;
  //const int DUMP_SIZE = 30;
  //__threadfence_system();
  //printf("[%s][%d] before waiting %d\n", __FUNCTION__ , __LINE__, *flag);
  //WAIT_ON_MEM_NE(d_mem[2093068], 0);
  WAIT_ON_MEM(*flag, 1);
  //printf("[%s][%d] after waiting %d\n", __FUNCTION__ , __LINE__, *flag);

#if 1
  while(pkt_cnt < 30000) {
    //printf("[%s][%d] BEFORE WAIT_ON_MEM %d\n", __FUNCTION__ , __LINE__, threadIdx.x);
    int curr_index = 12 + 0x1000*threadIdx.x; // + 0x1000*(THREADS_PER_TB * pkt_cnt);
    //WAIT_ON_MEM_NE(d_mem[curr_index], 0);
    pkt_cnt++;
    if(*((uint16_t*)&d_mem[curr_index]) == 0x0608 || *((uint16_t*)&d_mem[curr_index]) == 0x0806 ) {
      printf("[%s][%d] Ethernet type 0x%02x%02x on %dth memory. (threadIdx.x:%d %dpkts)\n", __FUNCTION__ , __LINE__, d_mem[curr_index], d_mem[curr_index+1], curr_index, threadIdx.x, pkt_cnt);
      *((uint16_t*)&d_mem[curr_index]) = 0;     
    }
  }
#endif

}

__global__ void pkt_consumer(int * pkt_queue, int pkt_queue_size, volatile int * server_done) 
{
  unsigned char * d_mem = (unsigned char*)pkt_queue;
  uint32_t offset_for_rx = 512 * 4096 ;
  d_mem += offset_for_rx;

  int pkt_cnt = 0;

  //__threadfence_system();
  int curr_index = 12 + 0x1000*threadIdx.x; // + 0x1000*(THREADS_PER_TB * pkt_cnt);
  while(!(*server_done)) { 
    if(*((uint16_t*)&d_mem[curr_index]) == 0x0608 || *((uint16_t*)&d_mem[curr_index]) == 0x0806) {
      pkt_cnt++;
      printf("[%s][%d] Ethernet type 0x%02x%02x on %dth memory. (threadIdx.x:%d %dpkts)\n", __FUNCTION__ , __LINE__, d_mem[curr_index], d_mem[curr_index+1],curr_index, threadIdx.x, pkt_cnt);
      // for now, found and initialize.
      *((uint16_t*)&d_mem[curr_index]) = 0;
      curr_index += 0x1000*THREADS_PER_TB;
    }
  }
}

__global__ void pkt_pumper(int* d_pkt_buffer, int size, int * pkt_queue, int pkt_queue_size, volatile int * server_done)
{
  // first block for check
  // TODO: use server_complete as in GPUnet
  int num_to_loop = 3000; 
  unsigned char * d_mem = (unsigned char*)d_pkt_buffer;
  unsigned char * d_mem2 = (unsigned char*)pkt_queue;
  uint32_t offset_for_rx = 512 * 4096;
  d_mem += offset_for_rx;

  int pkt_cnt = 0;

  //__threadfence_system();
  int my_index = 12 + 0x1000*blockIdx.x;
  //int my_index = 12 + 0x1000*threadIdx.x;
  //printf("server_done: %d\n", *server_done);
  printf("server_done: %d\n", *server_done);
  
  //while(!(*server_done)) { 
  while(num_to_loop--) { 
    printf("[%s][%d] blockIdx.x:%d %dpkts, index:%d\n", __FUNCTION__ , __LINE__, blockIdx.x, pkt_cnt, my_index);
    if(*((uint16_t*)&d_mem[my_index]) != 0) { 
      printf("[%s][%d] %dth memory != 0. (threadIdx.x:%d %dpkts)\n", __FUNCTION__ , __LINE__, my_index, threadIdx.x, pkt_cnt);
      memcpy((void*)(d_mem2 + my_index + 0x1000 * pkt_cnt), d_mem + my_index, 0x1000);
      pkt_cnt++;
      *((uint16_t*)&d_mem[my_index]) = 0;
    }
  }
  printf("server_done: %d\n", *server_done);
}

__device__ volatile int finished;

#if 0
__device__ void wait_for_something(volatile int * something_finished)
{
  BEGIN_SINGLE_THREAD_PART {
    while(!*something_finished) {
    }
  }
}
#endif

__device__ volatile int server_done;
#define NUM_TB 2 

__global__ void rx_handler(volatile int * tb_alloc_tbl)
{
  //printf("Entering rx_handler. (Block ID:%d)\n", blockIdx.x);
  //printf("server_done1:%d\n", server_done);
  int num = 0;
  if(blockIdx.x == 0) {
    BEGIN_SINGLE_THREAD_PART {
      for(int i=1; i<=NUM_TB * 50 * 10000; i++)
        if(i % (50*10000) == 0 ) {
          int block_num = i / (50*10000) ;
          printf("[%d] sets %d to %d(i:%d)\n", blockIdx.x, block_num, 20*block_num, i);
          tb_alloc_tbl[1] = 20*block_num;
        }
      //server_done = 1;
    } END_SINGLE_THREAD_PART;
  } else {
    do {
      while(!tb_alloc_tbl[blockIdx.x]) { } 
      printf("[%2d,%2d] %dth %d\n", blockIdx.x, threadIdx.x, num++, tb_alloc_tbl[blockIdx.x]);
      //tb_alloc_tbl[blockIdx.x] = 0;
      //if(threadIdx.x < tb_alloc_tbl[blockIdx.x]) {
      //  printf("[%2d,%2d]\n", blockIdx.x, threadIdx.x);
      //} else {
      //}
      if(tb_alloc_tbl[blockIdx.x] == 40)
        break;
      tb_alloc_tbl[blockIdx.x] = 0;
    } while(!tb_alloc_tbl[blockIdx.x]);
  }
  //printf("[%d] server_done2:%d\n", blockIdx.x, server_done);
}

union ixgbe_adv_tx_desc {
	struct {
		__le64 buffer_addr; /* Address of descriptor's data buf */
		__le32 cmd_type_len;
		__le32 olinfo_status;
	} read;
	struct {
		__le64 rsvd; /* Reserved */
		__le32 nxtseq_seed;
		__le32 status;
	} wb;
};


#define COMPILER_BARRIER() asm volatile("" ::: "memory")
#define cpu_to_le32(x) ((__le32)(__swab32)(x))


__global__ void doorbell_test(void * io_addr, void * desc, uint32_t curr, int* g_mem, int size, int* flag)
{
  *flag = 0;
  if (blockIdx.x == 0) {
    BEGIN_SINGLE_THREAD_PART {
      printf("[%s][%d] in doorbell_test First Block.\n", __FUNCTION__, __LINE__);
      if(desc == 0) {
        printf("[%s][%d]desc==NULL.\n", __FUNCTION__, __LINE__);
        return;
      }
      // ARP call
      uint8_t* pktBuf;
      pktBuf = (uint8_t *)malloc(60);

      // For now, static ip address 
      uint8_t src_tmp[] = {0x0a, 0x00, 0x00, 0x02};
      uint8_t dst_tmp[] = {0x0a, 0x00, 0x00, 0x01};
      uint32_t src_ip;                             
      memcpy(&src_ip, src_tmp, 4);                 
      uint32_t dst_ip;                             
      memcpy(&dst_ip, dst_tmp, 4);  
      unsigned char dst_haddr[ETH_ALEN];
      memset(dst_haddr, 0xFF, ETH_ALEN);

      //ARPOutput(pktBuf, arp_op_request, src_ip, dst_ip, dst_haddr);

      //DumpPacket(pktBuf, 60);


      for(int i=0; i< size/sizeof(int); i++) {
        g_mem[i] = 0;
      }

      for(int i=0; i< 512; i++) {
        memcpy(g_mem+4096*i/4, pktBuf, 60);
      }

      unsigned char *db[12];
      for(int i=0; i<12; i++)
        db[i] = ((unsigned char *)io_addr) + IXGBE_TDT(i);

      COMPILER_BARRIER();
      volatile union ixgbe_adv_tx_desc* tx_desc;
      tx_desc = (union ixgbe_adv_tx_desc*) desc;

      int num_packets = NUM_PACKETS;
      int index = curr;
      for(int i=0; i<curr; i++) {
        tx_desc++;
      }
      for(int i=0; i<num_packets; i++) {
        tx_desc->read.cmd_type_len |= 60;
        tx_desc->read.olinfo_status = 0xf0000;
        tx_desc++;
        index++;
        if(index == 512)
          tx_desc = (union ixgbe_adv_tx_desc*) desc;
      }
      unsigned long tail_val = (unsigned long)((curr + num_packets) % 512);
      *(volatile unsigned long*)db[0] = tail_val;

      COMPILER_BARRIER();

      //__threadfence_system();
      printf("[%s][%d] after flag=1.\n", __FUNCTION__, __LINE__);
      *flag = 1;
    } END_SINGLE_THREAD_PART;
    
  } else {
    // Second Block
    printf("[%s][%d] in doorbell_test Second Block.\n", __FUNCTION__, __LINE__);
    d_check_data(size, g_mem, flag);
  }
}

// YHOON~ for test
void yhoon_xmit_arp(int *g_mem, int size)
{
  printf("[%s][%d]START---yhoon_xmit_arp------------HOST.\n", __FUNCTION__, __LINE__);
  printf("[%s][%d] sizeof tx_desc:%lu \n", __FUNCTION__, __LINE__, sizeof(union ixgbe_adv_tx_desc));
  const char *myinode = "/dev/ixgbe";
  int fd = open(myinode, O_RDWR);
  //uint64_t ptr = 1234;
  //ioctl(fd, 0, &ptr);
  ioctl(fd, 1);

  //printf("cpu_to_le32 test:%x\n", htonl(60));
  void* dBAR;
  const size_t IXGBE_BAR0_SIZE = 4096*8; // A rough calculation
  void* ixgbe_bar0_host_addr = mmap(0, IXGBE_BAR0_SIZE*5 , PROT_READ | PROT_WRITE, MAP_SHARED, fd, 0);
  ASSERTRT(hipHostRegister(ixgbe_bar0_host_addr, IXGBE_BAR0_SIZE, hipHostRegisterIoMemory));
  ASSERTRT(hipHostGetDevicePointer((void**)&dBAR, (void*)ixgbe_bar0_host_addr, 0));

  unsigned char *db[12];
  for(int i=0; i<1; i++) {
    db[i] = ((unsigned char *)ixgbe_bar0_host_addr) + IXGBE_TDT(i);
#if 0
    printf("db[%d]: %d\n", i, *(volatile unsigned int *)db[i]);
#endif
  }

  ixgbe_adv_tx_desc* desc = (ixgbe_adv_tx_desc*)((unsigned char*)ixgbe_bar0_host_addr + IXGBE_BAR0_SIZE);
  ASSERTRT(hipHostRegister(desc, IXGBE_BAR0_SIZE * 4, hipHostRegisterIoMemory));
  //hipHostRegister(desc, sizeof(ixgbe_adv_tx_desc), hipHostRegisterIoMemory);
  void* tx_desc;
  hipMalloc(&tx_desc, IXGBE_BAR0_SIZE * 4);
  if(hipSuccess != hipHostGetDevicePointer((void**)&tx_desc, (void*)desc, 0)) {
    cudaCheckErrors("hipHostGetDevicePointer fails");
  }

  uint32_t curr_tx_index_q_zero = *(volatile unsigned int *)db[0];
  printf("curr_tx_index_q_zero: %u\n", curr_tx_index_q_zero);
  int *flag;
  hipMalloc(&flag, sizeof(int));

	hipStream_t hip_stream;
  ASSERT_CUDA(hipStreamCreate(&hip_stream));

  doorbell_test<<< 1, 1, 0, hip_stream >>>(dBAR, tx_desc, curr_tx_index_q_zero, g_mem, size, flag);

  //if(hipSuccess != hipDeviceSynchronize())
	//  cudaCheckErrors("doorbell_sync_error!");

  curr_num = curr_tx_index_q_zero;
  hipHostUnregister(desc);
  hipHostUnregister(ixgbe_bar0_host_addr);
  munmap(ixgbe_bar0_host_addr, IXGBE_BAR0_SIZE*5);

  // ~YHOON
  printf("[%s][%d]END-----yhoon_xmit_arp------------HOST.\n", __FUNCTION__, __LINE__);
}


int main(int argc, char *argv[])
{
  printf("[%s][%d] main\n", __FUNCTION__, __LINE__);
  int dev_id = 0;
  size_t _pkt_buffer_size = 2*512*4096; // 4MB, for rx,tx ring
  size_t pkt_queue_size = 50*1024*1024; // 50MB, for packet queue

  // CKJUNG, meaning of this?
  size_t pkt_buffer_size = (_pkt_buffer_size + GPU_PAGE_SIZE - 1) & GPU_PAGE_MASK;
  printf("[%s][%d]____CKJUNG__pkt_buffer_size: %lu\n", __FUNCTION__, __LINE__, pkt_buffer_size);

  int n_devices = 0;

  ASSERTRT(hipGetDeviceCount(&n_devices));

  hipDeviceProp_t prop;
  for (int n=0; n<n_devices; ++n) { hipGetDeviceProperties(&prop,n); OUT << "GPU id:" << n << " name:" << prop.name 
      << " PCI domain: " << prop.pciDomainID 
      << " bus: " << prop.pciBusID 
      << " device: " << prop.pciDeviceID << endl;
  }
  OUT << "selecting device " << dev_id << endl;
  OUT << "_pkt_buffer_size: " << _pkt_buffer_size << "  pkt_buffer_size: " << pkt_buffer_size << endl;

  ASSERTRT(hipSetDevice(dev_id));
  //ASSERTRT(hipSetDeviceFlags(hipDeviceMapHost));

  OUT << "Before dummy malloc" << endl;
  // Test
  void* dummy;
  ASSERTRT(hipMalloc(&dummy, 0));

  OUT << "Before d_pkt_buffer malloc" << endl;
  int* d_pkt_buffer;
  ASSERTRT(hipMalloc((void**)&d_pkt_buffer, pkt_buffer_size));
  
  unsigned int flag = 1;
  ASSERTDRV(hipPointerSetAttribute(&flag, HIP_POINTER_ATTRIBUTE_SYNC_MEMOPS, (hipDeviceptr_t) d_pkt_buffer));


  my_t g = my_open();

  ASSERT_NEQ(g, (void*)0);

  my_mh_t mh;
  if (my_pin_buffer(g, (hipDeviceptr_t)d_pkt_buffer, pkt_buffer_size, 0, 0, &mh)  != 0)
    OUT << "NOT_EQ" << endl;

  OUT << "Before xmit" << endl;
  // call ixgbe_xmit_yhoon in ixgbe_main.c
  yhoon_xmit_arp(d_pkt_buffer, pkt_buffer_size);

  if(hipSuccess != hipDeviceSynchronize())
	  cudaCheckErrors("make_pkt_error"); 

  OUT << "END" << endl;
  hipFree(d_pkt_buffer);
  return 0;
}
