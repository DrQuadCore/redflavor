#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <unistd.h>

#define MUTEX_LOCK(lock) while (atomicExch((int*)(&(lock)),1));
#define MUTEX_UNLOCK(lock) { atomicExch((int*)(&(lock)),0);}

struct toss
{
  volatile int flag;
  int _lock;
  __device__ void toggle();
  __device__ void check_and_return();
};

__device__ void mysleep(int64_t num_cycles)
{
  int64_t cycles = 0;
  int64_t start = clock64();
  while(cycles < num_cycles) {
    cycles = clock64() - start;
  }
}

__global__ void mywrapper(int cmd, struct toss* ptr, int* d_a) 
{
  switch(cmd) {
    case 0:
      ptr->toggle();
      break;
    case 1:
      ptr->check_and_return();
      break;
    case 2:
      ptr->flag = 0;
      ptr->_lock = 0;
      break;
  }
}

__device__ void toss::toggle() {
  printf("begin of toggle:%d\n", flag);
  if(flag == 0)
    flag = 1;
  else
    flag = 0;
  printf("end of toggle:%d\n", flag);
}

__device__ void toss::check_and_return()
{
  printf("begin of check:%d\n", flag);
  while(flag) {
  }
  printf("end of check:%d\n", flag);
}


int main() {
	int a, *d_a;
	int size = sizeof(int);

  hipStream_t stream1;
  hipStreamCreateWithFlags(&stream1,hipStreamNonBlocking);

	hipMalloc((void**)&d_a, size);
	
	a = 1;

  hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);

  struct toss* mytoss; 
  hipMalloc((void**)&mytoss, sizeof(struct toss));

  printf("%d\n", a);

  // init
  mywrapper<<< 1,1 >>> (2, mytoss, NULL);

  // toggle
  mywrapper<<< 1,1 >>> (0, mytoss, d_a);

  // check_and_run
  mywrapper<<< 1,1,0,stream1 >>> (1, mytoss, d_a);

  // toggle
  mywrapper<<< 1,1 >>> (0, mytoss, d_a);

  hipDeviceSynchronize();

  hipMemcpy(&a, d_a, size, hipMemcpyDeviceToHost);
	hipFree(d_a);
  hipStreamDestroy(stream1);
  printf("%d\n", a);
  

	return 0;
}
