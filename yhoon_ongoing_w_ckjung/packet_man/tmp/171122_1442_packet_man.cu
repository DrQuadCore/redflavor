#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <memory.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <stdarg.h>
#include <string.h>
#include <sys/types.h>
#include <sys/socket.h>
#include <sys/time.h>
#include <sys/stat.h>
#include <sys/mman.h>
#include <fcntl.h>
#include <errno.h>
#include <netdb.h>
#include <malloc.h>
#include <getopt.h>
#include <arpa/inet.h>
#include <sys/ioctl.h>
#include <time.h>
#include <asm/types.h>

#include <linux/if_ether.h>
#include <linux/ip.h>
#include <linux/udp.h>
#include "arp.h" 

#include <linux/netdevice.h>   /* struct device, and other headers */
#include <linux/tcp.h>         /* struct tcphdr */
#include <linux/in6.h>
#define ETH_ALEN  6 // YHOON
#define ARP_PAD_LEN 18 // YHOON

#define HTONS(n) (((((unsigned short)(n) & 0xFF)) << 8) | (((unsigned short)(n) & 0xFF00) >> 8))
#define NTOHS(n) (((((unsigned short)(n) & 0xFF)) << 8) | (((unsigned short)(n) & 0xFF00) >> 8))

#define HTONL(n) (((((unsigned long)(n) & 0xFF)) << 24) | \
	((((unsigned long)(n) & 0xFF00)) << 8) | \
	((((unsigned long)(n) & 0xFF0000)) >> 8) | \
		  ((((unsigned long)(n) & 0xFF000000)) >> 24))

#define NTOHL(n) (((((unsigned long)(n) & 0xFF)) << 24) | \
	((((unsigned long)(n) & 0xFF00)) << 8) | \
	((((unsigned long)(n) & 0xFF0000)) >> 8) | \
		  ((((unsigned long)(n) & 0xFF000000)) >> 24))


#define cudaCheckErrors(msg) do { hipError_t __err = hipGetLastError(); if (__err != hipSuccess) { \
			fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
					msg, hipGetErrorString(__err), \
					__FILE__, __LINE__); \
				fprintf(stderr, "*** FAILED - ABORTING\n"); \
				exit(1); \
		} \
	} while (0)

#include <asm/types.h>
using namespace std;

#include "packet_man.h"
#include "mydrv/mydrv.h"
#include "common.hpp"


#define OUT cout


#define HTONS(n) (((((unsigned short)(n) & 0xFF)) << 8) | (((unsigned short)(n) & 0xFF00) >> 8))
#define NTOHS(n) (((((unsigned short)(n) & 0xFF)) << 8) | (((unsigned short)(n) & 0xFF00) >> 8))
enum mycopy_msg_level {
    MYCOPY_MSG_DEBUG = 1,
    MYCOPY_MSG_INFO,
    MYCOPY_MSG_WARN,
    MYCOPY_MSG_ERROR
};

#define IXGBE_TDT(_i)   (0x06018 + ((_i) * 0x40))

struct my {
  int fd;
};

/*----------------------------------------------------------------------------*/
enum arp_hrd_format
{
	arp_hrd_ethernet = 1
};
/*----------------------------------------------------------------------------*/
enum arp_opcode
{
	arp_op_request = 1, 
	arp_op_reply = 2, 
};

// YHOON
struct arphdr
{
  uint16_t ar_hrd;      /* hardware address format */
  uint16_t ar_pro;      /* protocol address format */
  uint8_t ar_hln;       /* hardware address length */
  uint8_t ar_pln;       /* protocol address length */
  uint16_t ar_op;       /* arp opcode */

  uint8_t ar_sha[ETH_ALEN]; /* sender hardware address */
  uint32_t ar_sip;      /* sender ip address */
  uint8_t ar_tha[ETH_ALEN]; /* targe hardware address */
  uint32_t ar_tip;      /* target ip address */

  uint8_t pad[ARP_PAD_LEN];
} __attribute__ ((packed));

static int my_msg_level = MYCOPY_MSG_ERROR;
static int my_enable_logging = 1;

static void my_msg(enum mycopy_msg_level lvl, const char* fmt, ...)
{
    if (-1 == my_enable_logging) {
        const char *env = getenv("MYCOPY_ENABLE_LOGGING");
        if (env)
            my_enable_logging = 1;
        else
            my_enable_logging = 0;

        env = getenv("MYCOPY_LOG_LEVEL");
        if (env)
            my_msg_level = atoi(env);
    }
    if (my_enable_logging) {
        if (lvl >= my_msg_level) {
            va_list ap;
            va_start(ap, fmt);
            vfprintf(stderr, fmt, ap);
        }
    }
}

#define my_dbg(FMT, ARGS...)  my_msg(MYCOPY_MSG_DEBUG, "DBG:  " FMT, ## ARGS)
#define my_dbgc(C, FMT, ARGS...)  do { static int my_dbg_cnt=(C); if (my_dbg_cnt) { my_dbg(FMT, ## ARGS); --my_dbg_cnt; }} while (0)
#define my_info(FMT, ARGS...) my_msg(MYCOPY_MSG_INFO,  "INFO: " FMT, ## ARGS)
#define my_warn(FMT, ARGS...) my_msg(MYCOPY_MSG_WARN,  "WARN: " FMT, ## ARGS)
#define my_err(FMT, ARGS...)  my_msg(MYCOPY_MSG_ERROR, "ERR:  " FMT, ## ARGS)

#define DBGMSG 1

void h_DumpARPPacket(struct arphdr *arph)
{
	uint8_t *t;

	printf("ARP header: \n");
	printf("Hardware type: %d (len: %d), "
			"protocol type: %d (len: %d), opcode: %d\n", 
			//ntohs(arph->ar_hrd), arph->ar_hln, 
			NTOHS(arph->ar_hrd), arph->ar_hln, 
			//ntohs(arph->ar_pro), arph->ar_pln, ntohs(arph->ar_op));
			NTOHS(arph->ar_pro), arph->ar_pln, NTOHS(arph->ar_op));
	t = (uint8_t *)&arph->ar_sip;
	printf("Sender IP: %u.%u.%u.%u, "
			"haddr: %02X:%02X:%02X:%02X:%02X:%02X\n", 
			t[0], t[1], t[2], t[3], 
			arph->ar_sha[0], arph->ar_sha[1], arph->ar_sha[2], 
			arph->ar_sha[3], arph->ar_sha[4], arph->ar_sha[5]);
	t = (uint8_t *)&arph->ar_tip;
	printf("Target IP: %u.%u.%u.%u, "
			"haddr: %02X:%02X:%02X:%02X:%02X:%02X\n", 
			t[0], t[1], t[2], t[3], 
			arph->ar_tha[0], arph->ar_tha[1], arph->ar_tha[2], 
			arph->ar_tha[3], arph->ar_tha[4], arph->ar_tha[5]);
}






__device__ void DumpARPPacket(struct arphdr *arph)
//void DumpARPPacket(struct arphdr *arph)
{
	uint8_t *t;

	printf("ARP header: \n");
	printf("Hardware type: %d (len: %d), "
			"protocol type: %d (len: %d), opcode: %d\n", 
			//ntohs(arph->ar_hrd), arph->ar_hln, 
			NTOHS(arph->ar_hrd), arph->ar_hln, 
			//ntohs(arph->ar_pro), arph->ar_pln, ntohs(arph->ar_op));
			NTOHS(arph->ar_pro), arph->ar_pln, NTOHS(arph->ar_op));
	t = (uint8_t *)&arph->ar_sip;
	printf("Sender IP: %u.%u.%u.%u, "
			"haddr: %02X:%02X:%02X:%02X:%02X:%02X\n", 
			t[0], t[1], t[2], t[3], 
			arph->ar_sha[0], arph->ar_sha[1], arph->ar_sha[2], 
			arph->ar_sha[3], arph->ar_sha[4], arph->ar_sha[5]);
	t = (uint8_t *)&arph->ar_tip;
	printf("Target IP: %u.%u.%u.%u, "
			"haddr: %02X:%02X:%02X:%02X:%02X:%02X\n", 
			t[0], t[1], t[2], t[3], 
			arph->ar_tha[0], arph->ar_tha[1], arph->ar_tha[2], 
			arph->ar_tha[3], arph->ar_tha[4], arph->ar_tha[5]);
}

void h_DumpPacket(uint8_t *buf, int len)
{
  printf("<<<h_DumpPacket>>>\n");
	struct ethhdr *ethh;
	struct iphdr *iph;
	struct udphdr *udph;
	//struct tcphdr *tcph;
	uint8_t *t;

	ethh = (struct ethhdr *)buf;
	//if (ntohs(ethh->h_proto) != ETH_P_IP) {
	if (NTOHS(ethh->h_proto) != ETH_P_IP) {
		printf("%02X:%02X:%02X:%02X:%02X:%02X -> %02X:%02X:%02X:%02X:%02X:%02X ",
				ethh->h_source[0],
				ethh->h_source[1],
				ethh->h_source[2],
				ethh->h_source[3],
				ethh->h_source[4],
				ethh->h_source[5],
				ethh->h_dest[0],
				ethh->h_dest[1],
				ethh->h_dest[2],
				ethh->h_dest[3],
				ethh->h_dest[4],
				ethh->h_dest[5]);

		//printf("protocol %04hx  \n", ntohs(ethh->h_proto));
		printf("protocol %04hx  \n", NTOHS(ethh->h_proto));

    //if(ntohs(ethh->h_proto) == ETH_P_ARP)
    if(NTOHS(ethh->h_proto) == ETH_P_ARP)
      h_DumpARPPacket((struct arphdr *) (ethh + 1));
		goto done;
	}

	iph = (struct iphdr *)(ethh + 1);
	udph = (struct udphdr *)((uint32_t *)iph + iph->ihl);
	//tcph = (struct tcphdr *)((uint32_t *)iph + iph->ihl);

	t = (uint8_t *)&iph->saddr;
	printf("%u.%u.%u.%u", t[0], t[1], t[2], t[3]);
	if (iph->protocol == IPPROTO_TCP || iph->protocol == IPPROTO_UDP)
		//printf("(%d)", ntohs(udph->source));
		printf("(%d)", NTOHS(udph->source));

	printf(" -> ");

	t = (uint8_t *)&iph->daddr;
	printf("%u.%u.%u.%u", t[0], t[1], t[2], t[3]);
	if (iph->protocol == IPPROTO_TCP || iph->protocol == IPPROTO_UDP)
		//printf("(%d)", ntohs(udph->dest));
		printf("(%d)", NTOHS(udph->dest));

	//printf(" IP_ID=%d", ntohs(iph->id));
	printf(" IP_ID=%d", NTOHS(iph->id));
	printf(" TTL=%d ", iph->ttl);

	switch (iph->protocol) {
	case IPPROTO_TCP:
		printf("TCP ");
		break;
	case IPPROTO_UDP:
		printf("UDP ");
		break;
	default:
		printf("protocol %d ", iph->protocol);
		goto done;
	}
done:
	printf("len=%d\n", len);
}





__device__ void DumpPacket(uint8_t *buf, int len)
//void DumpPacket(uint8_t *buf, int len)
{
  printf("<<<DumpPacket>>>----------------------------------------------------------------\n");
	struct ethhdr *ethh;
	struct iphdr *iph;
	struct udphdr *udph;
	//struct tcphdr *tcph;
	uint8_t *t;

	ethh = (struct ethhdr *)buf;
	//if (ntohs(ethh->h_proto) != ETH_P_IP) {
	if (NTOHS(ethh->h_proto) != ETH_P_IP) {
		printf("%02X:%02X:%02X:%02X:%02X:%02X -> %02X:%02X:%02X:%02X:%02X:%02X ",
				ethh->h_source[0],
				ethh->h_source[1],
				ethh->h_source[2],
				ethh->h_source[3],
				ethh->h_source[4],
				ethh->h_source[5],
				ethh->h_dest[0],
				ethh->h_dest[1],
				ethh->h_dest[2],
				ethh->h_dest[3],
				ethh->h_dest[4],
				ethh->h_dest[5]);

		//printf("protocol %04hx  \n", ntohs(ethh->h_proto));
		printf("protocol %04hx  \n", NTOHS(ethh->h_proto));

    //if(ntohs(ethh->h_proto) == ETH_P_ARP)
    if(NTOHS(ethh->h_proto) == ETH_P_ARP)
      DumpARPPacket((struct arphdr *) (ethh + 1));
		goto done;
	}

	iph = (struct iphdr *)(ethh + 1);
	udph = (struct udphdr *)((uint32_t *)iph + iph->ihl);
	//tcph = (struct tcphdr *)((uint32_t *)iph + iph->ihl);

	t = (uint8_t *)&iph->saddr;
	printf("%u.%u.%u.%u", t[0], t[1], t[2], t[3]);
	if (iph->protocol == IPPROTO_TCP || iph->protocol == IPPROTO_UDP)
		//printf("(%d)", ntohs(udph->source));
		printf("(%d)", NTOHS(udph->source));

	printf(" -> ");

	t = (uint8_t *)&iph->daddr;
	printf("%u.%u.%u.%u", t[0], t[1], t[2], t[3]);
	if (iph->protocol == IPPROTO_TCP || iph->protocol == IPPROTO_UDP)
		//printf("(%d)", ntohs(udph->dest));
		printf("(%d)", NTOHS(udph->dest));

	//printf(" IP_ID=%d", ntohs(iph->id));
	printf(" IP_ID=%d", NTOHS(iph->id));
	printf(" TTL=%d ", iph->ttl);

	switch (iph->protocol) {
	case IPPROTO_TCP:
		printf("TCP ");
		break;
	case IPPROTO_UDP:
		printf("UDP ");
		break;
	default:
		printf("protocol %d ", iph->protocol);
		goto done;
	}
done:
	printf("len=%d\n", len);
  printf("<<<DumpPacket>>>-----------------------------------END--------------------------\n");

}


uint8_t * h_EthernetOutput(uint8_t *buf, uint16_t h_proto, unsigned char* src_haddr, unsigned char* dst_haddr, uint16_t iplen)
{
	struct ethhdr *ethh;
	int i;

	ethh = (struct ethhdr *)buf;

#if 1
	printf("dst_hwaddr: %02X:%02X:%02X:%02X:%02X:%02X\n",
				dst_haddr[0], dst_haddr[1], 
				dst_haddr[2], dst_haddr[3], 
				dst_haddr[4], dst_haddr[5]);
	printf("src_hwaddr: %02X:%02X:%02X:%02X:%02X:%02X\n",
				src_haddr[0], src_haddr[1], 
				src_haddr[2], src_haddr[3], 
				src_haddr[4], src_haddr[5]);
#endif

	for (i = 0; i < ETH_ALEN; i++) {
		ethh->h_source[i] = src_haddr[i];
		ethh->h_dest[i] = dst_haddr[i];
	}
	ethh->h_proto = HTONS(h_proto);

	return (uint8_t *)(ethh + 1);
}


__device__ uint8_t * EthernetOutput(uint8_t *buf, uint16_t h_proto, unsigned char* src_haddr, unsigned char* dst_haddr, uint16_t iplen)
//uint8_t * EthernetOutput(uint8_t *buf, uint16_t h_proto, unsigned char* src_haddr, unsigned char* dst_haddr, uint16_t iplen)
{
	struct ethhdr *ethh;
	int i;

	ethh = (struct ethhdr *)buf;

#if 1
	printf("dst_hwaddr: %02X:%02X:%02X:%02X:%02X:%02X\n",
				dst_haddr[0], dst_haddr[1], 
				dst_haddr[2], dst_haddr[3], 
				dst_haddr[4], dst_haddr[5]);
	printf("src_hwaddr: %02X:%02X:%02X:%02X:%02X:%02X\n",
				src_haddr[0], src_haddr[1], 
				src_haddr[2], src_haddr[3], 
				src_haddr[4], src_haddr[5]);
#endif

	for (i = 0; i < ETH_ALEN; i++) {
		ethh->h_source[i] = src_haddr[i];
		ethh->h_dest[i] = dst_haddr[i];
	}
	ethh->h_proto = HTONS(h_proto);

	return (uint8_t *)(ethh + 1);
}

static int h_ARPOutput(uint8_t * buf, int opcode, uint32_t src_ip, uint32_t dst_ip, unsigned char *dst_haddr)
{
	if (!dst_haddr)
		return -1;
  printf("ARPOUTPUT\n");
	/* Allocate a buffer */

#if 1
  uint8_t src_haddr[ETH_ALEN];
  // ckjung: 00:1b:21:bc:11:52
  src_haddr[0] = 0x00;
  src_haddr[1] = 0x1b;
  src_haddr[2] = 0x21;
  src_haddr[3] = 0xbc;
  src_haddr[4] = 0x11;
  src_haddr[5] = 0x52;

	struct arphdr *arph = (struct arphdr *)(uintptr_t)h_EthernetOutput(
    buf, ETH_P_ARP, src_haddr, dst_haddr, sizeof(struct arphdr));

	if (!arph) {
    printf("ERROR\n");
		return -1;
	}
#else 
	struct arphdr *arph = NULL;
#endif
	/* Fill arp header */
	arph->ar_hrd = HTONS(arp_hrd_ethernet);
	arph->ar_pro = HTONS(ETH_P_IP);
	//arph->ar_pro = htons(0x0800);
	arph->ar_hln = ETH_ALEN;
	arph->ar_pln = 4;
	arph->ar_op = HTONS(opcode);

	/* Fill arp body */
#if 0 // HONESTCHOI : TODO
	arph->ar_sip = CONFIG.eths[nif].ip_addr;
#endif 
	arph->ar_sip = src_ip;
	arph->ar_tip = dst_ip;

#if 0 // HONESTCHOI : TODO
	memcpy(arph->ar_sha, CONFIG.eths[nif].haddr, arph->ar_hln);
	if (target_haddr) {
		memcpy(arph->ar_tha, target_haddr, arph->ar_hln);
	} else {
		memcpy(arph->ar_tha, dst_haddr, arph->ar_hln);
	}
#endif
	memcpy(arph->ar_sha, src_haddr, arph->ar_hln);
  memcpy(arph->ar_tha, dst_haddr, arph->ar_hln);
	memset(arph->pad, 0, ARP_PAD_LEN);

#if DBGMSG
//	DumpARPPacket(arph);
#endif

	return 0;
}

__device__ static int ARPOutput(uint8_t * buf, int opcode, uint32_t src_ip, uint32_t dst_ip, unsigned char *dst_haddr)
//static int ARPOutput(uint8_t * buf, int opcode, uint32_t src_ip, uint32_t dst_ip, unsigned char *dst_haddr)
{
	if (!dst_haddr)
		return -1;
  printf("ARPOUTPUT\n");
	/* Allocate a buffer */

#if 1
  uint8_t src_haddr[ETH_ALEN];
  // ckjung: 00:1b:21:bc:11:52
  src_haddr[0] = 0x00;
  src_haddr[1] = 0x1b;
  src_haddr[2] = 0x21;
  src_haddr[3] = 0xbc;
  src_haddr[4] = 0x11;
  src_haddr[5] = 0x52;

	struct arphdr *arph = (struct arphdr *)(uintptr_t)EthernetOutput(
    buf, ETH_P_ARP, src_haddr, dst_haddr, sizeof(struct arphdr));

	if (!arph) {
    printf("ERROR\n");
		return -1;
	}
#else 
	struct arphdr *arph = NULL;
#endif
	/* Fill arp header */
	arph->ar_hrd = HTONS(arp_hrd_ethernet);
	arph->ar_pro = HTONS(ETH_P_IP);
	//arph->ar_pro = htons(0x0800);
	arph->ar_hln = ETH_ALEN;
	arph->ar_pln = 4;
	arph->ar_op = HTONS(opcode);

	/* Fill arp body */
#if 0 // HONESTCHOI : TODO
	arph->ar_sip = CONFIG.eths[nif].ip_addr;
#endif 
	arph->ar_sip = src_ip;
	arph->ar_tip = dst_ip;

#if 0 // HONESTCHOI : TODO
	memcpy(arph->ar_sha, CONFIG.eths[nif].haddr, arph->ar_hln);
	if (target_haddr) {
		memcpy(arph->ar_tha, target_haddr, arph->ar_hln);
	} else {
		memcpy(arph->ar_tha, dst_haddr, arph->ar_hln);
	}
#endif
	memcpy(arph->ar_sha, src_haddr, arph->ar_hln);
  memcpy(arph->ar_tha, dst_haddr, arph->ar_hln);
	memset(arph->pad, 0, ARP_PAD_LEN);

#if DBGMSG
	DumpARPPacket(arph);
#endif

	return 0;
}
int my_pin_buffer(my_t g, unsigned long addr, size_t size, uint64_t p2p_token, uint32_t va_space, my_mh_t *handle)
{
    int ret = 0;
    int retcode;

    struct MYDRV_IOC_PIN_BUFFER_PARAMS params;
    params.addr = addr;
    params.size = size;
    params.p2p_token = p2p_token;
    params.va_space = va_space;
    params.handle = 0;
    //params.buf_name = bname;

    retcode = ioctl(g->fd, MYDRV_IOC_PIN_BUFFER, &params);
    if (0 != retcode) {
        ret = errno;
        my_err("ioctl error (errno=%d)\n", ret);
    }
    *handle = params.handle;

    return ret;
}

my_t my_open()
{
    my_t m = NULL;
    const char *myinode = "/dev/mydrv";

    m = (my_t) calloc(1, sizeof(*m));
    if (!m) {
        //my_err("error while allocating memory\n");
        return NULL;
    }

    int fd = open(myinode, O_RDWR);
    if (-1 == fd ) {
        int ret = errno;
        //my_err("error opening driver (errno=%d/%s)\n", ret, strerror(ret));
        free(m);
        return NULL;
    }

    m->fd = fd;

    return m;
}
#if 0
static inline uint32_t myrand(uint64_t *seed) 
{
	*seed = *seed * 1103515245 + 12345;
	return (uint32_t)(*seed >> 32);
}
#endif


// CKJUNG ~

__global__ void make_pkt(int* g_mem, int size)
{
	printf("[%s][%d]----------------------------------------START---make_pkt------------DEVICE.\n", __FUNCTION__, __LINE__);

	// ARP call
	uint8_t* pktBuf;
	pktBuf = (uint8_t *)malloc(60);
	
	// For now, static ip address 
	uint8_t src_tmp[] = {0x0a, 0x00, 0x00, 0x02};
	uint8_t dst_tmp[] = {0x0a, 0x00, 0x00, 0x01};
	uint32_t src_ip;                             
	memcpy(&src_ip, src_tmp, 4);                 
	uint32_t dst_ip;                             
	memcpy(&dst_ip, dst_tmp, 4);  
	unsigned char dst_haddr[ETH_ALEN];
	memset(dst_haddr, 0xFF, ETH_ALEN);

// Just for check the value of variables
#if 0	
	uint8_t *t,*s;
	t = (uint8_t *)&src_ip;	
	s = (uint8_t *)&dst_haddr;	
	printf("[%s][%d]____CKJUNG___src_ip: %u.%u.%u.%u.\n", __FUNCTION__, __LINE__, t[0],t[1],t[2],t[3]);
	printf("[%s][%d]____CKJUNG___dst_haddr: %u.%u.%u.%u.\n", __FUNCTION__, __LINE__, s[0],s[1],s[2],s[3]);
#endif
	ARPOutput(pktBuf, arp_op_request, src_ip, dst_ip, dst_haddr);
	
	DumpPacket(pktBuf, 60);
	
#if 1
	for(int i=0; i< size/sizeof(int); i++) {
		g_mem[i] = 0;
	}

	printf("[%s][%d]____CKJUNG__ HERE\n", __FUNCTION__, __LINE__);

	for(int i=0; i< 512; i++) { // making 100 arpreq packets
		for(int j=0; j < 15; j++) { 
			//g_mem[4096*i+j] = pktBuf[j];
			memcpy(g_mem+4096*i+j, pktBuf + 4*j, 4);
		}
	}
	// For check
#if 0
	int count = 0;
	for(int j = 0; j < 4096*512; j++) {
		if(*((uint16_t*)&g_mem[j]) == 0x0608 || *((uint16_t*)&g_mem[j]) == 0x0806 ){
			count++;
			printf("[%s][%d] %d We found ethernet type 0x%02x%02x on %dth memory.\n", 
					__FUNCTION__ , __LINE__, count,g_mem[j], g_mem[j+1], j);
		}
	}
#endif
#else
#endif
	printf("[%s][%d]------------------------------------------END---make_pkt------------DEVICE.\n", __FUNCTION__, __LINE__);

}

// ~CKJUNG


#if 0
__global__ void init_data(int size, int* d_A)
#else
void init_data(int size, unsigned char* h_mem, int* d_A)
#endif
{
	for(int i=0; i<size/2; i++)
		h_mem[i] = 0;
//	hipMemcpy(d_A, h_mem, size, hipMemcpyHostToDevice);

	char * arp_req = (char*) malloc(100*sizeof(char));

	//uint64_t seed = 0;
	//build_packet(arp_req, 1000, &seed);  

	uint8_t* buf;
	buf = (uint8_t *) malloc(60);
	uint8_t src_tmp[] = {0x0a, 0x00, 0x00, 0x02};
	uint8_t dst_tmp[] = {0x0a, 0x00, 0x00, 0x01};
	uint32_t src_ip;
	memcpy(&src_ip, src_tmp, 4);
	uint32_t dst_ip;
	memcpy(&dst_ip, dst_tmp, 4);

	unsigned char dst_haddr[ETH_ALEN];
	memset(dst_haddr, 0xFF, ETH_ALEN);

	h_ARPOutput(buf, arp_op_request, src_ip, dst_ip, dst_haddr);

	//DumpPacket(buf, 60);

	for(int i=0; i< 512; i++) { // making 100 arpreq packets
		for(int j=0; j < 60; j++) { 
			 h_mem[4096*i+j] = buf[j];
			//d_A[4096*i+j] = buf[j];
		}
	}

	// for check
	for(int j = 0; j < 4096*512; j++) {
		if(*((uint16_t*)&h_mem[j]) == 0x0608 || *((uint16_t*)&h_mem[j]) == 0x0806 ){
			printf("[%s][%d] We found ethernet type 0x%02x%02x on %dth memory.\n", 
					__FUNCTION__ , __LINE__, h_mem[j], h_mem[j+1], j);
		}
	}

	//hipMemcpy(d_A, h_mem, size, hipMemcpyHostToDevice);

}

int curr_num;
#if 0
__global__ void check_data(int size, unsigned char* h_mem, int* d_A)
#else
void check_data(int size, unsigned char* h_mem, int* d_A)
#endif
{
  printf("[%s][%d]----------------------------------------START---check_data------------HOST.\n", __FUNCTION__, __LINE__);
  hipMemcpy(h_mem, d_A, size, hipMemcpyDeviceToHost);
  uint32_t offset_for_rx = 511 * 4096 * 12;
  h_mem += offset_for_rx;

#if 0
  for(int outer = 0; outer < size-4095; outer+=4096) {
	if(iter[outer+4095] != 0) {
      int inner = outer;
      while(iter[inner] != 0)
	    printf("%c", iter[inner++]);
	}
  }
#else
  int dirty_cnt = 0;
  int pkt_cnt = 0;
  //const int DUMP_SIZE = 30;
  for(int i = 0; i < size-offset_for_rx; i++) {
    if(*((uint16_t*)&h_mem[i]) == 0x0608 || *((uint16_t*)&h_mem[i]) == 0x0806 ) {//(h_mem[i] == 0x08 && h_mem[i+1] == 0x00) ) {
      pkt_cnt++;
      //printf("\n%s][%d] PACKET BEGINS-------------------------------------\n", __FUNCTION__, __LINE__);
      //printf("\n\n");
      printf("[%s][%d] We found ethernet type 0x%02x%02x on %dth memory. (%d pkts, curr:%d)\n", __FUNCTION__ , __LINE__, h_mem[i], h_mem[i+1],i, pkt_cnt, curr_num);
      //printf("[%s][%d] We found ethernet type 0x%04x on %dth memory.\n", __FUNCTION__ , __LINE__, *((uint16_t*)&h_mem[i]), i);
      
     // h_DumpPacket((uint8_t*)&h_mem[i-12], 60);
    } else if (h_mem[i] == 0xAB || h_mem[i] == 0xBA) {
      for(int packet_iter = i; packet_iter < (i+20); packet_iter++) {
        if((packet_iter-i) % 4 == 0)
          printf("\n");
        printf("%3d:0x%02x\t", packet_iter-i, h_mem[packet_iter]);
      }
	}
  }
  for(int i = 0; i < size-offset_for_rx; i++) 
    if(h_mem[i] != 0) 
      dirty_cnt++;
#endif
  //printf("[%s][%d] dirty_cnt:[%d]\n", __FUNCTION__ , __LINE__, dirty_cnt);
  printf("[%s][%d]----------------------------------------END---check_data------------HOST.\n", __FUNCTION__, __LINE__);
}

union ixgbe_adv_tx_desc {
	struct {
		__le64 buffer_addr; /* Address of descriptor's data buf */
		__le32 cmd_type_len;
		__le32 olinfo_status;
	} read;
	struct {
		__le64 rsvd; /* Reserved */
		__le32 nxtseq_seed;
		__le32 status;
	} wb;
};


#define COMPILER_BARRIER() asm volatile("" ::: "memory")
#define cpu_to_le32(x) ((__le32)(__swab32)(x))

#ifndef __USE_GPU__
void doorbell_test(void * io_addr)
#else
__global__ void doorbell_test(void * io_addr, void * desc, uint32_t curr)
#endif
{
  //printf("[%s][%d] curr:%d\n", __FUNCTION__, __LINE__, curr);
  printf("[%s][%d] curr:%d---------------------------START---doorbell_test------------DEVICE.\n", __FUNCTION__, __LINE__,curr);

  unsigned char *db[12];
  for(int i=0; i<12; i++)
    db[i] = ((unsigned char *)io_addr) + IXGBE_TDT(i);

  COMPILER_BARRIER();
#if 0
  for(int i=0; i<12;i++)
    printf("[%s][%d] %d\n", __FUNCTION__, __LINE__, *(volatile unsigned int *)db[i] );
#endif
  volatile union ixgbe_adv_tx_desc* tx_desc;
  tx_desc = (union ixgbe_adv_tx_desc*) desc;

  int num_packets = 5;
  int index = curr;
  for(int i=0; i<curr; i++)
    tx_desc++;
  for(int i=0; i<num_packets; i++) {
    tx_desc->read.cmd_type_len |= 60;
    tx_desc->read.olinfo_status = 0xf0000;
    tx_desc++;
    index++;
    if(index == 512)
      tx_desc = (union ixgbe_adv_tx_desc*) desc;
  }
  unsigned long tail_val = (unsigned long)((curr + num_packets) % 512);
  printf("[%s][%d] tail_val:%x %x\n", __FUNCTION__, __LINE__, tail_val, tail_val >> 8);

  //*(volatile unsigned long*)db[0] = (unsigned long)((curr + num_packets) % 512);
  *(volatile unsigned long*)db[0] = tail_val;
  //*(volatile unsigned long*)(db[0]+1) = tail_val >> 8;
  printf("[%s][%d] curr:%d---------------------------END---doorbell_test------------DEVICE.\n", __FUNCTION__, __LINE__,curr);
}

#if 1
// YHOON~ for test
void yhoon_xmit_arp()
{
  printf("[%s][%d]---------------------------START---yhoon_xmit_arp------------HOST.\n", __FUNCTION__, __LINE__);
  printf("[%s][%d] sizeof tx_desc:%lu \n", __FUNCTION__, __LINE__, sizeof(union ixgbe_adv_tx_desc));
  const char *myinode = "/dev/ixgbe";
  int fd = open(myinode, O_RDWR);
  //uint64_t ptr = 1234;
  //ioctl(fd, 0, &ptr);
  ioctl(fd, 1);

  printf("cpu_to_le32 test:%x\n", htonl(60));
#ifndef __USE_GPU__
  void* dummy2;
  ASSERTRT(hipMalloc(&dummy2, 4096*8));


#error "use_gpu"

  doorbell_test(dummy2);

#else
  void* dBAR;
  const size_t IXGBE_BAR0_SIZE = 4096*8; // A rough calculation
  void* ixgbe_bar0_host_addr = mmap(0, IXGBE_BAR0_SIZE*5 , PROT_READ | PROT_WRITE, MAP_SHARED, fd, 0);
  hipHostRegister(ixgbe_bar0_host_addr, IXGBE_BAR0_SIZE, hipHostRegisterIoMemory);
  hipHostGetDevicePointer((void**)&dBAR, (void*)ixgbe_bar0_host_addr, 0);

  unsigned char *db[12];
  for(int i=0; i<1; i++) {
    db[i] = ((unsigned char *)ixgbe_bar0_host_addr) + IXGBE_TDT(i);
#if 1
    printf("db[%d]: %d\n", i, *(volatile unsigned int *)db[i]);
#endif
  }

  ixgbe_adv_tx_desc* desc = (ixgbe_adv_tx_desc*)((unsigned char*)ixgbe_bar0_host_addr + IXGBE_BAR0_SIZE);
  hipHostRegister(desc, IXGBE_BAR0_SIZE * 4, hipHostRegisterIoMemory);
  //hipHostRegister(desc, sizeof(ixgbe_adv_tx_desc), hipHostRegisterIoMemory);
  void* tx_desc;
  hipHostGetDevicePointer((void**)&tx_desc, (void*)desc, 0);

  uint32_t curr_tx_index_q_zero = *(volatile unsigned int *)db[0];
  //printf("curr_tx_index_q_zero: %u\n", curr_tx_index_q_zero);
  doorbell_test<<< 1,1 >>>(dBAR, tx_desc, curr_tx_index_q_zero);
  //if(hipSuccess != hipDeviceSynchronize())
  //	cudaCheckErrors("doorbell_sync_error!");
  curr_num = curr_tx_index_q_zero;
#endif
  // ~YHOON
  printf("[%s][%d]---------------------------END---yhoon_xmit_arp------------HOST.\n", __FUNCTION__, __LINE__);
}
#else
// YHOON~ for test
void yhoon_xmit_arp()
{
  const char *myinode = "/dev/ixgbe";
  int fd = open(myinode, O_RDWR);
  uint64_t ptr = 1234;
  ioctl(fd, 0, &ptr);
  // ~YHOON
}
#endif

int main(int argc, char *argv[])
{
  printf("[%s][%d] main\n", __FUNCTION__, __LINE__);
  int dev_id = 0;
  size_t _size = 50*1024*1024; //50*1024*1024;

  // CKJUNG, meaning of this?
  size_t size = (_size + GPU_PAGE_SIZE - 1) & GPU_PAGE_MASK;
  //printf("[%s][%d]____CKJUNG__size: %lu\n", __FUNCTION__, __LINE__, size);

  int n_devices = 0;

  unsigned char* h_mem = (unsigned char*)malloc(size*sizeof(unsigned char)); 


  hipGetDeviceCount(&n_devices);

  hipDeviceProp_t prop;
  for (int n=0; n<n_devices; ++n) {
    hipGetDeviceProperties(&prop,n);
    OUT << "GPU id:" << n << " name:" << prop.name 
      << " PCI domain: " << prop.pciDomainID 
      << " bus: " << prop.pciBusID 
      << " device: " << prop.pciDeviceID << endl;
  }
  OUT << "selecting device " << dev_id << endl;
  OUT << "_size: " << _size << "  size: " << size << endl;

  ASSERTRT(hipSetDevice(dev_id));

  // Test
  void* dummy;
  ASSERTRT(hipMalloc(&dummy, 0));

  int* d_A;
  ASSERTRT(hipMalloc((void**)&d_A, size));
  OUT << "device ptr: " << hex << d_A << dec << endl;
  
  unsigned int flag = 1;
  ASSERTDRV(hipPointerSetAttribute(&flag, HIP_POINTER_ATTRIBUTE_SYNC_MEMOPS, (hipDeviceptr_t) d_A));

  my_t g = my_open();

  ASSERT_NEQ(g, (void*)0);

  my_mh_t mh;
  if (my_pin_buffer(g, (hipDeviceptr_t)d_A, size, 0, 0, &mh)  != 0)
    OUT << "NOT_EQ" << endl;

  //hipMemcpy(&h_tmp, (int *)d_A, sizeof(int), hipMemcpyDeviceToHost);
  //OUT << "after pinning: " << h_tmp << endl;

  //init_data(size, h_mem, d_A);
  make_pkt<<<1,1>>>(d_A, size);


  // call ixgbe_xmit_yhoon in ixgbe_main.c
  yhoon_xmit_arp();

  if(hipSuccess != hipDeviceSynchronize())
	 cudaCheckErrors("make_pkt_error"); 

  int count = 0;
  while(count < 1) {
    check_data(size, h_mem, d_A);
    usleep(1*1000*1000);
    count++;
  }
  OUT << "END" << endl;
  hipFree(d_A);

  return 0;
}
