#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <memory.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <stdarg.h>
#include <string.h>
#include <sys/types.h>
#include <sys/socket.h>
#include <sys/time.h>
#include <sys/stat.h>
#include <sys/mman.h>
#include <fcntl.h>
#include <errno.h>
#include <netdb.h>
#include <malloc.h>
#include <getopt.h>
#include <arpa/inet.h>
#include <sys/ioctl.h>
#include <time.h>
#include <asm/types.h>

#include <linux/if_ether.h>
#include <linux/ip.h>
#include <linux/udp.h>
#include "arp.h" 

#include <linux/netdevice.h>   /* struct device, and other headers */
#include <linux/tcp.h>         /* struct tcphdr */
#include <linux/in6.h>
#define ETH_ALEN  6 // YHOON
#define ARP_PAD_LEN 18 // YHOON

#define HTONS(n) (((((unsigned short)(n) & 0xFF)) << 8) | (((unsigned short)(n) & 0xFF00) >> 8))
#define NTOHS(n) (((((unsigned short)(n) & 0xFF)) << 8) | (((unsigned short)(n) & 0xFF00) >> 8))

#define HTONL(n) (((((unsigned long)(n) & 0xFF)) << 24) | \
	((((unsigned long)(n) & 0xFF00)) << 8) | \
	((((unsigned long)(n) & 0xFF0000)) >> 8) | \
		  ((((unsigned long)(n) & 0xFF000000)) >> 24))

#define NTOHL(n) (((((unsigned long)(n) & 0xFF)) << 24) | \
	((((unsigned long)(n) & 0xFF00)) << 8) | \
	((((unsigned long)(n) & 0xFF0000)) >> 8) | \
		  ((((unsigned long)(n) & 0xFF000000)) >> 24))


#define cudaCheckErrors(msg) do { hipError_t __err = hipGetLastError(); if (__err != hipSuccess) { \
			fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
					msg, hipGetErrorString(__err), \
					__FILE__, __LINE__); \
				fprintf(stderr, "*** FAILED - ABORTING\n"); \
				exit(1); \
		} \
	} while (0)

#include <asm/types.h>
using namespace std;

#include "packet_man.h"
#include "mydrv/mydrv.h"
#include "common.hpp"


#define OUT cout


#define HTONS(n) (((((unsigned short)(n) & 0xFF)) << 8) | (((unsigned short)(n) & 0xFF00) >> 8))
#define NTOHS(n) (((((unsigned short)(n) & 0xFF)) << 8) | (((unsigned short)(n) & 0xFF00) >> 8))
enum mycopy_msg_level {
    MYCOPY_MSG_DEBUG = 1,
    MYCOPY_MSG_INFO,
    MYCOPY_MSG_WARN,
    MYCOPY_MSG_ERROR
};

#define IXGBE_TDT(_i)   (0x06018 + ((_i) * 0x40))

//#define __USE_PKT_MONITOR__
#define NUM_PACKETS 500
#define THREADS_PER_TB 512 

struct my {
  int fd;
};

/*----------------------------------------------------------------------------*/
enum arp_hrd_format
{
	arp_hrd_ethernet = 1
};
/*----------------------------------------------------------------------------*/
enum arp_opcode
{
	arp_op_request = 1, 
	arp_op_reply = 2, 
};

// YHOON
struct arphdr
{
  uint16_t ar_hrd;      /* hardware address format */
  uint16_t ar_pro;      /* protocol address format */
  uint8_t ar_hln;       /* hardware address length */
  uint8_t ar_pln;       /* protocol address length */
  uint16_t ar_op;       /* arp opcode */

  uint8_t ar_sha[ETH_ALEN]; /* sender hardware address */
  uint32_t ar_sip;      /* sender ip address */
  uint8_t ar_tha[ETH_ALEN]; /* targe hardware address */
  uint32_t ar_tip;      /* target ip address */

  uint8_t pad[ARP_PAD_LEN];
} __attribute__ ((packed));

static int my_msg_level = MYCOPY_MSG_ERROR;
static int my_enable_logging = 1;

static void my_msg(enum mycopy_msg_level lvl, const char* fmt, ...)
{
    if (-1 == my_enable_logging) {
        const char *env = getenv("MYCOPY_ENABLE_LOGGING");
        if (env)
            my_enable_logging = 1;
        else
            my_enable_logging = 0;

        env = getenv("MYCOPY_LOG_LEVEL");
        if (env)
            my_msg_level = atoi(env);
    }
    if (my_enable_logging) {
        if (lvl >= my_msg_level) {
            va_list ap;
            va_start(ap, fmt);
            vfprintf(stderr, fmt, ap);
        }
    }
}

#define my_dbg(FMT, ARGS...)  my_msg(MYCOPY_MSG_DEBUG, "DBG:  " FMT, ## ARGS)
#define my_dbgc(C, FMT, ARGS...)  do { static int my_dbg_cnt=(C); if (my_dbg_cnt) { my_dbg(FMT, ## ARGS); --my_dbg_cnt; }} while (0)
#define my_info(FMT, ARGS...) my_msg(MYCOPY_MSG_INFO,  "INFO: " FMT, ## ARGS)
#define my_warn(FMT, ARGS...) my_msg(MYCOPY_MSG_WARN,  "WARN: " FMT, ## ARGS)
#define my_err(FMT, ARGS...)  my_msg(MYCOPY_MSG_ERROR, "ERR:  " FMT, ## ARGS)

#define DBGMSG 1

__device__ uint32_t d_curr_of_processing_queue = 0;
__device__ uint32_t d_processing_queue_size = 8 * 512;

void h_DumpARPPacket(struct arphdr *arph)
{
	uint8_t *t;

	printf("ARP header: \n");
	printf("Hardware type: %d (len: %d), "
			"protocol type: %d (len: %d), opcode: %d\n", 
			//ntohs(arph->ar_hrd), arph->ar_hln, 
			NTOHS(arph->ar_hrd), arph->ar_hln, 
			//ntohs(arph->ar_pro), arph->ar_pln, ntohs(arph->ar_op));
			NTOHS(arph->ar_pro), arph->ar_pln, NTOHS(arph->ar_op));
	t = (uint8_t *)&arph->ar_sip;
	printf("Sender IP: %u.%u.%u.%u, "
			"haddr: %02X:%02X:%02X:%02X:%02X:%02X\n", 
			t[0], t[1], t[2], t[3], 
			arph->ar_sha[0], arph->ar_sha[1], arph->ar_sha[2], 
			arph->ar_sha[3], arph->ar_sha[4], arph->ar_sha[5]);
	t = (uint8_t *)&arph->ar_tip;
	printf("Target IP: %u.%u.%u.%u, "
			"haddr: %02X:%02X:%02X:%02X:%02X:%02X\n", 
			t[0], t[1], t[2], t[3], 
			arph->ar_tha[0], arph->ar_tha[1], arph->ar_tha[2], 
			arph->ar_tha[3], arph->ar_tha[4], arph->ar_tha[5]);
}

__device__ void DumpARPPacket(struct arphdr *arph)
//void DumpARPPacket(struct arphdr *arph)
{
	uint8_t *t;

	printf("ARP header: \n");
	printf("Hardware type: %d (len: %d), "
			"protocol type: %d (len: %d), opcode: %d\n", 
			//ntohs(arph->ar_hrd), arph->ar_hln, 
			NTOHS(arph->ar_hrd), arph->ar_hln, 
			//ntohs(arph->ar_pro), arph->ar_pln, ntohs(arph->ar_op));
			NTOHS(arph->ar_pro), arph->ar_pln, NTOHS(arph->ar_op));
	t = (uint8_t *)&arph->ar_sip;
	printf("Sender IP: %u.%u.%u.%u, "
			"haddr: %02X:%02X:%02X:%02X:%02X:%02X\n", 
			t[0], t[1], t[2], t[3], 
			arph->ar_sha[0], arph->ar_sha[1], arph->ar_sha[2], 
			arph->ar_sha[3], arph->ar_sha[4], arph->ar_sha[5]);
	t = (uint8_t *)&arph->ar_tip;
	printf("Target IP: %u.%u.%u.%u, "
			"haddr: %02X:%02X:%02X:%02X:%02X:%02X\n", 
			t[0], t[1], t[2], t[3], 
			arph->ar_tha[0], arph->ar_tha[1], arph->ar_tha[2], 
			arph->ar_tha[3], arph->ar_tha[4], arph->ar_tha[5]);
}


__device__ uint8_t * EthernetOutput(uint8_t *buf, uint16_t h_proto, unsigned char* src_haddr, unsigned char* dst_haddr, uint16_t iplen)
//uint8_t * EthernetOutput(uint8_t *buf, uint16_t h_proto, unsigned char* src_haddr, unsigned char* dst_haddr, uint16_t iplen)
{
	struct ethhdr *ethh;
	int i;

	ethh = (struct ethhdr *)buf;

#if 0
	printf("dst_hwaddr: %02X:%02X:%02X:%02X:%02X:%02X\n",
				dst_haddr[0], dst_haddr[1], 
				dst_haddr[2], dst_haddr[3], 
				dst_haddr[4], dst_haddr[5]);
	printf("src_hwaddr: %02X:%02X:%02X:%02X:%02X:%02X\n",
				src_haddr[0], src_haddr[1], 
				src_haddr[2], src_haddr[3], 
				src_haddr[4], src_haddr[5]);
#endif

	for (i = 0; i < ETH_ALEN; i++) {
		ethh->h_source[i] = src_haddr[i];
		ethh->h_dest[i] = dst_haddr[i];
	}
	ethh->h_proto = HTONS(h_proto);

	return (uint8_t *)(ethh + 1);
}

__device__ void DumpPacket(uint8_t *buf, int len)
//void DumpPacket(uint8_t *buf, int len)
{
  printf("<<<DumpPacket>>>----------------------------------------------------------------\n");
	struct ethhdr *ethh;
	struct iphdr *iph;
	struct udphdr *udph;
	//struct tcphdr *tcph;
	uint8_t *t;

	ethh = (struct ethhdr *)buf;
	//if (ntohs(ethh->h_proto) != ETH_P_IP) {
	if (NTOHS(ethh->h_proto) != ETH_P_IP) {
		printf("%02X:%02X:%02X:%02X:%02X:%02X -> %02X:%02X:%02X:%02X:%02X:%02X ",
				ethh->h_source[0],
				ethh->h_source[1],
				ethh->h_source[2],
				ethh->h_source[3],
				ethh->h_source[4],
				ethh->h_source[5],
				ethh->h_dest[0],
				ethh->h_dest[1],
				ethh->h_dest[2],
				ethh->h_dest[3],
				ethh->h_dest[4],
				ethh->h_dest[5]);

		//printf("protocol %04hx  \n", ntohs(ethh->h_proto));
		printf("protocol %04hx  \n", NTOHS(ethh->h_proto));

    //if(ntohs(ethh->h_proto) == ETH_P_ARP)
    if(NTOHS(ethh->h_proto) == ETH_P_ARP)
      DumpARPPacket((struct arphdr *) (ethh + 1));
		goto done;
	}

	iph = (struct iphdr *)(ethh + 1);
	udph = (struct udphdr *)((uint32_t *)iph + iph->ihl);
	//tcph = (struct tcphdr *)((uint32_t *)iph + iph->ihl);

	t = (uint8_t *)&iph->saddr;
	printf("%u.%u.%u.%u", t[0], t[1], t[2], t[3]);
	if (iph->protocol == IPPROTO_TCP || iph->protocol == IPPROTO_UDP)
		//printf("(%d)", ntohs(udph->source));
		printf("(%d)", NTOHS(udph->source));

	printf(" -> ");

	t = (uint8_t *)&iph->daddr;
	printf("%u.%u.%u.%u", t[0], t[1], t[2], t[3]);
	if (iph->protocol == IPPROTO_TCP || iph->protocol == IPPROTO_UDP)
		//printf("(%d)", ntohs(udph->dest));
		printf("(%d)", NTOHS(udph->dest));

	//printf(" IP_ID=%d", ntohs(iph->id));
	printf(" IP_ID=%d", NTOHS(iph->id));
	printf(" TTL=%d ", iph->ttl);

	switch (iph->protocol) {
	case IPPROTO_TCP:
		printf("TCP ");
		break;
	case IPPROTO_UDP:
		printf("UDP ");
		break;
	default:
		printf("protocol %d ", iph->protocol);
		goto done;
	}
done:
	printf("len=%d\n", len);
  printf("<<<DumpPacket>>>-----------------------------------END--------------------------\n");

}


__device__ static int ARPOutput(uint8_t * d_tx_pkt_buffer, int opcode, uint32_t src_ip, uint32_t dst_ip, unsigned char *dst_haddr)
{
	if (!dst_haddr)
		return -1;

  //printf("\n\n\n[%s][%d] Enters\n", __FUNCTION__, __LINE__);
  // ckjung: 00:1b:21:bc:11:52
  uint8_t src_haddr[ETH_ALEN] = {0x00, 0x1b, 0x21, 0xbc, 0x11, 0x52};
	struct arphdr *arph = 
    (struct arphdr *)(uintptr_t)EthernetOutput(d_tx_pkt_buffer, ETH_P_ARP, src_haddr, dst_haddr, sizeof(struct arphdr));

	if (!arph) {
    printf("ERROR\n");
		return -1;
	}
	/* Fill arp header */
	arph->ar_hrd = HTONS(arp_hrd_ethernet);
	arph->ar_pro = HTONS(ETH_P_IP);
	arph->ar_hln = ETH_ALEN;
	arph->ar_pln = 4;
	arph->ar_op = HTONS(opcode);

	/* Fill arp body */
#if 0 // HONESTCHOI : TODO
	arph->ar_sip = CONFIG.eths[nif].ip_addr;
#endif 
	arph->ar_sip = src_ip;
	arph->ar_tip = dst_ip;

#if 0 // HONESTCHOI : TODO
	memcpy(arph->ar_sha, CONFIG.eths[nif].haddr, arph->ar_hln);
	if (target_haddr) {
		memcpy(arph->ar_tha, target_haddr, arph->ar_hln);
	} else {
		memcpy(arph->ar_tha, dst_haddr, arph->ar_hln);
	}
#endif
	memcpy(arph->ar_sha, src_haddr, arph->ar_hln);
  memcpy(arph->ar_tha, dst_haddr, arph->ar_hln);
	memset(arph->pad, 0, ARP_PAD_LEN);

#if 0
	DumpARPPacket(arph);
#endif

	return 0;
}

__device__ uint32_t offset_for_rx = 512 * 0x1000;
__device__ static volatile uint8_t *tx_tail_for_queue_zero;

__device__ static int ProcessARPRequest(struct arphdr *arph, uint8_t* d_tx_pkt_buffer)
{
  //printf("[%s][%d] Enters", __FUNCTION__, __LINE__);
	ARPOutput(d_tx_pkt_buffer, arp_op_reply, arph->ar_tip, arph->ar_sip, arph->ar_sha);
  return 0;
}

__device__ static int ProcessARPReply(struct arphdr *arph)
{
  DumpARPPacket(arph);
	//unsigned char *temp;
	return 0;
}

__device__ int ProcessARPPacket(unsigned char* d_tx_pkt_buffer, unsigned char *pkt_data, int len)
{
	struct arphdr *arph = (struct arphdr *)(pkt_data + sizeof(struct ethhdr));

  switch (NTOHS(arph->ar_op)) {
    case arp_op_request:
      printf("[%s][%d] arp_op_request\n", __FUNCTION__, __LINE__);
      ProcessARPRequest(arph, d_tx_pkt_buffer);
      break;

    case arp_op_reply:
      printf("[%s][%d] arp_op_reply\n", __FUNCTION__, __LINE__);
      ProcessARPReply(arph);
      break;

    default:
      printf("[%s][%d] ERROR. KNOWN OP CODE (%d)\n", __FUNCTION__, __LINE__, NTOHS(arph->ar_op));
      DumpPacket(pkt_data, 1500);
      break;
  }

  return 1;
}
uint8_t * h_EthernetOutput(uint8_t *buf, uint16_t h_proto, unsigned char* src_haddr, unsigned char* dst_haddr, uint16_t iplen)
{
	struct ethhdr *ethh;
	int i;

	ethh = (struct ethhdr *)buf;

#if 1
	printf("dst_hwaddr: %02X:%02X:%02X:%02X:%02X:%02X\n",
				dst_haddr[0], dst_haddr[1], 
				dst_haddr[2], dst_haddr[3], 
				dst_haddr[4], dst_haddr[5]);
	printf("src_hwaddr: %02X:%02X:%02X:%02X:%02X:%02X\n",
				src_haddr[0], src_haddr[1], 
				src_haddr[2], src_haddr[3], 
				src_haddr[4], src_haddr[5]);
#endif

	for (i = 0; i < ETH_ALEN; i++) {
		ethh->h_source[i] = src_haddr[i];
		ethh->h_dest[i] = dst_haddr[i];
	}
	ethh->h_proto = HTONS(h_proto);

	return (uint8_t *)(ethh + 1);
}

int my_pin_buffer(my_t g, unsigned long addr, size_t size, uint64_t p2p_token, uint32_t va_space, my_mh_t *handle)
{
    int ret = 0;
    int retcode;

    struct MYDRV_IOC_PIN_BUFFER_PARAMS params;
    params.addr = addr;
    params.size = size;
    params.p2p_token = p2p_token;
    params.va_space = va_space;
    params.handle = 0;
    //params.buf_name = bname;

    retcode = ioctl(g->fd, MYDRV_IOC_PIN_BUFFER, &params);
    if (0 != retcode) {
        ret = errno;
        my_err("ioctl error (errno=%d)\n", ret);
    }
    *handle = params.handle;

    return ret;
}

my_t my_open()
{
    my_t m = NULL;
    //const char *myinode = "/dev/mydrv";
    const char *myinode = "/dev/ixgbe";

    m = (my_t) calloc(1, sizeof(*m));
    if (!m) {
        //my_err("error while allocating memory\n");
        return NULL;
    }

    int fd = open(myinode, O_RDWR);
    if (-1 == fd ) {
        int ret = errno;
        //my_err("error opening driver (errno=%d/%s)\n", ret, strerror(ret));
        free(m);
        return NULL;
    }

    m->fd = fd;

    return m;
}
#if 0
static inline uint32_t myrand(uint64_t *seed) 
{
	*seed = *seed * 1103515245 + 12345;
	return (uint32_t)(*seed >> 32);
}
#endif

int curr_num;
__device__ void d_check_data(int size, volatile int* d_pkt_buffer, volatile int* flag)
{
  printf("[%s][%d] \n", __FUNCTION__ , __LINE__);
  volatile unsigned char * d_mem = (volatile unsigned char*)d_pkt_buffer;
  d_mem += offset_for_rx;

  int pkt_cnt = 0;
  //const int DUMP_SIZE = 30;
  //__threadfence_system();
  //printf("[%s][%d] before waiting %d\n", __FUNCTION__ , __LINE__, *flag);
  //WAIT_ON_MEM_NE(d_mem[2093068], 0);
  //WAIT_ON_MEM(*flag, 1);
  //printf("[%s][%d] after waiting %d\n", __FUNCTION__ , __LINE__, *flag);

#if 1
  while(pkt_cnt < 30000) {
    //printf("[%s][%d] BEFORE WAIT_ON_MEM %d\n", __FUNCTION__ , __LINE__, threadIdx.x);
    int head_index = 12 + 0x1000*threadIdx.x; // + 0x1000*(THREADS_PER_TB * pkt_cnt);
    //WAIT_ON_MEM_NE(d_mem[head_index], 0);
    pkt_cnt++;
    if(*((uint16_t*)&d_mem[head_index]) == 0x0608 || *((uint16_t*)&d_mem[head_index]) == 0x0806 ) {
      printf("[%s][%d] Ethernet type 0x%02x%02x on %dth memory. (threadIdx.x:%d %dpkts)\n", __FUNCTION__ , __LINE__, d_mem[head_index], d_mem[head_index+1], head_index, threadIdx.x, pkt_cnt);
      *((uint16_t*)&d_mem[head_index]) = 0;     
    }
  }
#endif

}

#if 0
__device__ void wait_for_something(volatile int * something_finished)
{
  BEGIN_SINGLE_THREAD_PART {
    while(!*something_finished) {
    }
  }
}
#endif

__device__ unsigned long tail_val;
__device__ volatile int server_done;
#define NUM_TB 2 
#define NUM_THREADS 512 

__global__ void clean_buffer(unsigned char* buffer, unsigned char* buffer2, int size, char* bm_worked_thread) 
{
  for(int i=0; i<size; i++) {
    buffer[i] = 0;
  }
  for(int i=0; i<4*size; i++) {
    buffer2[i] = 0;
  }
  for(int i=0; i<NUM_THREADS; i++) {
    bm_worked_thread[i] = 0;
  }

}

#define NUM_TURN 2
#define COMPILER_BARRIER() asm volatile("" ::: "memory")
#define cpu_to_le32(x) ((__le32)(__swab32)(x))

union ixgbe_adv_tx_desc {
	struct {
		__le64 buffer_addr; /* Address of descriptor's data buf */
		__le32 cmd_type_len;
		__le32 olinfo_status;
	} read;
	struct {
		__le64 rsvd; /* Reserved */
		__le32 nxtseq_seed;
		__le32 status;
	} wb;
};

__global__ void packet_processor(unsigned char* d_pkt_processing_queue, unsigned char* d_tx_pkt_buffer, int * tb_alloc_tbl, volatile int* num_turns, volatile uint8_t* io_addr)
{
  if(blockIdx.x == 0) {
    // can be placed somewhere else.
    BEGIN_SINGLE_THREAD_PART {
      tx_tail_for_queue_zero = io_addr + IXGBE_TDT(0);
      printf("tx_tail_for_queue_zero: %d\n", *tx_tail_for_queue_zero);
    } END_SINGLE_THREAD_PART;

    while(*num_turns < NUM_TURN) {
      while(!readNoCache(tb_alloc_tbl + 2)) { } 
      // currently d_curr_of_processing_queue is fixed to zero.
      unsigned char* rx_packet = &d_pkt_processing_queue[d_curr_of_processing_queue * 512 * 0x1000 + 0x1000*threadIdx.x];
      unsigned char* tx_packet = &d_tx_pkt_buffer[0x1000*threadIdx.x];
      //__threadfence_system();
      if(*(uint16_t*)(rx_packet+12) != 0) {
        ProcessARPPacket(tx_packet, rx_packet, 60);
        printf("[%s][%d] %d thread setting tx packet\n", __FUNCTION__, __LINE__, threadIdx.x);
        //DumpPacket((uint8_t*)tx_packet, 60);
      }
      tb_alloc_tbl[3] = 1; // set for tx 
      tb_alloc_tbl[2] = 0; // set for pp
    }
  }
}

__global__ void tx_handler(volatile unsigned char* d_pkt_buffer, int * tb_alloc_tbl, volatile uint8_t* io_addr, volatile union ixgbe_adv_tx_desc* tx_desc, volatile int* num_turns)
{
  if(blockIdx.x == 0) {
#if 1
    BEGIN_SINGLE_THREAD_PART {
      printf("TX [%2d,%2d]\n", blockIdx.x, threadIdx.x);
      while(*num_turns < NUM_TURN) {
        while(!readNoCache(tb_alloc_tbl + 3)) { } 
        tb_alloc_tbl[3]=0;

        //volatile unsigned char* tx_packet = &d_pkt_buffer[0x1000*threadIdx.x];
        //DumpPacket((uint8_t*)tx_packet, 60);
#if 1
        for(int i=0; i<512; i++) {
          if(*(uint16_t*)(d_pkt_buffer+ 0x1000*i) != 0) {
            printf("%dth memory, tx handler finds a packet to send.\n", i);
            DumpPacket((uint8_t*)(d_pkt_buffer+0x1000*i), 60);

            // TODO: currently, back to back. batching need to be implemented
            printf("TX: Try to send packets using %dth tx_desc.\n",i);
            //printf("tx_tail_for_queue_zero:%p\n", tx_tail_for_queue_zero);
            volatile union ixgbe_adv_tx_desc * desc = tx_desc + i;
            desc->read.cmd_type_len |= 60;
            desc->read.olinfo_status = 0xf0000;
            *(volatile unsigned long*) tx_tail_for_queue_zero = (unsigned long)i;
          }
        }
#endif
        //__threadfence_system();
      }
    } END_SINGLE_THREAD_PART;
#else
    while(*num_turns < NUM_TURN) { 
      while(!readNoCache(tb_alloc_tbl + 3)) { } 
      tb_alloc_tbl[3]=0;

      volatile unsigned char* tx_packet = &d_pkt_buffer[0x1000*threadIdx.x];
      if(*((uint16_t*)tx_packet) != 0) {
          DumpPacket((uint8_t*)(tx_packet), 60);
        }
      }
      //__threadfence_system();
    BEGIN_SINGLE_THREAD_PART {
      tx_tail_for_queue_zero = io_addr + IXGBE_TDT(0);
    } END_SINGLE_THREAD_PART;
#endif
  }
}

__global__ void rx_handler(volatile unsigned char* d_pkt_buffer, int * tb_alloc_tbl, char *bm_worked_thread, volatile int* num_turns, int fd, unsigned char* d_pkt_processing_queue) // bm: bitmap
{
  *num_turns = 0;
  //volatile unsigned char * d_mem = (volatile unsigned char*)d_pkt_buffer;
  tb_alloc_tbl[1] = 0;
  volatile unsigned char* rx_buf = d_pkt_buffer + offset_for_rx;

  if(blockIdx.x == 0) {
    BEGIN_SINGLE_THREAD_PART {
      printf("Entering rx_handler. (Block ID:%d)\n", blockIdx.x);
      int mem_index = 0; // why 12??
      while(*num_turns < NUM_TURN) { 
        if(readNoCache(((uint16_t*)&rx_buf[mem_index])) != 0 ) {
          //for(int i=0; i<120; i=i+2) {
          //  printf("%d:0x%02x%02x\n", i, rx_buf[mem_index+i],rx_buf[mem_index+i+1]);
          //}
          //printf("1[%2d,%2d] %d, %d\n", blockIdx.x, threadIdx.x, readNoCache((uint16_t*)&rx_buf[mem_index]), mem_index);
          //printf("Setting tb_alloc_tbl[1] = 1 %d(%d)\n", (uint16_t)rx_buf[mem_index], mem_index);
          tb_alloc_tbl[1] = 1;
        }
        mem_index += 0x1000;
        //__threadfence_system();

        if(mem_index >= offset_for_rx) {
          mem_index -= offset_for_rx;
        }
      }
    } END_SINGLE_THREAD_PART;
  } else {
    while(*num_turns < NUM_TURN) {
      while(!readNoCache(tb_alloc_tbl + 1)) { } 
      //__threadfence_system();
      int mem_index = 0x1000 * threadIdx.x;
      if(readNoCache((uint16_t*)&rx_buf[mem_index]) != 0) {
        //printf("3[%2d,%2d] %d from %d\n", blockIdx.x, threadIdx.x, tb_alloc_tbl[blockIdx.x], *num_turns);
        printf("RX [%2d,%2d] %d\n", blockIdx.x, threadIdx.x, mem_index);
        //DumpPacket((uint8_t*)&rx_buf[mem_index], 60);
        memcpy(d_pkt_processing_queue + mem_index,(const void*)(rx_buf + mem_index), 0x1000);
        for(int i=mem_index; i<mem_index+0x1000; i++)
          rx_buf[i] = 0;
        bm_worked_thread[threadIdx.x] = 1;
      }
#if 1
      BEGIN_SINGLE_THREAD_PART {
        //printf("[%2d,%2d] %d from %d\n", blockIdx.x, threadIdx.x, tb_alloc_tbl[blockIdx.x], *num_turns);
        //__threadfence_system();
        tb_alloc_tbl[1] = 0; // set for rx
        tb_alloc_tbl[2] = 1; // set for pp
        (*num_turns)++;
        int num_worked_threads = 0;
        for(int i=0; i<NUM_THREADS; i++) {
          if(bm_worked_thread[i]) {
            num_worked_threads++;
            bm_worked_thread[i] = 0;
          }
        }
        printf("RX [%2d,%2d] %d from %d, num_worked_threads:%d\n", blockIdx.x, threadIdx.x, tb_alloc_tbl[blockIdx.x], *num_turns, num_worked_threads);
      } END_SINGLE_THREAD_PART;
#endif
    }
  }
}




__global__ void doorbell_test(void * io_addr, void * desc, uint32_t curr, int* d_mem, int size)
{
  printf("[%s][%d]\n", __FUNCTION__, __LINE__);
  if (blockIdx.x == 0) {
    BEGIN_SINGLE_THREAD_PART {
      printf("[%s][%d] in doorbell_test First Block.\n", __FUNCTION__, __LINE__);
      if(desc == 0) {
        printf("[%s][%d]desc==NULL.\n", __FUNCTION__, __LINE__);
        return;
      }
      // ARP call
      uint8_t* pktBuf;
      pktBuf = (uint8_t *)malloc(60);

      // For now, static ip address 
      uint8_t src_tmp[] = {0x0a, 0x00, 0x00, 0x02};
      uint8_t dst_tmp[] = {0x0a, 0x00, 0x00, 0x01};
      uint32_t src_ip;                             
      memcpy(&src_ip, src_tmp, 4);                 
      uint32_t dst_ip;                             
      memcpy(&dst_ip, dst_tmp, 4);  
      unsigned char dst_haddr[ETH_ALEN];
      memset(dst_haddr, 0xFF, ETH_ALEN);

      for(int i=0; i< size/sizeof(int); i++) {
        d_mem[i] = 0;
      }

      for(int i=0; i< 512; i++) {
        memcpy(d_mem+4096*i/4, pktBuf, 60);
      }

      unsigned char *db[12];
      for(int i=0; i<12; i++)
        db[i] = ((unsigned char *)io_addr) + IXGBE_TDT(i);

      COMPILER_BARRIER();
      volatile union ixgbe_adv_tx_desc* tx_desc;
      tx_desc = (union ixgbe_adv_tx_desc*) desc;

      int num_packets = NUM_PACKETS;
      int index = curr;
      for(int i=0; i<curr; i++) {
        tx_desc++;
      }
      for(int i=0; i<num_packets; i++) {
        tx_desc->read.cmd_type_len |= 60;
        tx_desc->read.olinfo_status = 0xf0000;
        tx_desc++;
        index++;
        if(index == 512)
          tx_desc = (union ixgbe_adv_tx_desc*) desc;
      }
      tail_val = (unsigned long)((curr + num_packets) % 512);
      *(volatile unsigned long*)db[0] = tail_val;

      COMPILER_BARRIER();

      //__threadfence_system();
    } END_SINGLE_THREAD_PART;
    
  } else {
#ifndef __USE_PKT_MONITOR__
    // Second Block
    //printf("[%s][%d] in doorbell_test Second Block.\n", __FUNCTION__, __LINE__);
    //d_check_data(size, d_mem, flag);
#endif
  }
}

// YHOON~ for test
int tx_rx_ring_setup()
{
  const char *myinode = "/dev/ixgbe";
  int fd = open(myinode, O_RDWR);
  ioctl(fd, 1);
  return fd;
}

void yhoon_initializer(int fd, void *ixgbe_bar0_host_addr, ixgbe_adv_tx_desc* desc_addr, void **io_addr, void **tx_desc)
{
  const size_t IXGBE_BAR0_SIZE = 4096*8; // A rough calculation

  ixgbe_bar0_host_addr = mmap(0, IXGBE_BAR0_SIZE*5 , PROT_READ | PROT_WRITE, MAP_SHARED, fd, 0);
  ASSERTRT(hipHostRegister(ixgbe_bar0_host_addr, IXGBE_BAR0_SIZE, hipHostRegisterIoMemory));
  ASSERTRT(hipHostGetDevicePointer((void**)io_addr, (void*)ixgbe_bar0_host_addr, 0));
  printf("io_addr: %p\n", *io_addr);

  desc_addr = (ixgbe_adv_tx_desc*)((unsigned char*)ixgbe_bar0_host_addr + IXGBE_BAR0_SIZE);
  ASSERTRT(hipHostRegister(desc_addr, IXGBE_BAR0_SIZE * 4, hipHostRegisterIoMemory));
  //ASSERTRT(hipMalloc(&tx_desc, IXGBE_BAR0_SIZE * 4));
  ASSERTRT(hipHostGetDevicePointer((void**)tx_desc, (void*)desc_addr, 0));
  printf("tx_desc: %p\n", *tx_desc);

}

void yhoon_finalizer(void* ixgbe_bar0_host_addr, ixgbe_adv_tx_desc* desc_addr)
{
  const size_t IXGBE_BAR0_SIZE = 4096*8; // A rough calculation
  hipHostUnregister(desc_addr);
  hipHostUnregister(ixgbe_bar0_host_addr);
  munmap(ixgbe_bar0_host_addr, IXGBE_BAR0_SIZE*5);
}


void yhoon_xmit_arp(int *d_mem, int size, int fd)
{
  printf("[%s][%d]START---yhoon_xmit_arp------------HOST.\n", __FUNCTION__, __LINE__);
  printf("[%s][%d] sizeof tx_desc:%lu \n", __FUNCTION__, __LINE__, sizeof(union ixgbe_adv_tx_desc));
  // sample code for copying address to drivers
  //uint64_t ptr = 1234;
  //ioctl(fd, 0, &ptr);

  //printf("cpu_to_le32 test:%x\n", htonl(60));

  void* dBAR;
  const size_t IXGBE_BAR0_SIZE = 4096*8; // A rough calculation
  void* ixgbe_bar0_host_addr = mmap(0, IXGBE_BAR0_SIZE*5 , PROT_READ | PROT_WRITE, MAP_SHARED, fd, 0);
  ASSERTRT(hipHostRegister(ixgbe_bar0_host_addr, IXGBE_BAR0_SIZE, hipHostRegisterIoMemory));
  ASSERTRT(hipHostGetDevicePointer((void**)&dBAR, (void*)ixgbe_bar0_host_addr, 0));

  unsigned char *db[12];
  for(int i=0; i<1; i++) {
    db[i] = ((unsigned char *)ixgbe_bar0_host_addr) + IXGBE_TDT(i);
#if 0
    printf("db[%d]: %d\n", i, *(volatile unsigned int *)db[i]);
#endif
  }

  ixgbe_adv_tx_desc* desc = (ixgbe_adv_tx_desc*)((unsigned char*)ixgbe_bar0_host_addr + IXGBE_BAR0_SIZE);
  ASSERTRT(hipHostRegister(desc, IXGBE_BAR0_SIZE * 4, hipHostRegisterIoMemory));
  //hipHostRegister(desc, sizeof(ixgbe_adv_tx_desc), hipHostRegisterIoMemory);
  void* tx_desc;
  ASSERTRT(hipMalloc(&tx_desc, IXGBE_BAR0_SIZE * 4));
  if(hipSuccess != hipHostGetDevicePointer((void**)&tx_desc, (void*)desc, 0)) {
    cudaCheckErrors("hipHostGetDevicePointer fails");
  }

	hipStream_t cuda_stream2;
  ASSERT_CUDA(hipStreamCreate(&cuda_stream2));

  uint32_t curr_tx_index_q_zero = *(volatile unsigned int *)db[0];
  printf("curr_tx_index_q_zero: %u\n", curr_tx_index_q_zero);
  doorbell_test<<< 1, 1, 0, cuda_stream2 >>>(dBAR, tx_desc, curr_tx_index_q_zero, d_mem, size);

  //if(hipSuccess != hipDeviceSynchronize())
	//  cudaCheckErrors("doorbell_sync_error!");

  curr_num = curr_tx_index_q_zero;
  hipHostUnregister(desc);
  hipHostUnregister(ixgbe_bar0_host_addr);
  munmap(ixgbe_bar0_host_addr, IXGBE_BAR0_SIZE*5);

  // ~YHOON
  printf("[%s][%d]END-----yhoon_xmit_arp------------HOST.\n", __FUNCTION__, __LINE__);
}

void check_data(int size, unsigned char* h_mem, int* d_A)
{
  printf("[%s][%d] BEGINS--------------------------------------------------------------------------------\n", __FUNCTION__, __LINE__);
  hipMemcpy(h_mem, d_A, size, hipMemcpyDeviceToHost);
#if 0
  for(int outer = 0; outer < size-4095; outer+=4096) {
	if(iter[outer+4095] != 0) {
      int inner = outer;
      while(iter[inner] != 0)
	    printf("%c", iter[inner++]);
	}
  }
#else
  int dirty_cnt = 0;
  uint32_t offset_for_rx = 512 * 4096;
  //const int DUMP_SIZE = 30;
  for(int i = offset_for_rx; i < size; i++) {
    if(*((uint16_t*)&h_mem[i]) == 0x0608 || *((uint16_t*)&h_mem[i]) == 0x0008 ) {//(h_mem[i] == 0x08 && h_mem[i+1] == 0x00) ) {
      i = i;
      //printf("\n%s][%d] PACKET BEGINS-------------------------------------\n", __FUNCTION__, __LINE__);
      printf("[%s][%d] We found ethernet type 0x%02x%02x on %dth memory.\n", __FUNCTION__ , __LINE__, h_mem[i], h_mem[i+1],i-offset_for_rx);
      printf("[%s][%d] We found ethernet type 0x%04x on %dth memory.\n", __FUNCTION__ , __LINE__, *((uint16_t*)&h_mem[i]), i-offset_for_rx);
    } else if (h_mem[i] == 0xAB || h_mem[i] == 0xBA) {
      for(int packet_iter = i; packet_iter < (i+20); packet_iter++) {
        if((packet_iter-i) % 4 == 0)
          printf("\n");
        printf("%3d:0x%02x\t", packet_iter-i, h_mem[packet_iter]);
      }
	}
  }
  for(int i = 0; i < size; i++) 
    if(h_mem[i] != 0) 
      dirty_cnt++;
#endif
  printf("[%s][%d] dirty_cnt:[%d]\n", __FUNCTION__ , __LINE__, dirty_cnt);
  printf("[%s][%d] ENDS\n\n\n\n", __FUNCTION__, __LINE__);
}

int main(int argc, char *argv[])
{
  printf("[%s][%d] main\n", __FUNCTION__, __LINE__);
  int dev_id = 0;
  size_t _pkt_buffer_size = 2*512*4096; // 4MB, for rx,tx ring

  // CKJUNG, meaning of this?
  size_t pkt_buffer_size = (_pkt_buffer_size + GPU_PAGE_SIZE - 1) & GPU_PAGE_MASK;
  printf("[%s][%d]____CKJUNG__pkt_buffer_size: %lu\n", __FUNCTION__, __LINE__, pkt_buffer_size);

  int n_devices = 0;

  ASSERTRT(hipGetDeviceCount(&n_devices));

  hipDeviceProp_t prop;
  for (int n=0; n<n_devices; ++n) {
    hipGetDeviceProperties(&prop,n);
    OUT << "GPU id:" << n << " name:" << prop.name 
      << " PCI domain: " << prop.pciDomainID 
      << " bus: " << prop.pciBusID 
      << " device: " << prop.pciDeviceID << endl;
  }
  OUT << "selecting device " << dev_id << endl;
  OUT << "_pkt_buffer_size: " << _pkt_buffer_size << "  pkt_buffer_size: " << pkt_buffer_size << endl;

  ASSERTRT(hipSetDevice(dev_id));
  ASSERTRT(hipSetDeviceFlags(hipDeviceMapHost));

  unsigned char* d_pkt_buffer;
  unsigned char* d_pkt_processing_queue;
  ASSERTRT(hipMalloc((void**)&d_pkt_buffer, pkt_buffer_size));
  ASSERTRT(hipMalloc((void**)&d_pkt_processing_queue, 4*pkt_buffer_size));
  ASSERTRT(hipMemset(d_pkt_buffer, 0, pkt_buffer_size));
  ASSERTRT(hipMemset(d_pkt_processing_queue, 0, 4*pkt_buffer_size));
 

  
  unsigned int flag = 1;
  ASSERTDRV(hipPointerSetAttribute(&flag, HIP_POINTER_ATTRIBUTE_SYNC_MEMOPS, (hipDeviceptr_t) d_pkt_buffer));


  my_t g = my_open();

  ASSERT_NEQ(g, (void*)0);

  my_mh_t mh;
  if (my_pin_buffer(g, (hipDeviceptr_t)d_pkt_buffer, pkt_buffer_size, 0, 0, &mh)  != 0)
    OUT << "NOT_EQ" << endl;

  int *h_tmp = (int*) malloc(sizeof(int) * pkt_buffer_size);
  for(int i=0; i<pkt_buffer_size; i++)
    h_tmp[i] = 0;

  hipMemcpy((void *)d_pkt_buffer, (const void*)h_tmp, pkt_buffer_size * sizeof(int), hipMemcpyHostToDevice);

  for(int i=512*4096; i<512*4096*2; i++) {
    if(h_tmp[i] != 0) {
      printf("I%d %c\n", i, (uint16_t)h_tmp[i]);
    }
  }

  hipMemcpy((void*)h_tmp, (const void *)d_pkt_buffer, pkt_buffer_size * sizeof(int), hipMemcpyDeviceToHost);

  for(int i=512*4096; i<512*4096*2; i++) {
    if(h_tmp[i] != 0) {
      printf("O%d %c\n", i, (uint16_t)h_tmp[i]);
    }
  }

  void *ixgbe_bar0_host_addr = 0, *io_addr=0, *tx_desc=0;
  ixgbe_adv_tx_desc* desc_addr=0;
  int fd = tx_rx_ring_setup();
  yhoon_initializer(fd, ixgbe_bar0_host_addr, desc_addr, &io_addr, &tx_desc);

	hipStream_t cuda_stream1;
  ASSERT_CUDA(hipStreamCreate(&cuda_stream1));
	hipStream_t cuda_stream2;
  ASSERT_CUDA(hipStreamCreate(&cuda_stream2));
	hipStream_t cuda_stream3;
  ASSERT_CUDA(hipStreamCreate(&cuda_stream3));

	int *dev_tb_alloc_tbl, *num_turns;
  char *bm_worked_thread;
	ASSERT_CUDA(hipMalloc(&dev_tb_alloc_tbl, NUM_THREADS * sizeof(*dev_tb_alloc_tbl)));
	ASSERT_CUDA(hipMemset(dev_tb_alloc_tbl, 0, NUM_THREADS* sizeof(*dev_tb_alloc_tbl)));
  ASSERTRT(hipMalloc((void**)&num_turns, sizeof(int)));
  ASSERTRT(hipMalloc((void**)&bm_worked_thread, NUM_THREADS * sizeof(char)));

  clean_buffer<<< 1, 1 >>> (d_pkt_buffer, d_pkt_processing_queue, pkt_buffer_size, bm_worked_thread);

  if(hipSuccess != hipDeviceSynchronize())
	  cudaCheckErrors("hipDeviceSynchronize Error"); 

#if 1
  rx_handler<<< NUM_TB, NUM_THREADS, 0, cuda_stream1 >>> (d_pkt_buffer, dev_tb_alloc_tbl, bm_worked_thread, num_turns, fd, d_pkt_processing_queue);
  packet_processor<<< NUM_TB, NUM_THREADS, 0, cuda_stream2 >>> (d_pkt_processing_queue, d_pkt_buffer, dev_tb_alloc_tbl, num_turns, (volatile uint8_t *)io_addr);
  tx_handler<<< NUM_TB, NUM_THREADS, 0, cuda_stream3 >>> (d_pkt_buffer, dev_tb_alloc_tbl, (volatile uint8_t*)io_addr, (volatile union ixgbe_adv_tx_desc*) tx_desc, num_turns);
#endif

  // call ixgbe_xmit_yhoon in ixgbe_main.c
  //yhoon_xmit_arp(d_pkt_buffer, pkt_buffer_size, fd);

#if 0
  int one = 1;
  usleep(1*1000*1000);
  hipMemcpyToSymbol(HIP_SYMBOL(server_done), &one, sizeof(int));

  unsigned char* h_mem = (unsigned char*)malloc(pkt_buffer_size*sizeof(unsigned char)); 
  hipMemcpy(d_pkt_buffer, h_mem, pkt_buffer_size, hipMemcpyHostToDevice);
  int count = 0;
  while(count < 1000) {
    check_data(pkt_buffer_size, h_mem, d_pkt_buffer);
    usleep(1*1000*1000);
    count++;
  }
#endif

 
  if(hipSuccess != hipDeviceSynchronize())
	  cudaCheckErrors("hipDeviceSynchronize Error"); 

  yhoon_finalizer(ixgbe_bar0_host_addr, desc_addr);

  ASSERT_CUDA(hipFree(dev_tb_alloc_tbl));
  ASSERT_CUDA(hipFree(d_pkt_buffer));
  return 0;
}
