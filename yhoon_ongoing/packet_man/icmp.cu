#include <stdint.h>
#include <sys/types.h>
#include <netinet/ip.h>
#include <stdio.h>

#include "icmp.cu.h"

#define IP_NEXT_PTR(iph) ((uint8_t *)iph + (iph->ihl << 2))

#ifndef TRUE
#define TRUE (1)
#endif

#ifndef FALSE
#define FALSE (0)
#endif

#ifndef ERROR
#define ERROR (-1)
#endif



/*----------------------------------------------------------------------------*/

/*----------------------------------------------------------------------------*/

#undef IP_NEXT_PTR
