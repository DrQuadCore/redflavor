#include <stdint.h>
#include <sys/types.h>
#include <netinet/ip.h>
#include <stdio.h>

#include "icmp.h"

#define IP_NEXT_PTR(iph) ((uint8_t *)iph + (iph->ihl << 2))

#ifndef TRUE
#define TRUE (1)
#endif

#ifndef FALSE
#define FALSE (0)
#endif

#ifndef ERROR
#define ERROR (-1)
#endif



/*----------------------------------------------------------------------------*/
__device__ void 
DumpICMPPacket(struct icmphdr *icmph, uint32_t saddr, uint32_t daddr)
{
	uint8_t *t;
	
	printf("ICMP header: \n");
	printf("Type: %d, "
		"Code: %d, ID: %d, Sequence: %d\n", 
		icmph->icmp_type, icmph->icmp_code,
		NTOHS(ICMP_ECHO_GET_ID(icmph)), NTOHS(ICMP_ECHO_GET_SEQ(icmph)));
	
	t = (uint8_t *)&saddr;
	printf("Sender IP: %u.%u.%u.%u\n",
		t[0], t[1], t[2], t[3]);
	
	t = (uint8_t *)&daddr;
	printf("Target IP: %u.%u.%u.%u\n",
		t[0], t[1], t[2], t[3]);
}
/*----------------------------------------------------------------------------*/

__device__ int 
ProcessICMPPacket(struct iphdr *iph, int len)
{
  printf("[%s][%d]\n",__FUNCTION__, __LINE__);
#if 1
	struct icmphdr *icmph = (struct icmphdr *) IP_NEXT_PTR(iph);
	int i;
  // TODO : should we do the following?
#if 0
	int to_me = -1;
	
	/* process the icmp messages destined to me */
	for (i = 0; i < CONFIG.eths_num; i++) {
		if (iph->daddr == CONFIG.eths[i].ip_addr) {
			to_me = TRUE;
		}
	}
	
	if (!to_me)
		return TRUE;
#endif
	
	switch (icmph->icmp_type) {
        case ICMP_ECHO:
          printf("[%s][%d] [INFO] ICMP_ECHO received\n", __FUNCTION__, __LINE__);
          //ProcessICMPECHORequest(iph, len);
          break;
		
        case ICMP_DEST_UNREACH:
          printf("[INFO] ICMP Destination Unreachable message received\n");
          break;
		
        case ICMP_TIME_EXCEEDED:
          printf("[INFO] ICMP Time Exceeded message received\n");
          break;

        default:
          printf("[INFO] Unsupported ICMP message type %x received\n", icmph->icmp_type);
          break;
  }
#endif
  return TRUE;
}


#undef IP_NEXT_PTR
