#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <iostream>
#include <memory.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <stdarg.h>
#include <string.h>
#include <sys/types.h>
#include <sys/socket.h>
#include <sys/time.h>
#include <sys/stat.h>
#include <sys/mman.h>
#include <fcntl.h>
#include <errno.h>
#include <netdb.h>
#include <malloc.h>
#include <getopt.h>
#include <arpa/inet.h>
#include <sys/ioctl.h>
#include <time.h>
#include <asm/types.h>

#include <linux/if_ether.h>
#include <linux/ip.h>
#include <linux/udp.h>
#include "arp.h" 

#include <linux/netdevice.h>   /* struct device, and other headers */
#include <linux/tcp.h>         /* struct tcphdr */
#include <linux/in6.h>
#define ETH_ALEN  6 // YHOON
#define ARP_PAD_LEN 18 // YHOON

#define HTONS(n) (((((unsigned short)(n) & 0xFF)) << 8) | (((unsigned short)(n) & 0xFF00) >> 8))
#define NTOHS(n) (((((unsigned short)(n) & 0xFF)) << 8) | (((unsigned short)(n) & 0xFF00) >> 8))

#define HTONL(n) (((((unsigned long)(n) & 0xFF)) << 24) | \
	((((unsigned long)(n) & 0xFF00)) << 8) | \
	((((unsigned long)(n) & 0xFF0000)) >> 8) | \
		  ((((unsigned long)(n) & 0xFF000000)) >> 24))

#define NTOHL(n) (((((unsigned long)(n) & 0xFF)) << 24) | \
	((((unsigned long)(n) & 0xFF00)) << 8) | \
	((((unsigned long)(n) & 0xFF0000)) >> 8) | \
		  ((((unsigned long)(n) & 0xFF000000)) >> 24))



#include <asm/types.h>
using namespace std;

#include "packet_man.h"
#include "mydrv/mydrv.h"
#include "common.hpp"


#define OUT cout


#define HTONS(n) (((((unsigned short)(n) & 0xFF)) << 8) | (((unsigned short)(n) & 0xFF00) >> 8))
#define NTOHS(n) (((((unsigned short)(n) & 0xFF)) << 8) | (((unsigned short)(n) & 0xFF00) >> 8))
enum mycopy_msg_level {
    MYCOPY_MSG_DEBUG = 1,
    MYCOPY_MSG_INFO,
    MYCOPY_MSG_WARN,
    MYCOPY_MSG_ERROR
};

#define IXGBE_TDT(_i)   (0x06018 + ((_i) * 0x40))

struct my {
  int fd;
};

/*----------------------------------------------------------------------------*/
enum arp_hrd_format
{
	arp_hrd_ethernet = 1
};
/*----------------------------------------------------------------------------*/
enum arp_opcode
{
	arp_op_request = 1, 
	arp_op_reply = 2, 
};

// YHOON
struct arphdr
{
  uint16_t ar_hrd;      /* hardware address format */
  uint16_t ar_pro;      /* protocol address format */
  uint8_t ar_hln;       /* hardware address length */
  uint8_t ar_pln;       /* protocol address length */
  uint16_t ar_op;       /* arp opcode */

  uint8_t ar_sha[ETH_ALEN]; /* sender hardware address */
  uint32_t ar_sip;      /* sender ip address */
  uint8_t ar_tha[ETH_ALEN]; /* targe hardware address */
  uint32_t ar_tip;      /* target ip address */

  uint8_t pad[ARP_PAD_LEN];
} __attribute__ ((packed));

static int my_msg_level = MYCOPY_MSG_ERROR;
static int my_enable_logging = 1;

static void my_msg(enum mycopy_msg_level lvl, const char* fmt, ...)
{
    if (-1 == my_enable_logging) {
        const char *env = getenv("MYCOPY_ENABLE_LOGGING");
        if (env)
            my_enable_logging = 1;
        else
            my_enable_logging = 0;

        env = getenv("MYCOPY_LOG_LEVEL");
        if (env)
            my_msg_level = atoi(env);
    }
    if (my_enable_logging) {
        if (lvl >= my_msg_level) {
            va_list ap;
            va_start(ap, fmt);
            vfprintf(stderr, fmt, ap);
        }
    }
}

#define my_dbg(FMT, ARGS...)  my_msg(MYCOPY_MSG_DEBUG, "DBG:  " FMT, ## ARGS)
#define my_dbgc(C, FMT, ARGS...)  do { static int my_dbg_cnt=(C); if (my_dbg_cnt) { my_dbg(FMT, ## ARGS); --my_dbg_cnt; }} while (0)
#define my_info(FMT, ARGS...) my_msg(MYCOPY_MSG_INFO,  "INFO: " FMT, ## ARGS)
#define my_warn(FMT, ARGS...) my_msg(MYCOPY_MSG_WARN,  "WARN: " FMT, ## ARGS)
#define my_err(FMT, ARGS...)  my_msg(MYCOPY_MSG_ERROR, "ERR:  " FMT, ## ARGS)

#define DBGMSG 1

void 
DumpARPPacket(struct arphdr *arph)
{
	uint8_t *t;

	printf("ARP header: \n");
	printf("Hardware type: %d (len: %d), "
			"protocol type: %d (len: %d), opcode: %d\n", 
			ntohs(arph->ar_hrd), arph->ar_hln, 
			ntohs(arph->ar_pro), arph->ar_pln, ntohs(arph->ar_op));
	t = (uint8_t *)&arph->ar_sip;
	printf("Sender IP: %u.%u.%u.%u, "
			"haddr: %02X:%02X:%02X:%02X:%02X:%02X\n", 
			t[0], t[1], t[2], t[3], 
			arph->ar_sha[0], arph->ar_sha[1], arph->ar_sha[2], 
			arph->ar_sha[3], arph->ar_sha[4], arph->ar_sha[5]);
	t = (uint8_t *)&arph->ar_tip;
	printf("Target IP: %u.%u.%u.%u, "
			"haddr: %02X:%02X:%02X:%02X:%02X:%02X\n", 
			t[0], t[1], t[2], t[3], 
			arph->ar_tha[0], arph->ar_tha[1], arph->ar_tha[2], 
			arph->ar_tha[3], arph->ar_tha[4], arph->ar_tha[5]);
}

void
DumpPacket(uint8_t *buf, int len)
{
  printf("\n<<<DumpPacket>>>\n");
	struct ethhdr *ethh;
	struct iphdr *iph;
	struct udphdr *udph;
	//struct tcphdr *tcph;
	uint8_t *t;

	ethh = (struct ethhdr *)buf;
	if (ntohs(ethh->h_proto) != ETH_P_IP) {
		printf("%02X:%02X:%02X:%02X:%02X:%02X -> %02X:%02X:%02X:%02X:%02X:%02X ",
				ethh->h_source[0],
				ethh->h_source[1],
				ethh->h_source[2],
				ethh->h_source[3],
				ethh->h_source[4],
				ethh->h_source[5],
				ethh->h_dest[0],
				ethh->h_dest[1],
				ethh->h_dest[2],
				ethh->h_dest[3],
				ethh->h_dest[4],
				ethh->h_dest[5]);

		printf("protocol %04hx  \n", ntohs(ethh->h_proto));

    if(ntohs(ethh->h_proto) == ETH_P_ARP)
      DumpARPPacket((struct arphdr *) (ethh + 1));
		goto done;
	}

	iph = (struct iphdr *)(ethh + 1);
	udph = (struct udphdr *)((uint32_t *)iph + iph->ihl);
	//tcph = (struct tcphdr *)((uint32_t *)iph + iph->ihl);

	t = (uint8_t *)&iph->saddr;
	printf("%u.%u.%u.%u", t[0], t[1], t[2], t[3]);
	if (iph->protocol == IPPROTO_TCP || iph->protocol == IPPROTO_UDP)
		printf("(%d)", ntohs(udph->source));

	printf(" -> ");

	t = (uint8_t *)&iph->daddr;
	printf("%u.%u.%u.%u", t[0], t[1], t[2], t[3]);
	if (iph->protocol == IPPROTO_TCP || iph->protocol == IPPROTO_UDP)
		printf("(%d)", ntohs(udph->dest));

	printf(" IP_ID=%d", ntohs(iph->id));
	printf(" TTL=%d ", iph->ttl);

	switch (iph->protocol) {
	case IPPROTO_TCP:
		printf("TCP ");
		break;
	case IPPROTO_UDP:
		printf("UDP ");
		break;
	default:
		printf("protocol %d ", iph->protocol);
		goto done;
	}
done:
	printf("len=%d\n", len);
}

uint8_t *
EthernetOutput(uint8_t *buf, uint16_t h_proto, 
		unsigned char* src_haddr, unsigned char* dst_haddr, uint16_t iplen)
{
	struct ethhdr *ethh;
	int i;

	ethh = (struct ethhdr *)buf;

#if 1
	printf("dst_hwaddr: %02X:%02X:%02X:%02X:%02X:%02X\n",
				dst_haddr[0], dst_haddr[1], 
				dst_haddr[2], dst_haddr[3], 
				dst_haddr[4], dst_haddr[5]);
	printf("src_hwaddr: %02X:%02X:%02X:%02X:%02X:%02X\n",
				src_haddr[0], src_haddr[1], 
				src_haddr[2], src_haddr[3], 
				src_haddr[4], src_haddr[5]);
#endif

	for (i = 0; i < ETH_ALEN; i++) {
		ethh->h_source[i] = src_haddr[i];
		ethh->h_dest[i] = dst_haddr[i];
	}
	ethh->h_proto = HTONS(h_proto);

	return (uint8_t *)(ethh + 1);
}

static int 
ARPOutput(uint8_t * buf, int opcode,
		uint32_t src_ip, uint32_t dst_ip, unsigned char *dst_haddr)
{
	if (!dst_haddr)
		return -1;
  printf("ARPOUTPUT\n");
	/* Allocate a buffer */

#if 1
  uint8_t src_haddr[ETH_ALEN];
  src_haddr[0] = 0xa0;
  src_haddr[1] = 0x36;
  src_haddr[2] = 0x9f;
  src_haddr[3] = 0x9c;
  src_haddr[4] = 0x8c;
  src_haddr[5] = 0x26;

	struct arphdr *arph = (struct arphdr *)(uintptr_t)EthernetOutput(
    buf, ETH_P_ARP, src_haddr, dst_haddr, sizeof(struct arphdr));

	if (!arph) {
    printf("ERROR\n");
		return -1;
	}
#else 
	struct arphdr *arph = NULL;
#endif
	/* Fill arp header */
	arph->ar_hrd = HTONS(arp_hrd_ethernet);
	arph->ar_pro = HTONS(ETH_P_IP);
	//arph->ar_pro = htons(0x0800);
	arph->ar_hln = ETH_ALEN;
	arph->ar_pln = 4;
	arph->ar_op = HTONS(opcode);

	/* Fill arp body */
#if 0 // HONESTCHOI : TODO
	arph->ar_sip = CONFIG.eths[nif].ip_addr;
#endif 
	arph->ar_sip = src_ip;
	arph->ar_tip = dst_ip;

#if 0 // HONESTCHOI : TODO
	memcpy(arph->ar_sha, CONFIG.eths[nif].haddr, arph->ar_hln);
	if (target_haddr) {
		memcpy(arph->ar_tha, target_haddr, arph->ar_hln);
	} else {
		memcpy(arph->ar_tha, dst_haddr, arph->ar_hln);
	}
#endif
	memcpy(arph->ar_sha, src_haddr, arph->ar_hln);
  memcpy(arph->ar_tha, dst_haddr, arph->ar_hln);
	memset(arph->pad, 0, ARP_PAD_LEN);

#if DBGMSG
	DumpARPPacket(arph);
#endif

	return 0;
}
int my_pin_buffer(my_t g, unsigned long addr, size_t size, uint64_t p2p_token, uint32_t va_space, my_mh_t *handle)
{
    int ret = 0;
    int retcode;

    struct MYDRV_IOC_PIN_BUFFER_PARAMS params;
    params.addr = addr;
    params.size = size;
    params.p2p_token = p2p_token;
    params.va_space = va_space;
    params.handle = 0;

    retcode = ioctl(g->fd, MYDRV_IOC_PIN_BUFFER, &params);
    if (0 != retcode) {
        ret = errno;
        my_err("ioctl error (errno=%d)\n", ret);
    }
    *handle = params.handle;

    return ret;
}

my_t my_open()
{
    my_t m = NULL;
    const char *myinode = "/dev/mydrv";

    m = (my_t) calloc(1, sizeof(*m));
    if (!m) {
        //my_err("error while allocating memory\n");
        return NULL;
    }

    int fd = open(myinode, O_RDWR);
    if (-1 == fd ) {
        int ret = errno;
        //my_err("error opening driver (errno=%d/%s)\n", ret, strerror(ret));
        free(m);
        return NULL;
    }

    m->fd = fd;

    return m;
}
#if 0
static inline uint32_t myrand(uint64_t *seed) 
{
	*seed = *seed * 1103515245 + 12345;
	return (uint32_t)(*seed >> 32);
}
#endif

#if 0 // BUILD_PACKET
void build_packet(char *buf, int size, uint64_t *seed)
{
	struct ethhdr *eth;
	struct iphdr *ip;
	struct udphdr *udp;

	uint32_t rand_val;

	//memset(buf, 0, size);

	/* build an ethernet header */
	eth = (struct ethhdr *)buf;
  /*
  char arp_req[] = {0xff, 0xff, 0xff, 0xff, 0xff, 0xff, 
                    0xab, 0xcd, 0xef, 0x01, 0x23, 0x45,
                    //0xa0, 0x36, 0x9f, 0x9c, 0x8c, 0x26,
                    0x06, 0x08, 
                    0x00, 0x01, 
                    0x08, 0x00, 
                    0x06, 
                    0x04, 
                    0x00, 0x01,  // op code : 1 for req
                    0xa0, 0x36, 0x9f, 0x9c, 0x8c, 0x26,
                    0x01, 0x01, 0x01, 0x0b, 
                    0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 
                    0x01, 0x01, 0x01, 0x15,
                    0xab, 0xcd, 0xef, 0x01, 0x23, 0x45};
                    */


	eth->h_dest[0] = 0xa0;
	eth->h_dest[1] = 0x36;
	eth->h_dest[2] = 0x9f;
	eth->h_dest[3] = 0x9c;
	eth->h_dest[4] = 0x93;
	eth->h_dest[5] = 0x06;

	eth->h_source[0] = 0xab;
	eth->h_source[1] = 0xcd;
	eth->h_source[2] = 0xef;
	eth->h_source[3] = 0x01;
	eth->h_source[4] = 0x23;
	eth->h_source[5] = 0x45;

	eth->h_proto = HTONS(0x0800);

	/* build an IP header */
	ip = (struct iphdr *)(buf + sizeof(*eth));

	ip->version = 4;
	ip->ihl = 5;
	ip->tos = 0;
	ip->tot_len = HTONS(size - sizeof(*eth));
	ip->id = 0;
	ip->frag_off = 0;
	ip->ttl = 32;
	ip->protocol = IPPROTO_UDP;
	ip->saddr = HTONL(0x0101010b);
	ip->daddr = HTONL(0x01010115);
	//ip->daddr = HTONL(myrand(seed));
	ip->check = 0;
	//ip->check = ip_fast_csum(ip, ip->ihl);
#if 1
	udp = (struct udphdr *)((char *)ip + sizeof(*ip));

	rand_val = myrand(seed);
	udp->source = HTONS(rand_val & 0xFFFF);
	udp->dest = HTONS((rand_val >> 16) & 0xFFFF);

	udp->len = HTONS(size - sizeof(*eth) - sizeof(*ip));
	udp->check = 0;
#endif
}
#endif // BUILD_PACKET

#if 0
__global__ void init_data(int size, unsigned char* h_mem, int* d_A)
#else
void init_data(int size, unsigned char* h_mem, int* d_A)
#endif
{
  //printf("[%s][%d]\n", __FUNCTION__, __LINE__);
  for(int i=0; i < size; i++) 
    h_mem[i] = 0;
/* for router
  char arp_req[] = {0xff, 0xff, 0xff, 0xff, 0xff, 0xff, 
                    0xa0, 0x36, 0x9f, 0x9c, 0x8c, 0x26,
                    0x08, 0x06, 
                    0x00, 0x01, 
                    0x08, 0x00, 
                    0x06, 
                    0x04, 
                    0x00, 0x01,  // op code : 1 for req
                    0xa0, 0x36, 0x9f, 0x9c, 0x8c, 0x26,
                    0x01, 0x01, 0x01, 0x0b, 
                    0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 
                    0x01, 0x01, 0x01, 0x01};

                    */

  // for snow
  /*
  char arp_req[] = {0xff, 0xff, 0xff, 0xff, 0xff, 0xff, 
                    0xab, 0xcd, 0xef, 0x01, 0x23, 0x45,
                    //0xa0, 0x36, 0x9f, 0x9c, 0x8c, 0x26,
                    0x06, 0x08, 
                    0x00, 0x01, 
                    0x08, 0x00, 
                    0x06, 
                    0x04, 
                    0x00, 0x01,  // op code : 1 for req
                    0xa0, 0x36, 0x9f, 0x9c, 0x8c, 0x26,
                    0x01, 0x01, 0x01, 0x0b, 
                    0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 
                    0x01, 0x01, 0x01, 0x15
                    };
                    */

  char * arp_req = (char*) malloc(100*sizeof(char));
 
  //uint64_t seed = 0;
  //build_packet(arp_req, 1000, &seed);  

  uint8_t* buf;
  buf = (uint8_t *) malloc(60);
  uint8_t src_tmp[] = {0x01, 0x01, 0x01, 0x0b};
  uint8_t dst_tmp[] = {0x01, 0x01, 0x01, 0x01};
  uint32_t src_ip;
  memcpy(&src_ip, src_tmp, 4);
  uint32_t dst_ip;
  memcpy(&dst_ip, dst_tmp, 4);

	unsigned char dst_haddr[ETH_ALEN];
	memset(dst_haddr, 0xFF, ETH_ALEN);

  ARPOutput(buf, arp_op_request, src_ip, dst_ip, dst_haddr);

  DumpPacket(buf, 60);

  for(int i=0; i < 60; i++) 
    h_mem[i] = buf[i];
    //h_mem[i] = arp_req[i];

  hipMemcpy(d_A, h_mem, size, hipMemcpyHostToDevice);
}

#if 0
__global__ void check_data(int size, unsigned char* h_mem, int* d_A)
#else
void check_data(int size, unsigned char* h_mem, int* d_A)
#endif
{
  printf("[%s][%d] BEGINS--------------------------------------------------------------------------------\n", __FUNCTION__, __LINE__);
  hipMemcpy(h_mem, d_A, size, hipMemcpyDeviceToHost);
#if 0
  for(int outer = 0; outer < size-4095; outer+=4096) {
	if(iter[outer+4095] != 0) {
      int inner = outer;
      while(iter[inner] != 0)
	    printf("%c", iter[inner++]);
	}
  }
#else
  int dirty_cnt = 0;
  //const int DUMP_SIZE = 30;
  for(int i = 0; i < size; i++) {
    if(*((uint16_t*)&h_mem[i]) == 0x0608 || *((uint16_t*)&h_mem[i]) == 0x0008 ) {//(h_mem[i] == 0x08 && h_mem[i+1] == 0x00) ) {
      i = i;
      //printf("\n%s][%d] PACKET BEGINS-------------------------------------\n", __FUNCTION__, __LINE__);
      printf("[%s][%d] We found ethernet type 0x%02x%02x on %dth memory.\n", __FUNCTION__ , __LINE__, h_mem[i], h_mem[i+1],i);
      printf("[%s][%d] We found ethernet type 0x%04x on %dth memory.\n", __FUNCTION__ , __LINE__, *((uint16_t*)&h_mem[i]), i);
      
      //for(int packet_iter = i-20; packet_iter < (i+20+DUMP_SIZE); packet_iter++) {
      //  if((packet_iter-i) % 4 == 0)
      //    printf("\n");
      //  printf("%3d:0x%02x\t", packet_iter-i, h_mem[packet_iter]);
      //}
      DumpPacket((uint8_t*)&h_mem[i-12], 60);
      //printf("\n");
      //struct iphdr* iph = reinterpret_cast<iphdr*>(h_mem+i+2);
      //struct arphdr *arph = reinterpret_cast<arphdr*>(h_mem+i);
      //struct iphdr* iph = (struct iphdr *)(h_mem+i+2);
      //printf("[%s][%d] pid:[%x], protocol:[%x]\n", __FUNCTION__, __LINE__, iph->id, iph->protocol);
      //uint8_t *t = (uint8_t *)&iph->saddr;
      //uint8_t *saddr = (uint8_t *)&arph->ar_sip;
      //uint8_t *daddr = (uint8_t *)&arph->ar_tip;
      //printf("[%s][%d] src ip %u.%u.%u.%u\n", __FUNCTION__ ,__LINE__, t[0], t[1], t[2], t[3]);
      //printf("[%s][%d] src ip %x.%x.%x.%x\n", __FUNCTION__ ,__LINE__, t[0], t[1], t[2], t[3]);
      //printf("[%s][%d] src ip %u.%u.%u.%u\n", __FUNCTION__ ,__LINE__, saddr[0], saddr[1], saddr[2], saddr[3]);
      //printf("[%s][%d] dst ip %u.%u.%u.%u\n", __FUNCTION__ ,__LINE__, daddr[0], daddr[1], daddr[2], daddr[3]);
      //t = (uint8_t *)&iph->daddr;
      //printf("[%s][%d] dst ip %u.%u.%u.%u\n", __FUNCTION__ ,__LINE__, t[0], t[1], t[2], t[3]);
    } else if (h_mem[i] == 0xAB || h_mem[i] == 0xBA) {
      for(int packet_iter = i; packet_iter < (i+20); packet_iter++) {
        if((packet_iter-i) % 4 == 0)
          printf("\n");
        printf("%3d:0x%02x\t", packet_iter-i, h_mem[packet_iter]);
      }
	}
  }
  for(int i = 0; i < size; i++) 
    if(h_mem[i] != 0) 
      dirty_cnt++;
#endif
  printf("[%s][%d] dirty_cnt:[%d]\n", __FUNCTION__ , __LINE__, dirty_cnt);
  printf("[%s][%d] ENDS\n\n\n\n", __FUNCTION__, __LINE__);
}

#define COMPILER_BARRIER() asm volatile("" ::: "memory")
#ifndef __USE_GPU__
void doorbell_test(void * io_addr)
#else
__global__ void doorbell_test(void * io_addr)
#endif
{
  printf("[%s][%d] \n", __FUNCTION__, __LINE__);
  unsigned char *db0, *db1, *db2, *db3, *db4, *db5;
	db0 = ((unsigned char *)io_addr) + IXGBE_TDT(0);
	db1 = ((unsigned char *)io_addr) + IXGBE_TDT(1);
	db2 = ((unsigned char *)io_addr) + IXGBE_TDT(2);
	db3 = ((unsigned char *)io_addr) + IXGBE_TDT(3);
	db4 = ((unsigned char *)io_addr) + IXGBE_TDT(4);
	db5 = ((unsigned char *)io_addr) + IXGBE_TDT(5);


  printf("[%s][%d] %d\n", __FUNCTION__, __LINE__, *(volatile unsigned int *)db0 );
  printf("[%s][%d] %d\n", __FUNCTION__, __LINE__, *(volatile unsigned int *)db1 );
  printf("[%s][%d] %d\n", __FUNCTION__, __LINE__, *(volatile unsigned int *)db2 );
  printf("[%s][%d] %d\n", __FUNCTION__, __LINE__, *(volatile unsigned int *)db3 );
  printf("[%s][%d] %d\n", __FUNCTION__, __LINE__, *(volatile unsigned int *)db4 );
  printf("[%s][%d] %d\n", __FUNCTION__, __LINE__, *(volatile unsigned int *)db5 );

  COMPILER_BARRIER();
  *(volatile unsigned int *)db0 = 100;
  *(volatile unsigned int *)db1 = 100;
  *(volatile unsigned int *)db2 = 100;
  *(volatile unsigned int *)db3 = 100;
  *(volatile unsigned int *)db4 = 100;
  *(volatile unsigned int *)db5 = 100;
}

// YHOON~ for test
void yhoon_xmit_arp()
{
  const char *myinode = "/dev/ixgbe";
  int fd = open(myinode, O_RDWR);
  uint64_t ptr = 1234;
  //int retcode;
  //retcode = ioctl(fd, 0, &ptr);
  ioctl(fd, 0, &ptr);

#ifndef __USE_GPU__
  void* dummy2;
  ASSERTRT(hipMalloc(&dummy2, 4096*8));

  doorbell_test(dummy2);
#else
  void* dBAR;
  const size_t IXGBE_BAR0_SIZE = 4096*8; // A rough calculation
  void* ixgbe_bar0_host_addr = mmap(0, IXGBE_BAR0_SIZE , PROT_READ | PROT_WRITE, MAP_SHARED, fd, 0);
  hipHostRegister(ixgbe_bar0_host_addr, IXGBE_BAR0_SIZE, hipHostRegisterIoMemory);
  hipHostGetDevicePointer((void**)&dBAR, (void*)ixgbe_bar0_host_addr, 0);

  doorbell_test<<< 1,1 >>>(dBAR);
#endif
  // ~YHOON
}


int main(int argc, char *argv[])
{
  printf("[%s][%d] 2\n", __FUNCTION__, __LINE__);
  int dev_id = 0;
  size_t _size = 50*1024*1024; //50*1024*1024;

  size_t size = (_size + GPU_PAGE_SIZE - 1) & GPU_PAGE_MASK;

  int n_devices = 0;

  unsigned char* h_mem = (unsigned char*)malloc(size*sizeof(unsigned char)); 


  hipGetDeviceCount(&n_devices);

  hipDeviceProp_t prop;
  for (int n=0; n<n_devices; ++n) {
    hipGetDeviceProperties(&prop,n);
    OUT << "GPU id:" << n << " name:" << prop.name 
      << " PCI domain: " << prop.pciDomainID 
      << " bus: " << prop.pciBusID 
      << " device: " << prop.pciDeviceID << endl;
  }
  OUT << "selecting device " << dev_id << endl;
  OUT << "_size: " << _size << "  size: " << size << endl;

  ASSERTRT(hipSetDevice(dev_id));

  // Test
  void* dummy;
  ASSERTRT(hipMalloc(&dummy, 0));

  int* d_A;
  ASSERTRT(hipMalloc((void**)&d_A, size));
  OUT << "device ptr: " << hex << d_A << dec << endl;
  
  unsigned int flag = 1;
  ASSERTDRV(hipPointerSetAttribute(&flag, HIP_POINTER_ATTRIBUTE_SYNC_MEMOPS, (hipDeviceptr_t) d_A));

  my_t g = my_open();

  ASSERT_NEQ(g, (void*)0);

  my_mh_t mh;
  if (my_pin_buffer(g, (hipDeviceptr_t)d_A, size, 0, 0, &mh)  != 0)
    OUT << "NOT_EQ" << endl;

  //hipMemcpy(&h_tmp, (int *)d_A, sizeof(int), hipMemcpyDeviceToHost);
  //OUT << "after pinning: " << h_tmp << endl;

  init_data(size, h_mem, d_A);

  // call ixgbe_xmit_yhoon in ixgbe_main.c
  yhoon_xmit_arp();

  int count = 0;
  while(count < 1) {
    check_data(size, h_mem, d_A);
    usleep(1*1000*1000);
    count++;
  }
  OUT << "END" << endl;
  hipFree(d_A);

  return 0;
}
